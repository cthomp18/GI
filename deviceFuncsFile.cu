#include "hip/hip_runtime.h"
#include "deviceFuncsFile.cuh"

__device__ __noinline__ volatile float * volatile getMatInv(glm::vec3 iPt, SceneObject* obj, float *shF) {
   /*
   7: c
   8: s
   9: t
   */
   
   glm::mat3 matInv;
   glm::vec3 normal = obj->getNormal(obj, iPt, shF);
   glm::vec3 eRay = glm::vec3(0.0, 0.0, 1.0);
   
   normal = glm::normalize(normal);
   if (fabs(ELLIPSOID_SCALE - 1.0) > TOLERANCE &&
       (fabs(fabs(normal[0]) - eRay[0]) > TOLERANCE ||
        fabs(fabs(normal[1]) - eRay[1]) > TOLERANCE ||
        fabs(fabs(normal[2]) - eRay[2]) > TOLERANCE)) {
        
      glm::vec3 crossP = glm::cross(eRay, normal);
      crossP = glm::normalize(crossP);
      
      shF[6] = glm::dot(eRay, normal); shF[7] = sin(acos(glm::dot(eRay, normal))); shF[8] = 1.0 - shF[6];
      glm::mat3 mat = glm::mat3(shF[8]*crossP.x*crossP.x + shF[6], shF[8]*crossP.x*crossP.y - crossP.z*shF[7], shF[8]*crossP.x*crossP.z + crossP.y*shF[7],
                                shF[8]*crossP.x*crossP.y + crossP.z*shF[7], shF[8]*crossP.y*crossP.y + shF[6], shF[8]*crossP.y*crossP.z - crossP.x*shF[7],
                                shF[8]*crossP.x*crossP.z - crossP.y*shF[7], shF[8]*crossP.y*crossP.z + crossP.x*shF[7], shF[8]*crossP.z*crossP.z + shF[6]);
      matInv = glm::inverse(mat);
   } else {
      matInv = glm::mat3(1.0f);
   }
   
   //volatile float* volatile mInv;// = 
   return glm::value_ptr(matInv);
}

__device__ __noinline__ glm::vec3 accumulatePhotons(Photon **locateHeap, glm::vec3 iPt, SceneObject* obj, float *shF, int *shI) {
   glm::vec3 clr(0.0f, 0.0f, 0.0f);
   for (volatile int i = 0; i < shI[2]; i++) {
      //BRDF
      glm::vec3 normal = obj->getNormal(obj, iPt, shF);
      normal = glm::normalize(normal);
      float dotProd = glm::dot(-locateHeap[i]->incidence, normal);
      //if (shI[0] < causts) {
         //color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
      clr += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
      //} else {
         //color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
         //clr += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
      //}
   }
   return clr;
}

__device__ __noinline__ void getNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   glm::vec3 normal = obj->getNormal(obj, iPt, shF);
   normal = glm::normalize(normal);
   shF[3] = normal.x;
   shF[4] = normal.y;
   shF[5] = normal.z;
}
