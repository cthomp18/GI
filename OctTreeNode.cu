#include "hip/hip_runtime.h"
/*
   Cody Thompson
   CPE 473: Rendering
   Spring 2016
*/

#include "glm/glm.hpp"
#include "OctTreeNode.h"

#define TOLERANCE 0.001

using namespace std;

int sortAxisOct = 0;

bool sorterOct(SceneObject* s1, SceneObject* s2) { 
   if (sortAxisOct == 0) {
      return s1->boundingBox.middle.x < s2->boundingBox.middle.x;
   } else if (sortAxisOct == 1) {
      return s1->boundingBox.middle.y < s2->boundingBox.middle.y;
   } else if (sortAxisOct == 2) {
      return s1->boundingBox.middle.z < s2->boundingBox.middle.z;
   } else {
      std::cout << "fuck" << std::endl;
      return false; //fuck
   }
}

OctTreeNode::OctTreeNode(std::vector<SceneObject*> objects, int n, int depth) : SceneObject() {
   
   int middle, quarter, curInd = 0;
   uint octer;
   std::vector<SceneObject*> tempVect1, tempVect2, tempVect3;
   std::vector<SceneObject*> objs;
   std::vector<BoundingBox*> bbs;
   
   //SceneObject *hitObj = NULL;

   objs.clear();
   bbs.clear();
   if (!depth) { cout << "right " << sizeof(OctTreeNode) << endl; }
   for (int i = 0; i < 8; i++) {
   if (!depth && i == 7) cout << "which" << endl;
      octants[i] = NULL;
      if (!depth && i == 7) cout << "line" << endl;
      indeces[i] = -1;
      if (!depth && i == 7) cout << "dawg" << endl;
      objs.push_back(NULL);
   }
   if (!depth) { cout << "here" << endl; }
   /*std::cout << "Depth: " << depth << std::endl;
   for (int i = 0; i < objects.size(); i++) {
      objects[i]->printObj();
   }*/
   
   if (n <= 8) {
      //cout << n << endl;
      for (int i = 0; i < 8; i++) {
         if (i + 1 <= n) octants[i] = objs[i] = objects[i];
      }
      
      for (int i = 0; i < n; i++) {
         bbs.push_back(&(objs[i]->boundingBox));
      }
      
      boundingBox = combineBB(bbs);
   } else {      
      //std::cout << sortAxis << std::endl;
      //cout << n << endl;
      sortAxisOct = 0;
      std::sort(objects.begin(), objects.end(), sorterOct);
      sortAxisOct = 1;
      
      /*std::cout << "---------------New Recurse---------------------" << std::endl;
      std::cout << "SortAxis: " << sortAxis << std::endl;
      for (int i = 0; i < n; i++) {
         std::cout << objects[i]->boundingBox->middle.x() << " " << objects[i]->boundingBox->middle.y() << " " << objects[i]->boundingBox->middle.z() << std::endl;
      }*/
      
      middle = n / 2;
      quarter = middle / 2;
      octer = quarter / 2;
      
      tempVect1.clear();
      for (int i = 0; i < middle; i++) tempVect1.push_back(objects[i]);
      std::sort(tempVect1.begin(), tempVect1.end(), sorterOct);
      sortAxisOct = 2;
      
      tempVect2.clear();
      for (int i = 0; i < quarter; i++) tempVect2.push_back(tempVect1[i]);
      std::sort(tempVect2.begin(), tempVect2.end(), sorterOct);
      
      tempVect3.clear();
      for (uint i = 0; i < octer; i++) tempVect3.push_back(tempVect2[i]);
      if (octer) octants[curInd++] = new OctTreeNode(tempVect3, tempVect3.size(), depth + 1);
      
      tempVect3.clear();
      for (uint i = octer; i < tempVect2.size(); i++) tempVect3.push_back(tempVect2[i]);
      if (octer != tempVect2.size()) octants[curInd++] = new OctTreeNode(tempVect3, tempVect3.size(), depth + 1);
      
      tempVect2.clear();
      for (uint i = quarter; i < tempVect1.size(); i++) tempVect2.push_back(tempVect1[i]);
      std::sort(tempVect2.begin(), tempVect2.end(), sorterOct);
      sortAxisOct = 1;
      
      tempVect3.clear();
      for (uint i = 0; i < octer; i++) tempVect3.push_back(tempVect2[i]);
      if (octer) octants[curInd++] = new OctTreeNode(tempVect3, tempVect3.size(), depth + 1);
      
      tempVect3.clear();
      for (uint i = octer; i < tempVect2.size(); i++) tempVect3.push_back(tempVect2[i]);
      if (octer != tempVect2.size()) octants[curInd++] = new OctTreeNode(tempVect3, tempVect3.size(), depth + 1);
      
      
      tempVect1.clear();
      for (int i = middle; i < n; i++) tempVect1.push_back(objects[i]);
      std::sort(tempVect1.begin(), tempVect1.end(), sorterOct);
      sortAxisOct = 2;
      
      tempVect2.clear();
      for (int i = 0; i < quarter; i++) tempVect2.push_back(tempVect1[i]);
      std::sort(tempVect2.begin(), tempVect2.end(), sorterOct);
      
      tempVect3.clear();
      for (uint i = 0; i < octer; i++) tempVect3.push_back(tempVect2[i]);
      if (octer) octants[curInd++] = new OctTreeNode(tempVect3, tempVect3.size(), depth + 1);
      
      tempVect3.clear();
      for (uint i = octer; i < tempVect2.size(); i++) tempVect3.push_back(tempVect2[i]);
      if (octer != tempVect2.size()) octants[curInd++] = new OctTreeNode(tempVect3, tempVect3.size(), depth + 1);
            
      tempVect2.clear();
      for (uint i = quarter; i < tempVect1.size(); i++) tempVect2.push_back(tempVect1[i]);
      std::sort(tempVect2.begin(), tempVect2.end(), sorterOct);
      
      tempVect3.clear();
      for (uint i = 0; i < octer; i++) tempVect3.push_back(tempVect2[i]);
      if (octer) octants[curInd++] = new OctTreeNode(tempVect3, tempVect3.size(), depth + 1);
      
      tempVect3.clear();
      for (uint i = octer; i < tempVect2.size(); i++) tempVect3.push_back(tempVect2[i]);
      if (octer != tempVect2.size()) octants[curInd++] = new OctTreeNode(tempVect3, tempVect3.size(), depth + 1);
      
      for (int i = 0; i < curInd; i++) {
         bbs.push_back(&(octants[i]->boundingBox));
      }
      boundingBox = combineBB(bbs);
   }
   type = 8;
   checkCollision = (&checkOctTreeCollision);
   getNormal = (&getOctTreeNormal);
}

OctTreeNode::OctTreeNode(OctTreeNode* o) {
   int i;
   for (i = 0; i < 8; i++) {
      octants[i] = o->octants[i];
      indeces[i] = o->indeces[i];
   }
   copyData(o);
   checkCollision = (&checkOctTreeCollision);
   getNormal = (&getOctTreeNormal);
}

OctTreeNode::OctTreeNode() : SceneObject() {
   type = 8;
   //typedef float SceneObject::* fPtr;
   //checkCollision = static_cast<float (SceneObject::*)>(&OctTreeNode::collision);
   checkCollision = (&checkOctTreeCollision);
   getNormal = (&getOctTreeNormal);
}
OctTreeNode::~OctTreeNode() {
   int i;
   //printf("a suh dude\n");
   for (i = 0; i < 8; i++) {
      if (octants[i]) {
         delete octants[i];
      }
   }
}

/*float OctTreeNode::checkCollision2(glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
printf("otree collision\n");
return -1.0f;
   /*printf("pls\n");
   glm::vec4 startTransform;
   float t, tempT;
   SceneObject* tempObj;
   t = tempT = -1.0f;
   
   printf("Wanted to make sure\n");

  // if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   if (boundingBox.checkCollision(start, ray, time) < TOLERANCE) return -1.0f;
   
   if (octants[0]->transformed) {
      t = octants[0]->checkCollision(glm::vec3(octants[0]->transform * startTransform), glm::mat3(octants[0]->transform) * ray, time, &tempObj);
   } else {
      t = octants[0]->checkCollision(start, ray, time, &tempObj);
   }
   *object = tempObj;

   for (int i = 1; i < 8; i++) {
      if (octants[i]) {
         if (t >= TOLERANCE) {
            tempT = octants[i]->boundingBox.checkCollision(start, ray, time);
            if (tempT >= TOLERANCE && tempT < t) {
               if (octants[i]->transformed) {
                  tempT = octants[i]->checkCollision(glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj);
               } else {
                  tempT = octants[i]->checkCollision(start, ray, time, &tempObj);
               }
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  *object = tempObj;
               }
            }
         } else {
            t = octants[i]->checkCollision(glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj);
            *object = tempObj;
         }
      } //else break();
   }
   
   if (t < TOLERANCE) {
      *object = NULL;
   }
   
   return t;*/
//}
/*float OctTreeNode::checkCollision(glm::vec3 start, glm::vec3 ray, float time) {
   return -1.0f;
}*/

SceneObject* OctTreeNode::getObj() {
   return NULL;
}

/*float OctTreeNode::checkCollision(glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
//printf("%f %f %f\n", start.x, start.y, start.z);
//printf("%f %f %f\n", ray.x, ray.y, ray.z);
printf("otree collision\n");
//return -1.0f;
   //printf("pls\n");
   glm::vec4 startTransform;
   float t, tempT;
   //int i;
   SceneObject* tempObj;
   t = tempT = -1.0f;
   
   //printf("Wanted to make sure\n");

   //if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   //if (this == NULL) printf("Seriously what the fuck\n");
   printf("CURRENT MEM: %p\n", this);
   int pf = printf("Type? %d\n", this->type);
   printf("%d\n", pf);
   printf("blah? %d\n", this->blahblah);
   printf("amb? %f\n", this->ambient);
   printf("is it here?\n");
   if (boundingBox.checkCollision(start, ray, time) < TOLERANCE) { printf("bb not hit\n"); return -1.0f; }
   printf("nope\n");
   if (octants[0] == NULL) {
      printf("roll up in the club\n");
   } else {
      printf("like i got a fat...\n");
   }
   if (octants[0]) {
   if (octants[0]->transformed) {
      t = octants[0]->checkCollision(glm::vec3(octants[0]->transform * startTransform), glm::mat3(octants[0]->transform) * ray, time, &tempObj);
   } else {
      pf = printf("Type octant? %d\n", octants[0]->type);
      printf("%d\n", pf);
      printf("BLAH octant? %d\n", octants[0]->blahblah);
      printf("AMBi octant? %f\n", octants[0]->ambient);
      
      printf("OCTANT MEM: %p\n", &(this->octants[0]));
      octants[0]->type = 8;
      octants[0]->blahblah = 1;
      octants[0]->ambient = 4.0f;
      t = octants[0]->checkCollision(start, ray, time, &tempObj);
   }
   }
   *object = tempObj;
   printf("t: %f\n", t);
   printf("Type? %d\n", this->type);
   if (this->octants[1]) {
      printf("anything, really\n");
   } else {
      printf("pls\n");
   }
   printf("looking for something\n");
   
   int i = 0;
   for (i = 1; i < 8; i++) {
      printf("please\n");
      if (octants[i]) {
      printf("yo\n");
         if (t >= 0.001) {
            printf("hi\n");
            tempT = octants[i]->boundingBox.checkCollision(start, ray, time);
            if (tempT >= TOLERANCE && tempT < t) {
               if (octants[i]->transformed) {
                  tempT = octants[i]->checkCollision(glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj);
               } else {
                  printf("yo dawg\n");
                  if (octants[i]->type != 8) {
                     printf("triangle starting\n");
                  }
                  tempT = octants[i]->checkCollision(start, ray, time, &tempObj);
                  printf("k\n");
                  if (octants[i]->type != 8) {
                     printf("triangle ending\n");
                  }
               }
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  *object = tempObj;
               }
            }
         } else {
            t = octants[i]->checkCollision(glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj);
            *object = tempObj;
         }
      } //else break();
      printf("sup fam\n");
   }
   printf("whats happening\n");
   if (t < TOLERANCE) {
      *object = NULL;
   }
   
   return t;
}*/

/*CUDA_CALLABLE glm::vec3 OctTreeNode::getNormal(glm::vec3 iPt) {
   //std::cout << "Oh no! I fucked up!" << std::endl;
   printf("the fuck\n");
   return glm::vec3(0.0f, 0.0f, 0.0f);
}*/

/*void OctTreeNode::constructBB() {
   boundingBox = new Box(Eigen::Vector3f(position[0] - rad, position[1] - rad, position[2] - rad),
                         Eigen::Vector3f(position[0] + rad, position[1] + rad, position[2] + rad));
}*/

void OctTreeNode::printObj() {
   for (int i = 0; i < 8; i++) {
      if (octants[i]) octants[i]->printObj();
   }
}

BoundingBox OctTreeNode::combineBB(std::vector<BoundingBox*> boxes) {
   float xmin = boxes[0]->minPt[0], ymin = boxes[0]->minPt[1], zmin = boxes[0]->minPt[2];
   float xmax = boxes[0]->maxPt[0], ymax = boxes[0]->maxPt[1], zmax = boxes[0]->maxPt[2];
   for (uint i = 1; i < boxes.size(); i++) {
      xmin = fmin(xmin, boxes[i]->minPt[0]);
      ymin = fmin(ymin, boxes[i]->minPt[1]);
      zmin = fmin(zmin, boxes[i]->minPt[2]);
      xmax = fmax(xmax, boxes[i]->maxPt[0]);
      ymax = fmax(ymax, boxes[i]->maxPt[1]);
      zmax = fmax(zmax, boxes[i]->maxPt[2]);
   }
   return BoundingBox(glm::vec3(xmin, ymin, zmin), glm::vec3(xmax, ymax, zmax));
}

int OctTreeNode::treeLength() {
   int length = 1;
   
   for (int i = 0; i < 8; i++) {
      if (octants[i]) {
         if (octants[i]->type != 8) {
            length++;
         } else {
            length += static_cast<OctTreeNode*>(octants[i])->treeLength();
         }
      }
   }
   
   return length;
}

/*
This is done to transfer over most, if not all, objects into the GPU for CUDA.
Each node keeps track of its children through the indeces within the list.
The root node, once on the card, should be the very first element.
*/
void OctTreeNode::toSerialArray(Triangle* objectArray, int* currentIndex) {
   int i;
   int thisInd = *currentIndex;
   
   *currentIndex += 1;
   
   for (i = 0; i < 8; i++) {
      if (octants[i]) {
         indeces[i] = *currentIndex;
         if (octants[i]->type != 8) {
            if (*currentIndex >= 217450) printf("BAD ERROR ABORT\n");
            //printf("TYPE BURH: %d\n", octants[i]->type);
            assert(octants[i]->type == 2);
            memcpy(objectArray + (*currentIndex), octants[i], sizeof(Triangle));
            //printf("%p\n", (objectArray + (*currentIndex)));
            *currentIndex += 1;
         } else {
            reinterpret_cast<OctTreeNode*>(octants[i])->toSerialArray(objectArray, currentIndex);
         }
         //delete(octants[i]);
         //octants[i] = NULL;
      }
   }
   //printf("%d\n", *currentIndex);
   //Put the current node into the array at the end
   if (thisInd >= 217450) printf("BAD ERROR ABORT\n");
   memcpy(objectArray + thisInd, this, sizeof(OctTreeNode));
   //printf("%p\n", (objectArray + thisInd));
}
