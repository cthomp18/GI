#include "hip/hip_runtime.h"
/*
   Cody Thompson
   CPE 473: Rendering
   Spring 2016
*/

#include "glm/glm.hpp"
#include "QuadTreeNode.h"

#define TOLERANCE 0.001

using namespace std;

int sortAxisQuad = 0;

bool sorterQuad(SceneObject* s1, SceneObject* s2) { 
   if (sortAxisQuad == 0) {
      return s1->boundingBox.middle.x < s2->boundingBox.middle.x;
   } else if (sortAxisQuad == 2) {
      return s1->boundingBox.middle.z < s2->boundingBox.middle.z;
   } else {
      std::cout << "fuck" << std::endl;
      return false;
   }
}

QuadTreeNode::QuadTreeNode(std::vector<SceneObject*> objects, int n, int depth) : SceneObject() {
   int middle, quarter, curInd = 0;
   std::vector<SceneObject*> tempVect1, tempVect2;
   std::vector<SceneObject*> objs;
   std::vector<BoundingBox*> bbs;
   
   objs.clear();
   bbs.clear();
   
   for (int i = 0; i < 4; i++) {
      quadrants[i] = NULL;
      indeces[i] = -1;
      objs.push_back(NULL);
   }
   
   if (n <= 4) {
      for (int i = 0; i < 4; i++) {
         if (i + 1 <= n) quadrants[i] = objs[i] = objects[i];
      }
      
      for (int i = 0; i < n; i++) {
         bbs.push_back(&(objs[i]->boundingBox));
      }
      
      boundingBox = combineBB(bbs);
   } else {      
      sortAxisQuad = 0;
      std::sort(objects.begin(), objects.end(), sorterQuad);
      sortAxisQuad = 2;
      
      middle = n / 2;
      quarter = middle / 2;
      
      tempVect1.clear();
      for (int i = 0; i < middle; i++) tempVect1.push_back(objects[i]);
      std::sort(tempVect1.begin(), tempVect1.end(), sorterQuad);
      
      tempVect2.clear();
      for (int i = 0; i < quarter; i++) tempVect2.push_back(tempVect1[i]);
      if (quarter) quadrants[curInd++] = new QuadTreeNode(tempVect2, quarter, depth + 1);
      
      tempVect2.clear();
      for (int i = quarter; i < middle; i++) tempVect2.push_back(tempVect1[i]);
      if (quarter != tempVect1.size()) quadrants[curInd++] = new QuadTreeNode(tempVect2, tempVect2.size(), depth + 1);
      
      tempVect1.clear();
      for (int i = middle; i < n; i++) tempVect1.push_back(objects[i]);
      std::sort(tempVect1.begin(), tempVect1.end(), sorterQuad);
      
      tempVect2.clear();
      for (int i = 0; i < quarter; i++) tempVect2.push_back(tempVect1[i]);
      if (quarter) quadrants[curInd++] = new QuadTreeNode(tempVect2, quarter, depth + 1);
      
      tempVect2.clear();
      for (uint i = quarter; i < tempVect1.size(); i++) tempVect2.push_back(tempVect1[i]);
      if (quarter != tempVect1.size()) quadrants[curInd++] = new QuadTreeNode(tempVect2, tempVect2.size(), depth + 1);
      
      for (int i = 0; i < curInd; i++) {
         bbs.push_back(&(quadrants[i]->boundingBox));
      }
      boundingBox = combineBB(bbs);
   }
   type = 7;
   
   checkCollision = &(checkQuadTreeCollision);
   getNormal = &(getQuadTreeNormal);
}

QuadTreeNode::QuadTreeNode() : SceneObject() {}
QuadTreeNode::~QuadTreeNode() {
   for (int i = 0; i < 4; i++) {
      if (quadrants[i]) delete quadrants[i];
   }
}

void QuadTreeNode::printObj() {
   for (int i = 0; i < 4; i++) {
      if (quadrants[i]) quadrants[i]->printObj();
   }
}

BoundingBox QuadTreeNode::combineBB(std::vector<BoundingBox*> boxes) {
   float xmin = boxes[0]->minPt[0], ymin = boxes[0]->minPt[1], zmin = boxes[0]->minPt[2];
   float xmax = boxes[0]->maxPt[0], ymax = boxes[0]->maxPt[1], zmax = boxes[0]->maxPt[2];
   for (uint i = 1; i < boxes.size(); i++) {
      xmin = fmin(xmin, boxes[i]->minPt[0]);
      ymin = fmin(ymin, boxes[i]->minPt[1]);
      zmin = fmin(zmin, boxes[i]->minPt[2]);
      xmax = fmax(xmax, boxes[i]->maxPt[0]);
      ymax = fmax(ymax, boxes[i]->maxPt[1]);
      zmax = fmax(zmax, boxes[i]->maxPt[2]);
   }
   return BoundingBox(glm::vec3(xmin, ymin, zmin), glm::vec3(xmax, ymax, zmax));
}

int QuadTreeNode::treeLength() {
   int length = 1;
   
   for (int i = 0; i < 4; i++) {
      if (quadrants[i]) {
         if (quadrants[i]->type != 7) {
            length++;
         } else {
            length += static_cast<QuadTreeNode*>(quadrants[i])->treeLength();
         }
      }
   }
   
   return length;
}

void QuadTreeNode::toSerialArray(Triangle *objectArray, int *currentIndex) {
   int i;
   int thisInd = *currentIndex;
   
   *currentIndex += 1;
   
   for (i = 0; i < 4; i++) {
      if (quadrants[i]) {
         indeces[i] = *currentIndex;
         if (quadrants[i]->type != 7) {
            assert(quadrants[i]->type == 2);
            memcpy(objectArray + (*currentIndex), quadrants[i], sizeof(Triangle));
            *currentIndex += 1;
         } else {
            reinterpret_cast<QuadTreeNode*>(quadrants[i])->toSerialArray(objectArray, currentIndex);
         }
      }
   }
   
   memcpy(objectArray + thisInd, this, sizeof(QuadTreeNode));
}
