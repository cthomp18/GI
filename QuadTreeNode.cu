#include "hip/hip_runtime.h"
/*
   Cody Thompson
   CPE 473: Rendering
   Spring 2016
*/

#include "glm/glm.hpp"
#include "QuadTreeNode.h"

#define TOLERANCE 0.001

using namespace std;

int sortAxisQuad = 0;

bool sorterQuad(SceneObject* s1, SceneObject* s2) { 
   if (sortAxisQuad == 0) {
      return s1->boundingBox.middle.x < s2->boundingBox.middle.x;
   } else if (sortAxisQuad == 2) {
      return s1->boundingBox.middle.z < s2->boundingBox.middle.z;
   } else {
      std::cout << "fuck" << std::endl;
      return false; //fuck
   }
}

QuadTreeNode::QuadTreeNode(std::vector<SceneObject*> objects, int n, int depth) : SceneObject() {
   int middle, quarter;
   std::vector<SceneObject*> tempVect1, tempVect2;
   SceneObject *obj1, *obj2, *obj3, *obj4;
   
   q1 = q2 = q3 = q4 = obj1 = obj2 = obj3 = obj4 = NULL;
   
   /*std::cout << "Depth: " << depth << std::endl;
   for (int i = 0; i < objects.size(); i++) {
      objects[i]->printObj();
   }*/
   indeces[0] = indeces[1] = indeces[2] = indeces[3] = -1;
   if (n <= 4) {
      q1 = objects[0];
      obj1 = q1;
      if (n >= 2) q2 = obj2 = objects[1];
      else obj2 = objects[0];
      if (n >= 3) q3 = obj3 = objects[2];
      else obj3 = objects[0];
      if (n == 4) q4 = obj4 = objects[3];
      else obj4 = objects[0];

      boundingBox = combineBB(&(obj1->boundingBox), &(obj2->boundingBox), &(obj3->boundingBox), &(obj4->boundingBox));
   } else {      
      //std::cout << sortAxis << std::endl;
      sortAxisQuad = 0;
      std::sort(objects.begin(), objects.end(), sorterQuad);
      sortAxisQuad = 2;
      
      /*std::cout << "---------------New Recurse---------------------" << std::endl;
      std::cout << "SortAxis: " << sortAxis << std::endl;
      for (int i = 0; i < n; i++) {
         std::cout << objects[i]->boundingBox->middle.x() << " " << objects[i]->boundingBox->middle.y() << " " << objects[i]->boundingBox->middle.z() << std::endl;
      }*/
      
      middle = n / 2;
      quarter = middle / 2;
      
      tempVect1.clear();
      for (int i = 0; i < middle; i++) tempVect1.push_back(objects[i]);
      std::sort(tempVect1.begin(), tempVect1.end(), sorterQuad);
      
      tempVect2.clear();
      for (int i = 0; i < quarter; i++) tempVect2.push_back(tempVect1[i]);
      q1 = new QuadTreeNode(tempVect2, quarter, depth + 1);
      
      tempVect2.clear();
      for (int i = quarter; i < middle; i++) tempVect2.push_back(tempVect1[i]);
      q2 = new QuadTreeNode(tempVect2, tempVect2.size(), depth + 1);
      
      tempVect1.clear();
      for (int i = middle; i < n; i++) tempVect1.push_back(objects[i]);
      std::sort(tempVect1.begin(), tempVect1.end(), sorterQuad);
      
      tempVect2.clear();
      for (int i = 0; i < quarter; i++) tempVect2.push_back(tempVect1[i]);
      q3 = new QuadTreeNode(tempVect2, quarter, depth + 1);
      
      tempVect2.clear();
      for (uint i = quarter; i < tempVect1.size(); i++) tempVect2.push_back(tempVect1[i]);
      q4 = new QuadTreeNode(tempVect2, tempVect2.size(), depth + 1);
      
      boundingBox = combineBB(&(q1->boundingBox), &(q2->boundingBox), &(q3->boundingBox), &(q4->boundingBox));
   }
   type = 7;
   
   checkCollision = &(checkQuadTreeCollision);
   getNormal = &(getQuadTreeNormal);
}

QuadTreeNode::QuadTreeNode() : SceneObject() {}
QuadTreeNode::~QuadTreeNode() {
   if (q1) delete q1;
   if (q2) delete q2;
   if (q3) delete q3;
   if (q4) delete q4;
}

/*float QuadTreeNode::checkCollision(glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
   glm::vec4 startTransform;
   float t, tempT;
   SceneObject *obj1, *obj2, *obj3, *obj4;
   t = tempT = -1.0f;
   
  // if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   if (boundingBox.checkCollision(start, ray, time) < TOLERANCE) return -1.0f;
   

   if (q1->transformed) {
      t = q1->checkCollision(glm::vec3(q1->transform * startTransform), glm::mat3(q1->transform) * ray, time, &obj1);
   } else {
      t = q1->checkCollision(start, ray, time, &obj1);
   }
   *object = obj1;

   if (q2) {
      if (t >= TOLERANCE) {
         tempT = q2->boundingBox.checkCollision(start, ray, time);
         if (tempT >= TOLERANCE && tempT < t) {
            if (q2->transformed) {
               tempT = q2->checkCollision(glm::vec3(q2->transform * startTransform), glm::mat3(q2->transform) * ray, time, &obj2);
            } else {
               tempT = q2->checkCollision(start, ray, time, &obj2);
            }
            if (tempT >= TOLERANCE && tempT < t) {
               t = tempT;
               *object = obj2;
            }
         }
      } else {
         t = q2->checkCollision(glm::vec3(q2->transform * startTransform), glm::mat3(q2->transform) * ray, time, &obj2);
         *object = obj2;
      }
   }
   
   if (q3) {
      if (t >= TOLERANCE) {
         tempT = q3->boundingBox.checkCollision(start, ray, time);
         if (tempT >= TOLERANCE && tempT < t) {
            if (q3->transformed) {
               tempT = q3->checkCollision(glm::vec3(q3->transform * startTransform), glm::mat3(q3->transform) * ray, time, &obj3);
            } else {
               tempT = q3->checkCollision(start, ray, time, &obj3);
            }
            if (tempT >= TOLERANCE && tempT < t) {
               t = tempT;
               *object = obj3;
            }
         }
      } else {
         t = q3->checkCollision(glm::vec3(q3->transform * startTransform), glm::mat3(q3->transform) * ray, time, &obj3);
         *object = obj3;
      }
   }
   
   if (q4) {
      if (t >= TOLERANCE) {
         tempT = q4->boundingBox.checkCollision(start, ray, time);
         if (tempT >= TOLERANCE && tempT < t) {
            if (q4->transformed) {
               tempT = q4->checkCollision(glm::vec3(q4->transform * startTransform), glm::mat3(q4->transform) * ray, time, &obj4);
            } else {
               tempT = q4->checkCollision(start, ray, time, &obj4);
            }
            if (tempT >= TOLERANCE && tempT < t) {
               t = tempT;
               *object = obj4;
            }
         }
      } else {
         t = q4->checkCollision(glm::vec3(q4->transform * startTransform), glm::mat3(q4->transform) * ray, time, &obj4);
         *object = obj4;
      }
   }
   
   if (t < TOLERANCE) {
      *object = NULL;
   }
   
   return t;
}*/

/*glm::vec3 QuadTreeNode::getNormal(glm::vec3 iPt) {
   //std::cout << "Oh no! I fucked up!" << std::endl;
   return iPt;
}*/

/*void QuadTreeNode::constructBB() {
   boundingBox = new Box(Eigen::Vector3f(position[0] - rad, position[1] - rad, position[2] - rad),
                         Eigen::Vector3f(position[0] + rad, position[1] + rad, position[2] + rad));
}*/

void QuadTreeNode::printObj() {
   printf("hello?\n");
   if (q1->type == 2) { printf("wut\n"); }
   q1->printObj();
   if (q2) q2->printObj();
   if (q3) q3->printObj();
   if (q4) q4->printObj();
}

BoundingBox QuadTreeNode::combineBB(BoundingBox* box1, BoundingBox* box2, BoundingBox* box3, BoundingBox* box4) {
   return BoundingBox(glm::vec3(fmin(fmin(fmin(box1->minPt[0], box2->minPt[0]), box3->minPt[0]), box4->minPt[0]),
                                fmin(fmin(fmin(box1->minPt[1], box2->minPt[1]), box3->minPt[1]), box4->minPt[1]),
                                fmin(fmin(fmin(box1->minPt[2], box2->minPt[2]), box3->minPt[2]), box4->minPt[2])),
                      glm::vec3(fmax(fmax(fmax(box1->maxPt[0], box2->maxPt[0]), box3->maxPt[0]), box4->maxPt[0]),
                                fmax(fmax(fmax(box1->maxPt[1], box2->maxPt[1]), box3->maxPt[1]), box4->maxPt[1]),
                                fmax(fmax(fmax(box1->maxPt[2], box2->maxPt[2]), box3->maxPt[2]), box4->maxPt[2])));
}

int QuadTreeNode::treeLength() {
   int length = 1;
   
   if (q1) {
      if (q1->type != 7) {
         length++;
      } else {
         length += static_cast<QuadTreeNode*>(q1)->treeLength();
      }
   }
   if (q2) {
      if (q2->type != 7) {
         length++;
      } else {
         length += static_cast<QuadTreeNode*>(q2)->treeLength();
      }
   }
   if (q3) {
      if (q3->type != 7) {
         length++;
      } else {
         length += static_cast<QuadTreeNode*>(q3)->treeLength();
      }
   }
   if (q4) {
      if (q4->type != 7) {
         length++;
      } else {
         length += static_cast<QuadTreeNode*>(q4)->treeLength();
      }
   }

   return length;
}

void QuadTreeNode::toSerialArray(Triangle *objectArray, int *currentIndex) {
   int i;
   int thisInd = *currentIndex;
   
   *currentIndex += 1;
   
   if (q1) {
      indeces[0] = *currentIndex;
      if (q1->type != 7) {
         assert(q1->type == 2);
         memcpy(objectArray + (*currentIndex), q1, sizeof(Triangle));
         *currentIndex += 1;
      } else {
         reinterpret_cast<QuadTreeNode*>(q1)->toSerialArray(objectArray, currentIndex);
      }
   }
   if (q2) {
      indeces[1] = *currentIndex;
      if (q2->type != 7) {
         assert(q2->type == 2);
         memcpy(objectArray + (*currentIndex), q2, sizeof(Triangle));
         *currentIndex += 1;
      } else {
         reinterpret_cast<QuadTreeNode*>(q2)->toSerialArray(objectArray, currentIndex);
      }
   }
   if (q3) {
      indeces[2] = *currentIndex;
      if (q3->type != 7) {
         assert(q3->type == 2);
         memcpy(objectArray + (*currentIndex), q3, sizeof(Triangle));
         *currentIndex += 1;
      } else {
         reinterpret_cast<QuadTreeNode*>(q3)->toSerialArray(objectArray, currentIndex);
      }
   }
   if (q4) {
      indeces[3] = *currentIndex;
      if (q4->type != 7) {
         assert(q4->type == 2);
         memcpy(objectArray + (*currentIndex), q4, sizeof(Triangle));
         *currentIndex += 1;
      } else {
         reinterpret_cast<QuadTreeNode*>(q4)->toSerialArray(objectArray, currentIndex);
      }
   }
   //printf("%d\n", *currentIndex);
   //Put the current node into the array at the end
   //if (thisInd >= 217450) printf("BAD ERROR ABORT\n");
   memcpy(objectArray + thisInd, this, sizeof(QuadTreeNode));
   //printf("%p\n", (objectArray + thisInd));
}
