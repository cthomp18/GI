#include "hip/hip_runtime.h"
/*
   Cody Thompson
   Thesis
   Winter 2017
*/

#include "Box.h"

BoundingBox::BoundingBox(glm::vec3 cornerPt1, glm::vec3 cornerPt2) {
   for (int i = 0; i < 3; i++) {
      if (cornerPt1[i] < cornerPt2[i]) {
         minPt[i] = cornerPt1[i];
         maxPt[i] = cornerPt2[i];
      } else {
         minPt[i] = cornerPt2[i];
         maxPt[i] = cornerPt1[i];
      }
   }
   middle = glm::vec3((maxPt[0] + minPt[0]) / 2.0f, (maxPt[1] + minPt[1]) / 2.0f, (maxPt[2] + minPt[2]) / 2.0f);
}

BoundingBox::BoundingBox() {}
BoundingBox::~BoundingBox() {}

float BoundingBox::checkCollision(glm::vec3 ray, float time, int *shI, float *shF) {
   float tgmin = FLT_MIN, tgmax = FLT_MAX, t1, t2, temp, t = -1.0f;
   glm::vec3 start(shF[6], shF[7], shF[8]);
   for (int i = 0; i < 3; i++) {
      temp = start[i];
      
      if (fabs(ray[i]) < TOLERANCE) { // Ray along 2D Plane checks
         if (temp > maxPt[i] || temp < minPt[i]) return -1.0f;
      }
      
      t1 = (minPt[i] - temp) / ray[i];
      t2 = (maxPt[i] - temp) / ray[i];
      if (t2 < t1) {
         temp = t2;
         t2 = t1;
         t1 = temp;
      }
      if (t1 > tgmin) tgmin = t1;
      if (t2 < tgmax) tgmax = t2;
   }
   
   if (tgmin > tgmax) return -1.0f;
   if (tgmin < TOLERANCE) return tgmax;
   return tgmin;
}

float BoundingBox::checkCollision(glm::vec3 start, glm::vec3 ray, float time) {
   //std::cout << "Box Collision" << std::endl;
   float tgmin = FLT_MIN, tgmax = FLT_MAX, t1, t2, temp, t = -1.0f;

   for (int i = 0; i < 3; i++) {
      temp = start[i];
      
      if (fabs(ray[i]) < TOLERANCE) { // Ray along 2D Plane checks
         if (temp > maxPt[i] || temp < minPt[i]) return -1.0f;
      }
      
      t1 = (minPt[i] - temp) / ray[i];
      t2 = (maxPt[i] - temp) / ray[i];
      if (t2 < t1) {
         temp = t2;
         t2 = t1;
         t1 = temp;
      }
      if (t1 > tgmin) tgmin = t1;
      if (t2 < tgmax) tgmax = t2;
   }
   
   /*if (start.x() >= minPt.x() && start.x() <= maxPt.x() &&
       start.y() >= minPt.y() && start.y() <= maxPt.y() && 
       start.z() >= minPt.z() && start.z() <= maxPt.z()) {
       t = 10.0f;
       
       std::cout << "Inside!!!" << std::endl;
   }*/
   //if (tgmin < TOLERANCE) return new Collision(tgmin, this);
   //if (tgmin > tgmax || tgmax < 0.001f) return new Collision(t, this);
   if (tgmin > tgmax) return -1.0f;
   if (tgmin < TOLERANCE) return tgmax;
   return tgmin;
}
