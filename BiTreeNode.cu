#include "hip/hip_runtime.h"
/*
   Cody Thompson
   CPE 473: Rendering
   Spring 2016
*/

#include "BiTreeNode.h"

#define TOLERANCE 0.001

using namespace std;

int sortAxisBi = 0;

bool sorterBi(SceneObject* s1, SceneObject* s2) { 
   if (sortAxisBi == 0) {
      return s1->boundingBox.middle.x < s2->boundingBox.middle.x;
   } else if (sortAxisBi == 1) {
      return s1->boundingBox.middle.y < s2->boundingBox.middle.y;
   } else if (sortAxisBi == 2) {
      return s1->boundingBox.middle.z < s2->boundingBox.middle.z;
   } else {
      std::cout << "fuck" << std::endl;
      return false; //fuck
   }
}

BiTreeNode::BiTreeNode(std::vector<SceneObject*> objects, int axis, int n) : SceneObject() {
   int middle;
   std::vector<SceneObject*> tempVect1, tempVect2;
   
   sortAxisBi = 0;
   if (n == 1) {
      left = objects[0];
      right = NULL;
      boundingBox = objects[0]->boundingBox;
   } else if (n == 2) {
      left = objects[0];
      right = objects[1];
      boundingBox = combineBB(&(objects[0]->boundingBox), &(objects[1]->boundingBox));
   } else {      
      //tempVect.clear();
      float minx, miny, minz, maxx, maxy, maxz;
      minx = miny = minz = FLT_MAX;
      maxx = maxy = maxz = FLT_MIN;
      glm::vec3 mid;
      for (int i = 0; i < n; i++) {
         mid = objects[i]->boundingBox.middle;
         if (minx > mid.x) minx = mid.x;
         if (maxx < mid.x) maxx = mid.x;
         if (miny > mid.y) miny = mid.y;
         if (maxy < mid.y) maxy = mid.y;
         if (minz > mid.z) minz = mid.z;
         if (maxz < mid.z) maxz = mid.z;
      }
      
      if (maxx - minx > maxy - miny && maxx - minx > maxz - minz) {
         sortAxisBi = 0;
      } else {
         if (maxy - miny > maxz - minz) {
            sortAxisBi = 1;
         } else {
            sortAxisBi = 2;
         }
      }
      //std::cout << "---------------Un   Sorted---------------------" << std::endl;
      
      //std::cout << sortAxis << std::endl;
      std::sort(objects.begin(), objects.end(), sorterBi);
      
      /*std::cout << "---------------New Recurse---------------------" << std::endl;
      std::cout << "SortAxis: " << sortAxis << std::endl;
      for (int i = 0; i < n; i++) {
         std::cout << objects[i]->boundingBox->middle.x() << " " << objects[i]->boundingBox->middle.y() << " " << objects[i]->boundingBox->middle.z() << std::endl;
      }*/
      
      middle = n / 2;
      tempVect1.clear();
      tempVect2.clear();

      for (int i = 0; i < middle; i++) {
         tempVect1.push_back(objects[i]);
      }
      for (int i = middle; i < n; i++) {
         tempVect2.push_back(objects[i]);
      }

      //std::cout << n << " " << middle << std::endl;
      left = new BiTreeNode(tempVect1, (axis + 1) % 3, tempVect1.size());
      right = new BiTreeNode(tempVect2, (axis + 1) % 3, tempVect2.size());
      
      boundingBox = combineBB(&(left->boundingBox), &(right->boundingBox));
   }
   type = 8;
}

BiTreeNode::BiTreeNode() : SceneObject() {}
BiTreeNode::~BiTreeNode() {}

float BiTreeNode::checkCollision(glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
   glm::vec4 startTransform;
   float t, tLeft, tRight;
   SceneObject *lObj, *rObj;
   t = tLeft = tRight = -1.0f;
   
  // if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   if (boundingBox.checkCollision(start, ray, time) < TOLERANCE) return -1.0f;
   //if (left) {
   if (left->transformed) {
      tLeft = left->checkCollision(glm::vec3(left->transform * startTransform), glm::mat3(left->transform) * ray, time, &lObj);
   } else {
      tLeft = left->checkCollision(start, ray, time, &lObj);
   }
   if (right) {
      if (tLeft >= TOLERANCE) {
         tRight = right->boundingBox.checkCollision(start, ray, time);
         if (tRight >= TOLERANCE && tRight < tLeft) {
            if (right->transformed) {
               tRight = right->checkCollision(glm::vec3(right->transform * startTransform), glm::mat3(right->transform) * ray, time, &rObj);
            } else {
               tRight = right->checkCollision(start, ray, time, &rObj);
            }
         } else {
            tRight = -1.0f;
         }
      } else {
         tRight = right->checkCollision(glm::vec3(right->transform * startTransform), glm::mat3(right->transform) * ray, time, &rObj);
      }
   }
   //}
   
   if (tLeft < TOLERANCE) {
      if (tRight >= TOLERANCE) {
         t = tRight;
         *object = rObj;
      }
   } else {
      if (tRight < TOLERANCE) {
         t = tLeft;
         *object = lObj;
      } else {
         if (tLeft < tRight) {
            t = tLeft;
            *object = lObj;
         } else {
            t = tRight;
            *object = rObj;
         }
      }
   }
   
   return t;
}

glm::vec3 BiTreeNode::getNormal(glm::vec3 iPt) {
   //std::cout << "Oh no! I fucked up!" << std::endl;
   return glm::vec3(0.0f, 0.0f, 0.0f);
}

/*void BiTreeNode::constructBB() {
   boundingBox = new Box(Eigen::Vector3f(position[0] - rad, position[1] - rad, position[2] - rad),
                         Eigen::Vector3f(position[0] + rad, position[1] + rad, position[2] + rad));
}*/

void BiTreeNode::printTree() {
   
}

BoundingBox BiTreeNode::combineBB(BoundingBox* box1, BoundingBox* box2) {
   return BoundingBox(glm::vec3(fmin(box1->minPt[0], box2->minPt[0]), fmin(box1->minPt[1], box2->minPt[1]), fmin(box1->minPt[2], box2->minPt[2])),
                      glm::vec3(fmax(box1->maxPt[0], box2->maxPt[0]), fmax(box1->maxPt[1], box2->maxPt[1]), fmax(box1->maxPt[2], box2->maxPt[2])));
}
