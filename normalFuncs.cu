#include "hip/hip_runtime.h"
#include "normalFuncs.h"

glm::vec3 getOctTreeNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   OctTreeNode *thisObj = reinterpret_cast<OctTreeNode*>(obj);
   
   printf("the fuck\n");
   return glm::vec3(0.0f, 0.0f, 0.0f);
}

// glm::vec3 getOctTreeNormal2(SceneObject *thisObj, glm::vec3 iPt, float time);

//See detailed function below
glm::vec3 getTriNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   //F1: intersect point x
   //F2: intersect point y
   //F3: intersect point z
   //F4: alpha
   //F5: beta
   //F6: gamma
   //F7: dot product ABP
   //F8: dot product ACP
   
   Triangle *thisObj = reinterpret_cast<Triangle*>(obj);
   
   shF[1] = iPt.x;
   shF[2] = iPt.y;
   shF[3] = iPt.z;
   
   
   glm::vec3 normalConstructor = thisObj->normal;

   shF[7] = (thisObj->b.x - thisObj->a.x) * (shF[1] - thisObj->a.x);
   shF[7] += (thisObj->b.y - thisObj->a.y) * (shF[2] - thisObj->a.y);
   shF[7] += (thisObj->b.z - thisObj->a.z) * (shF[3] - thisObj->a.z);
   shF[8] = (thisObj->c.x - thisObj->a.x) * (shF[1] - thisObj->a.x);
   shF[8] += (thisObj->c.y - thisObj->a.y) * (shF[2] - thisObj->a.y);
   shF[8] += (thisObj->c.z - thisObj->a.z) * (shF[3] - thisObj->a.z);

   shF[4] = ((thisObj->dotAC * shF[7]) - (thisObj->dotABC * shF[8])) * thisObj->multiplier;
   shF[5] = ((thisObj->dotAB * shF[8]) - (thisObj->dotABC * shF[7])) * thisObj->multiplier;
   
   shF[6] = (1.0f - shF[4]) - shF[5];

   normalConstructor = thisObj->bNor * shF[4];
   normalConstructor += thisObj->cNor * shF[5];
   normalConstructor += thisObj->aNor * shF[6];
   
   normalConstructor = glm::normalize(normalConstructor);
   
   return normalConstructor;
}

/*glm::vec3 getTriNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   Triangle *thisObj = reinterpret_cast<Triangle*>(obj);
   
   float alpha, beta, gamma;
   float multiplier = thisObj->multiplier;
   float dotAC = thisObj->dotAC;
   float dotAB = thisObj->dotAB;
   float dotABC = thisObj->dotABC;
   glm::vec3 a = thisObj->a;
   glm::vec3 b = thisObj->b;
   glm::vec3 c = thisObj->c;
   glm::vec3 aNor = thisObj->aNor;
   glm::vec3 bNor = thisObj->bNor;
   glm::vec3 cNor = thisObj->cNor;
   
   glm::vec3 normalConstructor = thisObj->normal;
   if (thisObj->smooth) {
      float dotABP = glm::dot(b - a, iPt - a), dotACP = glm::dot(c - a, iPt - a);
      alpha = ((dotAC * dotABP) - (dotABC * dotACP)) * multiplier;
      beta = ((dotAB * dotACP) - (dotABC * dotABP)) * multiplier;
      gamma = (1.0f - alpha) - beta;
      
      normalConstructor = bNor * alpha;
      normalConstructor += cNor * beta;
      normalConstructor += aNor * gamma;
      normalConstructor = glm::normalize(normalConstructor);
   }
   
   return normalConstructor;
}*/

glm::vec3 getSphereNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   Sphere *thisObj = reinterpret_cast<Sphere*>(obj);
   glm::vec3 returnvec = (iPt - thisObj->position) / thisObj->radius;
   return returnvec;
}

glm::vec3 getPlaneNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   Plane *thisObj = reinterpret_cast<Plane*>(obj);
   
   return thisObj->normal;
}

glm::vec3 getBiTreeNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   BiTreeNode *thisObj = reinterpret_cast<BiTreeNode*>(obj);
   
   //std::cout << "Oh no! I fucked up!" << std::endl;
   return iPt;
}

glm::vec3 getBoxNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   Box *thisObj = reinterpret_cast<Box*>(obj);
   
   glm::vec3 normal = glm::vec3(0.0f, 0.0f, 0.0f);
   glm::vec3 minPt = thisObj->minPt;
   glm::vec3 maxPt = thisObj->maxPt;
   
   for (int i = 0; i < 3; i++) {
      if (fabs(iPt[i] - minPt[i]) < TOLERANCE) {
         normal[i] = -1.0f;
      } else if (fabs(iPt[i] - maxPt[i]) < TOLERANCE) {
         normal[i] = 1.0f;
      }
   }
   
   return normal;
}

//See better function below
//Currently this is at 34 regs, doesn't seem to be any fucking way to make it less (i already tried removing iPt as a parameter)
glm::vec3 getGWNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   //start:
   //F0: time
   //middle:
   //F2: temp vect x
   //F3: temp vect z
   //F4: intersect point x
   //F5: intersect point z
   //F6: wa
   //F7: wdpt
   //F8: coswa
   
   volatile GerstnerWave *thisObj = reinterpret_cast<volatile GerstnerWave*>(obj);
   glm::vec3 normal = glm::vec3(0.0f, 1.0f, 0.0f);

   shF[4] = iPt.x;
   shF[5] = iPt.z;
   
   volatile int i = 0;
   for ( ; i < thisObj->waves; i++) {
      //if (1) {
         //volatile float f1 = thisObj->frequency[i];
         //volatile float f2 = thisObj->amplitude[i];
         shF[6] = thisObj->frequency[i] * thisObj->amplitude[i];
         //shF[6] = f1 * f2;
         shF[2] = thisObj->direction[i].x * thisObj->frequency[i];
         //shF[2] = thisObj->direction[i].x * f1;
         shF[3] = thisObj->direction[i].z * thisObj->frequency[i];
         //shF[3] = thisObj->direction[i].z * f1;
         shF[7] = shF[2] * shF[4] + shF[3] * shF[5];
         shF[7] = shF[7] + shF[0] * thisObj->speedPC[i];
      //}
      //if (1) {
         //f1 = thisObj->speedPC[i];
         
         //volatile float f1 = cos(shF[7]);
         shF[8] = shF[6] * cos(shF[7]);
      //}
      //normal -= glm::vec3(thisObj->direction[i].x * shF[8], (thisObj->steepness[i] * shF[6] * sin(shF[7])), thisObj->direction[i].z * shF[8]);
      normal.x -= thisObj->direction[i].x * shF[8];
      normal.y -= thisObj->steepness[i] * shF[6] * sin(shF[7]);
      normal.z -= thisObj->direction[i].z * shF[8];
   }
   
   normal = glm::normalize(normal);
   
   return normal;
}

/*glm::vec3 getGWNormal(SceneObject *obj, glm::vec3 iPt, int *shI, float *shF) {
   GerstnerWave *thisObj = reinterpret_cast<GerstnerWave*>(obj);
   
   glm::vec3 normal = glm::vec3(0.0f, 1.0f, 0.0f);
   float wa, wdpt, coswa;
   
   int waves = thisObj->waves;
   
   for (int i = 0; i < waves; i++) {
      wa = thisObj->frequency[i] * thisObj->amplitude[i];
      wdpt = glm::dot(thisObj->frequency[i] * glm::vec3(thisObj->direction[i].x, 0.0f, thisObj->direction[i].z), iPt) + (shF[0] * thisObj->speedPC[i]);
      coswa = wa * cos(wdpt);
      normal -= glm::vec3(thisObj->direction[i].x * coswa, (thisObj->steepness[i] * wa * sin(wdpt)), thisObj->direction[i].z * coswa);
   }
   normal = glm::normalize(normal);
   
   return normal;
}*/

glm::vec3 getConeNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   Cone *thisObj = reinterpret_cast<Cone*>(obj);
   
   return glm::vec3(-1.0f, -1.0f, -1.0f);
}

glm::vec3 getQuadTreeNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   QuadTreeNode *thisObj = reinterpret_cast<QuadTreeNode*>(obj);
   
   return iPt;
}
