#include "hip/hip_runtime.h"
#include "normalFuncs.h"

glm::vec3 getOctTreeNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   //std::cout << "Oh no! I fucked up!" << std::endl;
   OctTreeNode *thisObj = reinterpret_cast<OctTreeNode*>(obj);
   
   printf("the fuck\n");
   return glm::vec3(0.0f, 0.0f, 0.0f);
}

// glm::vec3 getOctTreeNormal2(SceneObject *thisObj, glm::vec3 iPt, float time);

glm::vec3 getTriNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   Triangle *thisObj = reinterpret_cast<Triangle*>(obj);
   
   float alpha, beta, gamma;
   float multiplier = thisObj->multiplier;
   float dotAC = thisObj->dotAC;
   float dotAB = thisObj->dotAB;
   float dotABC = thisObj->dotABC;
   glm::vec3 a = thisObj->a;
   glm::vec3 b = thisObj->b;
   glm::vec3 c = thisObj->c;
   glm::vec3 aNor = thisObj->aNor;
   glm::vec3 bNor = thisObj->bNor;
   glm::vec3 cNor = thisObj->cNor;
   
   glm::vec3 normalConstructor = thisObj->normal;
   //printf("sup dawg\n");
   if (thisObj->smooth) {
      float dotABP = glm::dot(b - a, iPt - a), dotACP = glm::dot(c - a, iPt - a);
      alpha = ((dotAC * dotABP) - (dotABC * dotACP)) * multiplier;
      beta = ((dotAB * dotACP) - (dotABC * dotABP)) * multiplier;
      gamma = (1.0f - alpha) - beta;
      
      normalConstructor = bNor * alpha;
      normalConstructor += cNor * beta;
      normalConstructor += aNor * gamma;
      normalConstructor = glm::normalize(normalConstructor);
   }
   
   return normalConstructor;
}

glm::vec3 getSphereNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   Sphere *thisObj = reinterpret_cast<Sphere*>(obj);
   
   return (iPt - thisObj->position) / thisObj->radius;
}

glm::vec3 getPlaneNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   Plane *thisObj = reinterpret_cast<Plane*>(obj);
   
   return thisObj->normal;
}

glm::vec3 getBiTreeNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   BiTreeNode *thisObj = reinterpret_cast<BiTreeNode*>(obj);
   
   //std::cout << "Oh no! I fucked up!" << std::endl;
   return iPt;
}

glm::vec3 getBoxNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   Box *thisObj = reinterpret_cast<Box*>(obj);
   
   glm::vec3 normal = glm::vec3(0.0f, 0.0f, 0.0f);
   glm::vec3 minPt = thisObj->minPt;
   glm::vec3 maxPt = thisObj->maxPt;
   
   for (int i = 0; i < 3; i++) {
      if (fabs(iPt[i] - minPt[i]) < TOLERANCE) {
         normal[i] = -1.0f;
      } else if (fabs(iPt[i] - maxPt[i]) < TOLERANCE) {
         normal[i] = 1.0f;
      }
   }
   
   return normal;
}

glm::vec3 getGWNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   GerstnerWave *thisObj = reinterpret_cast<GerstnerWave*>(obj);
   
   glm::vec3 normal = glm::vec3(0.0f, 1.0f, 0.0f);
   float wa, wdpt, coswa;
   
   int waves = thisObj->waves;
   float *frequency = thisObj->frequency;
   float *amplitude = thisObj->amplitude;
   glm::vec3 *direction = thisObj->direction;
   float *speedPC = thisObj->speedPC;
   float *steepness = thisObj->steepness;
   
   for (int i = 0; i < waves; i++) {
      wa = frequency[i] * amplitude[i];
      wdpt = glm::dot(frequency[i] * glm::vec3(direction[i].x, 0.0f, direction[i].z), iPt) + (time * speedPC[i]);
      coswa = wa * cos(wdpt);
      normal -= glm::vec3(direction[i].x * coswa, (steepness[i] * wa * sin(wdpt)), direction[i].z * coswa);
   }
   normal = glm::normalize(normal);
   
   return normal;
}

glm::vec3 getConeNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   Cone *thisObj = reinterpret_cast<Cone*>(obj);
   
   return glm::vec3(-1.0f, -1.0f, -1.0f);
}

glm::vec3 getQuadTreeNormal(SceneObject *obj, glm::vec3 iPt, float time) {
   QuadTreeNode *thisObj = reinterpret_cast<QuadTreeNode*>(obj);
   
   //std::cout << "Oh no! I fucked up!" << std::endl;
   return iPt;
}
