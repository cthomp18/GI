#include "hip/hip_runtime.h"
#include "tracer.h"

//Chris Lupo's error handling fuction/macro
static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void toOctTree(Triangle *objectArray, int size, int gridDimension) {
   
   OctTreeNode tempO;
   Triangle* tempT;
   int threadInd = (blockIdx.y * (gridDimension * TILEWIDTH) * TILEWIDTH) + (blockIdx.x * TILEWIDTH) +
                   (threadIdx.y * (gridDimension * TILEWIDTH)) + threadIdx.x;//blockIdx.x;//*TILEWIDTH + threadIdx.x;
   int i;

   /*if (threadInd == 0) {
      printf("sup homie\n");
      tempO = new OctTreeNode((OctTreeNode*)(&(objectArray[0])));
      printf("suh\n");
      memcpy(objectArray, tempO, sizeof(OctTreeNode));
      printf(":)\n");
      delete(tempO);
   }*/
   //printf("%d\n", threadInd);
   //printf("xd\n");
   if (threadInd == 0) {
      printf(":(\n");
   if (threadInd == 0) { printf(":() %d\n", size); }
   if (threadInd == size - 1) { printf("UH SUH DUDE %d\n", size); }
   //__syncthreads();
   if (threadInd < size) {
      /*if (threadInd == size - 1) {
         printf("Thread Index: %d\n", threadInd);
         printf("Block? %d\n", blockIdx.x);
      }*/
      printf("suh\n");
      if (objectArray[threadInd].type == 8) {
         tempO = OctTreeNode((OctTreeNode*)(&(objectArray[threadInd])));
         /*for (i = 0; i < 8; i++) {
            tempO->octants[i] = NULL;
         }*/
         
         //if (threadInd == 0) {
         //printf("B)\n");
         //tempO = (OctTreeNode*)(objectArray + threadInd);
         //printf("xd\n");
         printf("fam\n");
         for (i = 0; i < 8; i++) {
            /*if (threadInd < 16) {
               printf("Index: %d\n", tempO->indeces[i]);
            }*/
            if (tempO.indeces[i] != -1) {
               printf("fuq\n");
               /*if (tempO->octants[i] == NULL) {
                  printf("Cool: %d\n", i);
               }*/
               
               tempO.octants[i] = (SceneObject*)(&(objectArray[tempO.indeces[i]]));
               /*if (tempO->octants[i] == NULL) {
                  printf("Not Cool: %d\n", i);
               }*/
               
            } else {
               tempO.octants[i] = NULL;
            }
         }
         if (threadInd == 1) {
            //printf("Index: %d\n", tempO->indeces[i]);
            /*Index: 1
Index: 27182
Index: 54363
Index: 81544
Index: 108725
Index: 135906
Index: 163087
Index: 190268
*/
/*Index: 2
Index: 3399
Index: 6797
Index: 10194
Index: 13592
Index: 16989
Index: 20387
Index: 23784
*/
            //s.y = 1.0;
            //printf("%f %f %f\n", s.x, s.y, s.z);
//printf("%f %f %f\n", t.x, t.y, t.z);
            //tempO->checkCollision(s, t, 0.0f, (SceneObject**)&tempT);
         }
         printf("diddly\n");
         memcpy(&(objectArray[threadInd]), &tempO, sizeof(OctTreeNode));
         printf("ding\n");
         //delete tempO;
         printf("dong\n");
         //}
      } else {
         //printf("Type: %d\n", objectArray[threadInd].type);
         tempT = new Triangle((Triangle*)(&(objectArray[threadInd])));
         memcpy(&(objectArray[threadInd]), tempT, sizeof(Triangle));
         delete tempT;
      }
      printf("damn\n");
   } else {
      printf("WHAT\n");
      printf("%d\n", threadInd);
      printf("%d\n", size);
      printf("COOL\n");
   }
   //printf("DAWG\n");
   //__syncthreads();
   //printf("%d\n", threadInd);
   //printf("WHAT\n", threadInd);
   if (threadInd == 1) {
      //printf("THE Fasdfadsf\n");
      //printf("OCTANT: %d\n", (&(objectArray[1]))->type);
      //printf("OCTANT: %d\n", ((OctTreeNode*)(&(objectArray[0])))->octants[0]->type);
      tempO = (OctTreeNode*)(&(objectArray[0]));
      
      for (i = 0; i < 8; i++) {
         /*if (tempO->octants[i]) {
            printf("good %d\n", tempO->indeces[i]);
         } else {
            printf("bad %d\n", tempO->indeces[i]);
         }*/
      }
         SceneObject **so;
         //tempO->checkCollision(s, t, 0.0f, so);
   }
   //memcpy(&(objectArray[threadInd]), tempO, sizeof(OctTreeNode));
   //printf("THE FUCK\n");
   //__syncthreads();
   }
   __syncthreads();
   //printf("FUCK\n");
}

__global__ void GIPhotonMapKernel(SceneObject **objArr, int *objSizes, int objSize, Pixel *pixelsD, Camera *camera, int width, int height, RayTracer raytracer) {
   //__shared__ TYPE Mds[TILEWIDTH][TILEWIDTH];
   //__shared__ TYPE Nds[TILEWIDTH][TILEWIDTH];
   int row = blockIdx.y*TILEWIDTH + threadIdx.y;
   int col = blockIdx.x*TILEWIDTH + threadIdx.x;
   if (row == 320 && col == 320) {
   //int c = 0;
   //printf("yo\n");
   glm::vec3 cPos = camera->getPosition();
   //glm::vec3 cPos = glm::vec3(0.0, 0.0, 0.0);
   //RayTracer* raytrace = new RayTracer(objArr, 0, 0, 0, 0, 0);
   //delete(raytrace);
   int currentImgInd;
   
   //Collision* col = new Collision();
   //delete(col);
   //for (int i = 0; i < width / TILEWIDTH; i++) {
   //i * imgheight + j;
      //int threadIndex = row*width1 + (m*TILEWIDTH + threadIdx.x);
      //Nds[threadIdx.y][threadIdx.x] = Nd[col + (m*TILEWIDTH + threadIdx.y)*width2];
      //cout << "i: " << i << endl;
   Collision* collision;
   glm::vec3 ray, tempColor;
      //for (int j = 0; j < height; j++) {
   currentImgInd = col * height + row;
   ray = pixelsD[currentImgInd].pt;// - cPos;
   ray = glm::normalize(ray);

   /*if (currentImgInd == 0) {
      //printf("First Obj Type: %d\n", objArr[0]->type);
      OctTreeNode* o = (OctTreeNode*)objArr[0];
      
      printf("Huh %d\n", o->type);
      printf("ImgInd: %d\n", currentImgInd);
      
      for(int i = 0; i < 8; i++) {
         if (o->octants[i] != NULL) {
            printf("Oct Type: %d\n", o->octants[i]->type);
         } else {
            printf("Null Oct: %d\n", i);
         }
      }
      printf("Tree size: %d\n", o->treeLength());
   }*/
//if (row == 320 && col == 320) {
   printf("hello?\n");
   collision = raytracer.trace(cPos, ray, false);
   printf("whats up?\n");
         //printf("Making sure ;)\n");
   if (collision->time > TOLERANCE) {
      //pixels[i][j].clr = raytrace->calcRadiance(cPos, cPos + ray * col->time, col->object, unit, 1.0f, 1.33f, 0.95f, 5); //Cam must start in air
      tempColor = collision->object->getNormal(cPos + ray * collision->time, 2.0f);
      pixelsD[currentImgInd].clr = (tempColor * 0.5f) + 0.5f;
      //printf("Anything here?\n");
   }
   else {
      pixelsD[currentImgInd].clr = glm::vec3(1.0f, 1.0f, 1.0f);
      //printf("Making sure\n");
   }
   
   delete(collision);
}
   //}
         //cout << "PIXCOL: " << pixels[i][j].clr.r << " " << pixels[i][j].clr.g << " " << pixels[i][j].clr.b << endl;
      //}
   //}
   __syncthreads();
   
   
   /*for (int i = 0; i < width * height; i++) {
      pixelsD[i].clr.x = pixelsD[i].clr.y = pixelsD[i].clr.z = 1.0;
   }*/
}

// Set the card up to run cuda
void RayTraceOnDevice(int width, int height, Pixel *pixels, std::vector<SceneObject*> objects, Camera *cam) {
   SceneObject** objArrD;
   SceneObject** objArrH;// = &objects[0];
   int* sizesH;
   int* sizesD;
   int gridDimension;
   
   Triangle* tempOctTree;
   Camera* cameraD;
   Pixel* pixelsD;
   RayTracer* raytracer;
   int tempSize, tempInd, pixelArrSize;
   
   dim3 dimGrid;
   dim3 dimBlock;
   
   pixelArrSize = width * height * sizeof(Pixel);
   printf("Transferring Data...\n");
   printf("Size parray: %d\n", pixelArrSize);

   HANDLE_ERROR(hipMalloc(&pixelsD, pixelArrSize));
   HANDLE_ERROR(hipMemcpy(pixelsD, pixels, pixelArrSize, hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&cameraD, sizeof(Camera)));
   HANDLE_ERROR(hipMemcpy(cameraD, cam, sizeof(Camera), hipMemcpyHostToDevice));
   
   //Allocate memory for all needed object pointers and their relative sizes
   HANDLE_ERROR(hipMalloc(&objArrD, objects.size() * sizeof(SceneObject*)));
   HANDLE_ERROR(hipMalloc(&sizesD, objects.size() * sizeof(int)));
   
   //Allocate memory on host for device pointers and object sizes
   printf("suh\n");
   objArrH = (SceneObject**)malloc(objects.size() * sizeof(SceneObject*));
   printf("dud\n");
   sizesH = (int*)malloc(objects.size() * sizeof(int));
   printf("bruh\n");
   
   //printf("OBJ SIZE: %d\n", objects.size());
   for (int i = 0; i < objects.size(); i++) {
      //printf("hello\n");
      if (objects[i]->type == 8) { //Oct tree handling
         //printf("yes\n");
         tempInd = 0;
         //Get the proper size of the tree
         tempSize = static_cast<OctTreeNode*>(objects[i])->treeLength();
         sizesH[i] = tempSize;
         
         //Copy over array represenation of tree
         
         tempOctTree = (Triangle*)malloc(tempSize * sizeof(Triangle));
         printf("frick\n");
         static_cast<OctTreeNode*>(objects[i])->toSerialArray(tempOctTree, &tempInd);
         printf("frack\n");
         HANDLE_ERROR(hipMalloc(&(objArrH[i]), tempSize * sizeof(Triangle)));
         printf("whats\n");
         HANDLE_ERROR(hipMemcpy(objArrH[i], tempOctTree, tempSize * sizeof(Triangle), hipMemcpyHostToDevice));
         printf("that\n");
         free(tempOctTree);
         printf("OH FUCK\n");
         
         //Rebuild tree on device
         gridDimension = int(ceil(sqrt((float(tempSize)) / (float(TILEWIDTH * TILEWIDTH)))));
         printf("GD: %d\n", gridDimension);
         dimGrid = dim3(gridDimension, gridDimension);//dim3((tempSize / TILEWIDTH) + 1, 1);
         dimBlock = dim3(TILEWIDTH,TILEWIDTH);//dim3(TILEWIDTH, 1);
         //printf("kk\n");
         toOctTree<<<dimGrid, dimBlock>>>((Triangle*)(objArrH[i]), tempSize, gridDimension);
         //printf("kkk\n");
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
   
      } else if (objects[i]->type == 5) { //Quad tree handling (Gerstner Wave Triangles)
         sizesH[i] = 1;
      } else { //Other object handling (mostly, if not all, planes)
         sizesH[i] = 1;
      }
   }
   
   //Copy device pointers from host array to device array and sizes of each obj pointer struct
   HANDLE_ERROR(hipMemcpy(objArrD, objArrH, objects.size() * sizeof(SceneObject*), hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMemcpy(sizesD, sizesH, objects.size() * sizeof(int), hipMemcpyHostToDevice));
   
   printf("hiya\n");
   raytracer = new RayTracer(objArrD, objects.size(), 0, 0, NULL, NULL);
   printf("fam\n");
   
   //Make blocks of dimension 32x32, with 32x32 threads on them
   dimGrid = dim3(width / TILEWIDTH, height / TILEWIDTH);
   dimBlock = dim3(TILEWIDTH, TILEWIDTH);
   
   printf("Calling Kernel...\n");
   //Run the Kernel code
   //hipDeviceSetLimit(hipLimitMallocHeapSize, size_t(3000000000));
   //GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, sizesD, objects.size(), pixelsD, cameraD, width, height, *raytracer);
   //GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, objects.size(), pixelsD, cameraD, width, height);
   HANDLE_ERROR(hipPeekAtLastError());
   HANDLE_ERROR(hipDeviceSynchronize());
   printf("SIZE GLM VEC: %d\n", sizeof(glm::vec3));
   printf("SIZE PIXEL: %d\n", sizeof(Pixel));
   //Copy modified pixel array from card back to host
   printf("Size parray: %d\n", pixelArrSize);
   HANDLE_ERROR(hipMemcpy(pixels, pixelsD, pixelArrSize, hipMemcpyDeviceToHost));
   
   //Free the data on the card
   printf("Freeing data on the card...\n");
   //Camera
   HANDLE_ERROR(hipFree(cameraD));
   printf("it\n");
   //Objects
   for (int i = 0; i < objects.size(); i++) {
      HANDLE_ERROR(hipFree(objArrH[i]));
   }
   printf("is\n");
   HANDLE_ERROR(hipFree(objArrD));
   HANDLE_ERROR(hipFree(sizesD));
   //Pixels
   HANDLE_ERROR(hipFree(pixelsD));
   //free(pixelsD);
   //KDTrees
   printf("where\n");
   
   //Free other data
   delete raytracer;
   free(sizesH);
   free(objArrH);
}
