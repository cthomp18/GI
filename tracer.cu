#include "hip/hip_runtime.h"
#include "tracer.h"
// http://stackoverflow.com/questions/3016077/how-to-spot-undefined-behavior
//Chris Lupo's error handling fuction/macro

__shared__ float sharedFloats[TILEWIDTH*TILEWIDTH * 9];
__shared__ int sharedInts[TILEWIDTH*TILEWIDTH * 3];


static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void editFuncPtrs(SceneObject *object) {
   if (object->type == 0) {         //SPHERE
      object->checkCollision = &(checkSphereCollision);
      object->getNormal = &(getSphereNormal);
   } else if (object->type == 1) {  //PLANE
      object->checkCollision = &(checkPlaneCollision);
      object->getNormal = &(getPlaneNormal);
   } else if (object->type == 2) {  //TRIANGLE
      object->checkCollision = &(checkTriCollision);
      object->getNormal = &(getTriNormal);
   } else if (object->type == 3) {  //BOX
      object->checkCollision = &(checkBoxCollision);
      object->getNormal = &(getBoxNormal);
   } else if (object->type == 4) {  //CONE
      object->checkCollision = &(checkConeCollision);
      object->getNormal = &(getConeNormal);
   }
}

__global__ void toOctTree(Triangle *objectArray, int size, int gridDimension) {
   //printf("xd\n");
   Triangle* tempT;
   OctTreeNode* tempO;
   
   int threadInd = (blockIdx.y * (gridDimension * TILEWIDTH) * TILEWIDTH) + (blockIdx.x * TILEWIDTH) +
                   (threadIdx.y * (gridDimension * TILEWIDTH)) + threadIdx.x;//blockIdx.x;//*TILEWIDTH + threadIdx.x;
   int i;

   if (threadInd == 0) { printf(":() %d\n", size); 
      printf("%f %f %f\n", objectArray->boundingBox.minPt.x, objectArray->boundingBox.minPt.y, objectArray->boundingBox.minPt.z);
      printf("%f %f %f\n", objectArray->boundingBox.maxPt.x, objectArray->boundingBox.maxPt.y, objectArray->boundingBox.maxPt.z);
   }
   if (threadInd == size - 1) { printf("UH SUH DUDE %d\n", size); }

   if (threadInd < size && threadInd >= 0) {
      if (objectArray[threadInd].type == 8) {
         tempO = reinterpret_cast<OctTreeNode*>(objectArray + threadInd);
         
         tempO->checkCollision = &(checkOctTreeCollision);
         tempO->getNormal = &(getOctTreeNormal);
         
         for (i = 0; i < 8; i++) {
            if (tempO->indeces[i] != -1) {
               tempO->octants[i] = reinterpret_cast<SceneObject*>(objectArray + tempO->indeces[i]);
            } else {
               tempO->octants[i] = NULL;
            }
         }
      } else {
         tempT = objectArray + threadInd;
         tempT->checkCollision = &(checkTriCollision);
         tempT->getNormal = &(getTriNormal);
      }
   } 
   
   __syncthreads();
}

__global__ void toQuadTree(Triangle *objectArray, int size, int gridDimension) {
   Triangle* tempT;
   QuadTreeNode* tempO;
   
   int threadInd = (blockIdx.y * (gridDimension * TILEWIDTH) * TILEWIDTH) + (blockIdx.x * TILEWIDTH) +
                   (threadIdx.y * (gridDimension * TILEWIDTH)) + threadIdx.x;
   int i;
   
   if (threadInd < size && threadInd >= 0) {
      if (objectArray[threadInd].type == 7) {
         tempO = reinterpret_cast<QuadTreeNode*>(objectArray + threadInd);
         
         tempO->checkCollision = &(checkQuadTreeCollision);
         tempO->getNormal = &(getQuadTreeNormal);

         for (i = 0; i < 4; i++) {
            if (tempO->indeces[i] != -1) {
               tempO->quadrants[i] = reinterpret_cast<SceneObject*>(objectArray + tempO->indeces[i]);
            } else {
               tempO->quadrants[i] = NULL;
            }
         }
      } else {
         tempT = objectArray + threadInd;
         
         tempT->checkCollision = &(checkTriCollision);
         tempT->getNormal = &(getTriNormal);
      }
   }
   __syncthreads();
}

__global__ void toKDTree(Photon *kdArray, int size, int gridDimension) {
   int row = blockIdx.y*TILEWIDTH + threadIdx.y;
   int col = blockIdx.x*TILEWIDTH + threadIdx.x;
   
   KDTreeNode* tempKD;
   
   int threadInd = (blockIdx.y * (gridDimension * TILEWIDTH) * TILEWIDTH) + (blockIdx.x * TILEWIDTH) +
                   (threadIdx.y * (gridDimension * TILEWIDTH)) + threadIdx.x;//blockIdx.x;//*TILEWIDTH + threadIdx.x;
   int i;

   if (threadInd < size && threadInd >= 0) {
      tempKD = reinterpret_cast<KDTreeNode*>(kdArray + (threadInd * 2));
      //tempO->checkCollision = &(checkOctTreeCollision);
      //tempO->getNormal = &(getOctTreeNormal);

      tempKD->photon = kdArray + (threadInd * 2) + 1;
      
      if (tempKD->leftInd != -1) {
         tempKD->left = reinterpret_cast<KDTreeNode*>(kdArray + tempKD->leftInd);
      } else {
         tempKD->left = NULL;
      }
      
      if (tempKD->rightInd != -1) {
         tempKD->right = reinterpret_cast<KDTreeNode*>(kdArray + tempKD->rightInd);
      } else {
         tempKD->right = NULL;
      }
   }
   __syncthreads();
}

__global__ __launch_bounds__( MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP ) void GIPhotonMapKernel(SceneObject **objArr, int *objSizes, int objSize, Pixel *pixelsD, Camera *camera, int width, int height, RayTracer *raytracer) {//, KDTreeNode *globalPhotons, KDTreeNode *causticPhotons) {
//__global__ void GIPhotonMapKernel(SceneObject **objArr, int *objSizes, int objSize, Pixel *pixelsD, Camera *camera, int width, int height, RayTracer *raytracer) {//, KDTreeNode *globalPhotons, KDTreeNode *causticPhotons) {
   //__shared__ TYPE Mds[TILEWIDTH][TILEWIDTH];
   //__shared__ TYPE Nds[TILEWIDTH][TILEWIDTH];
   
   //extern __shared__ float sh[];
   //extern __shared__ float uh[];
   //float *sh = NULL;
   
   int row = blockIdx.y*TILEWIDTH + threadIdx.y;
   int col = blockIdx.x*TILEWIDTH + threadIdx.x;
   //int currentImgInd = row * width + col;
   int currentImgInd = col * height + row;
   
   int threadNum = (threadIdx.y * TILEWIDTH) + threadIdx.x;
   
   int threadSpotF = threadNum * 9;
   int threadSpotI = threadNum * 3;
   
   glm::vec3 cPos = camera->getPosition();
   Collision* collision;
   glm::vec3 ray, tempColor;
   
   ray = pixelsD[currentImgInd].pt;// - cPos;
   ray = glm::normalize(ray);

   sharedFloats[threadSpotF+6] = cPos.x;
   sharedFloats[threadSpotF+7] = cPos.y;
   sharedFloats[threadSpotF+8] = cPos.z;
   collision = raytracer->trace(ray, sharedInts + threadSpotI, sharedFloats + threadSpotF);

   if (collision->time > TOLERANCE) {
      pixelsD[currentImgInd].clr = raytracer->calcRadiance(cPos, cPos + ray * collision->time, collision->object, false, 1.0f, 1.33f, 0.95f, threadNum, 0, sharedInts + threadSpotI, sharedFloats + threadSpotF); //Cam must start in air
   }
   else {
      pixelsD[currentImgInd].clr = glm::vec3(1.0f, 1.0f, 1.0f);
   }
   delete(collision);

   __syncthreads();
}

// Set the card up to run cuda
void RayTraceOnDevice(int width, int height, Pixel *pixels, std::vector<SceneObject*> objects, Camera *cam, KDTreeNode *globalPhotons, KDTreeNode *causticPhotons, time_t *startTime) {
   SceneObject** objArrD = NULL;
   SceneObject** objArrH = NULL;// = &objects[0];
   Photon* globalsD = NULL;
   Photon* causticsD = NULL;
   int gSize = 0;
   int cSize = 0;
   int* sizesH = NULL;
   int* sizesD = NULL;
   int gridDimension = 0;
   int depth;
      
   Triangle* tempOctTree = NULL;
   Photon* tempKDTree = NULL;
   Camera* cameraD = NULL;
   Pixel* pixelsD = NULL;
   RayTracer* raytracerD = NULL;
   RayTracer* raytracer = NULL;
   int tempSize = 0, tempInd = 0, pixelArrSize = 0;
   
   dim3 dimGrid = dim3(0,0);
   dim3 dimBlock = dim3(0,0);
   
   KDTreeNode **cudaPhotonStack;
   
   pixelArrSize = width * height * sizeof(Pixel);
   printf("~~~~~~~~~~~~~~FUNC LOC: %p\n", &checkOctTreeCollision);
   printf("Transferring Data...\n");
   printf("Size parray: %d\n", pixelArrSize);

   HANDLE_ERROR(hipMalloc(&pixelsD, pixelArrSize));
   HANDLE_ERROR(hipMemcpy(pixelsD, pixels, pixelArrSize, hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&cameraD, sizeof(Camera)));
   HANDLE_ERROR(hipMemcpy(cameraD, cam, sizeof(Camera), hipMemcpyHostToDevice));
   
   //Allocate memory for all needed object pointers and their relative sizes
   HANDLE_ERROR(hipMalloc(&objArrD, objects.size() * sizeof(SceneObject*)));
   HANDLE_ERROR(hipMalloc(&sizesD, objects.size() * sizeof(int)));
   
   //Allocate memory on host for device pointers and object sizes
   printf("suh\n");
   objArrH = (SceneObject**)malloc(objects.size() * sizeof(SceneObject*));
   printf("dud\n");
   sizesH = (int*)malloc(objects.size() * sizeof(int));
   printf("bruh\n");
   
   //printf("OBJ SIZE: %d\n", objects.size());
   printf("Objects...\n");
   for (uint i = 0; i < objects.size(); i++) {
      //printf("hello\n");
      if (objects[i]->type == 8) { //Oct tree handling
         //printf("yes\n");
         printf("OCT TREE START GPU\n");
         tempInd = 0; //yah
         //Get the proper size of the tree
         tempSize = static_cast<OctTreeNode*>(objects[i])->treeLength();
         sizesH[i] = tempSize;
         
         //Copy over array represenation of tree
         printf("TREE SIZE: %d\n", tempSize);
         
         
         tempOctTree = (Triangle*)calloc(tempSize, sizeof(Triangle));
         
         
         printf("ARR SIZE: %lu\n", tempSize * sizeof(Triangle));
         printf("ARR START POS: %p\n", tempOctTree);
         printf("frick\n");
         printf("SO SIZE %lu\n", sizeof(SceneObject));
         printf("TRI SIZE %lu\n", sizeof(Triangle));
         printf("OCT SIZE %lu\n", sizeof(OctTreeNode));
         
         
         reinterpret_cast<OctTreeNode*>(objects[i])->toSerialArray(tempOctTree, &tempInd);
         printf("BBS\n");
         printf("%f %f %f\n", objects[i]->boundingBox.minPt.x, objects[i]->boundingBox.minPt.y, objects[i]->boundingBox.minPt.z);
         printf("%f %f %f\n", objects[i]->boundingBox.maxPt.x, objects[i]->boundingBox.maxPt.y, objects[i]->boundingBox.maxPt.z);
         printf("%f %f %f\n", tempOctTree->boundingBox.minPt.x, tempOctTree->boundingBox.minPt.y, tempOctTree->boundingBox.minPt.z);
         printf("%f %f %f\n", tempOctTree->boundingBox.maxPt.x, tempOctTree->boundingBox.maxPt.y, tempOctTree->boundingBox.maxPt.z);
         
         printf("frack\n");
         
         HANDLE_ERROR(hipMalloc(&(objArrH[i]), tempSize * sizeof(Triangle)));
         
         printf("whats\n");
         
         HANDLE_ERROR(hipMemcpy(objArrH[i], tempOctTree, tempSize * sizeof(Triangle), hipMemcpyHostToDevice));
         
         printf("that\n");
         
         free(tempOctTree);
         
         printf("OH FUCK\n");
         
         //Rebuild tree on device
         gridDimension = int(ceil(sqrt((float(tempSize)) / (float(TILEWIDTH * TILEWIDTH)))));
         printf("GD: %d\n", gridDimension);
         dimGrid = dim3(gridDimension, gridDimension);//dim3((tempSize / TILEWIDTH) + 1, 1);
         dimBlock = dim3(TILEWIDTH,TILEWIDTH);//dim3(TILEWIDTH, 1);
         //printf("kk\n");
         
         
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
         
         
         toOctTree<<<dimGrid, dimBlock>>>((Triangle*)(objArrH[i]), tempSize, gridDimension);
         
         
         //printf("kkk\n");
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
   
      } else if (objects[i]->type == 7) { //Quad tree handling (Gerstner Wave Triangles)
         //printf("yes\n");
         printf("QUAD TREE START GPU\n");
         tempInd = 0; //yah
         //Get the proper size of the tree
         tempSize = static_cast<QuadTreeNode*>(objects[i])->treeLength();
         sizesH[i] = tempSize;
         
         //Copy over array represenation of tree
         printf("TREE SIZE: %d\n", tempSize);
         
         
         tempOctTree = (Triangle*)calloc(tempSize, sizeof(Triangle));
         
         
         printf("ARR SIZE: %lu\n", tempSize * sizeof(Triangle));
         printf("ARR START POS: %p\n", tempOctTree);
         printf("frick\n");
         printf("SO SIZE %lu\n", sizeof(SceneObject));
         printf("TRI SIZE %lu\n", sizeof(Triangle));
         printf("Quad SIZE %lu\n", sizeof(QuadTreeNode));
         
         
         reinterpret_cast<QuadTreeNode*>(objects[i])->toSerialArray(tempOctTree, &tempInd);
         printf("BBS\n");
         printf("%f %f %f\n", objects[i]->boundingBox.minPt.x, objects[i]->boundingBox.minPt.y, objects[i]->boundingBox.minPt.z);
         printf("%f %f %f\n", objects[i]->boundingBox.maxPt.x, objects[i]->boundingBox.maxPt.y, objects[i]->boundingBox.maxPt.z);
         printf("%f %f %f\n", tempOctTree->boundingBox.minPt.x, tempOctTree->boundingBox.minPt.y, tempOctTree->boundingBox.minPt.z);
         printf("%f %f %f\n", tempOctTree->boundingBox.maxPt.x, tempOctTree->boundingBox.maxPt.y, tempOctTree->boundingBox.maxPt.z);
         
         printf("frack\n");
         
         HANDLE_ERROR(hipMalloc(&(objArrH[i]), tempSize * sizeof(Triangle)));
         
         printf("whats\n");
         
         HANDLE_ERROR(hipMemcpy(objArrH[i], tempOctTree, tempSize * sizeof(Triangle), hipMemcpyHostToDevice));
         
         printf("that\n");
         
         free(tempOctTree);
         
         printf("OH FUCK\n");
         
         //Rebuild tree on device
         gridDimension = int(ceil(sqrt((float(tempSize)) / (float(TILEWIDTH * TILEWIDTH)))));
         printf("GD: %d\n", gridDimension);
         dimGrid = dim3(gridDimension, gridDimension);//dim3((tempSize / TILEWIDTH) + 1, 1);
         dimBlock = dim3(TILEWIDTH,TILEWIDTH);//dim3(TILEWIDTH, 1);
         //printf("kk\n");
         
         
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
         
         
         toQuadTree<<<dimGrid, dimBlock>>>((Triangle*)(objArrH[i]), tempSize, gridDimension);
         
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
      } else { //Other object handling (mostly, if not all, planes)
         if (objects[i]->type == 0) {         //SPHERE
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Sphere)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Sphere), hipMemcpyHostToDevice));
         } else if (objects[i]->type == 1) {  //PLANE
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Plane)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Plane), hipMemcpyHostToDevice));
         } else if (objects[i]->type == 2) {  //TRIANGLE
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Triangle)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Triangle), hipMemcpyHostToDevice));
         } else if (objects[i]->type == 3) {  //BOX
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Box)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Box), hipMemcpyHostToDevice));
         } else if (objects[i]->type == 4) {  //CONE
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Cone)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Cone), hipMemcpyHostToDevice));
         }
         editFuncPtrs<<<1,1>>>(objArrH[i]);
         sizesH[i] = 1;
      }
   }
   
   printf("Photon maps...\n");
   printf("SIZE KD: %lu\n", sizeof(KDTreeNode));
   printf("SIZE PHOTON: %lu\n", sizeof(Photon));
   
   if (globalPhotons) {
      printf("Porting global photons...\n");
      gSize = 0; //yah
      tempInd = 0;
      depth = 1;
      //Get the proper size of the tree
      gSize = globalPhotons->Treesize();
      //Copy over array represenation of tree
      printf("TREE SIZE: %d\n", gSize);
      
      
      tempKDTree = (Photon*)calloc(gSize * 2, sizeof(Photon));
      
      printf("GSIZE: %d\n", gSize);
      printf("ARR SIZE: %lu\n", gSize * sizeof(Photon));
      printf("ARR START POS: %p\n", tempKDTree);
      printf("frick\n");
      printf("P SIZE %lu\n", sizeof(Photon));
      printf("KD SIZE %lu\n", sizeof(KDTreeNode));
      
      globalPhotons->toSerialArray(tempKDTree, &tempInd);
      
      printf("TREE ARR END\n");
      
      printf("frack\n");
      
      HANDLE_ERROR(hipMalloc(&globalsD, gSize * 2 * sizeof(Photon)));
      
      printf("whats\n");
      
      HANDLE_ERROR(hipMemcpy(globalsD, tempKDTree, gSize * 2 * sizeof(Photon), hipMemcpyHostToDevice));
      
      printf("that\n");
      
      free(tempKDTree);
      
      printf("OH FUCK\n");
      
      //Rebuild tree on device
      gridDimension = int(ceil(sqrt((float(gSize)) / (float(TILEWIDTH * TILEWIDTH)))));
      printf("GD: %d\n", gridDimension);
      dimGrid = dim3(gridDimension, gridDimension);//dim3((tempSize / TILEWIDTH) + 1, 1);
      dimBlock = dim3(TILEWIDTH,TILEWIDTH);//dim3(TILEWIDTH, 1);
      
      
      HANDLE_ERROR(hipPeekAtLastError());
      HANDLE_ERROR(hipDeviceSynchronize());
      
      
      toKDTree<<<dimGrid, dimBlock>>>(globalsD, gSize, gridDimension);
      
      HANDLE_ERROR(hipPeekAtLastError());
      HANDLE_ERROR(hipDeviceSynchronize());
   }
   
   if (causticPhotons) {
      printf("Porting caustic photons...\n");
      cSize = 0;
   }
   
   //Copy device pointers from host array to device array and sizes of each obj pointer struct
   HANDLE_ERROR(hipMemcpy(objArrD, objArrH, objects.size() * sizeof(SceneObject*), hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMemcpy(sizesD, sizesH, objects.size() * sizeof(int), hipMemcpyHostToDevice));
   
   printf("hiya\n");
   printf("CSIZE: %d\n", cSize);
   
   //Make photon stack
   int treeLength = (gSize > cSize ? gSize : cSize);
   printf("GSIZE: %d\n", gSize);
   printf("CSIZE: %d\n", cSize);
   printf("TREELEN: %d\n", treeLength);
   while (treeLength != 0) {
      depth++;
      treeLength /= 2;
   }
   printf("DEPTH: %d\n", depth);
   
   //HANDLE_ERROR(hipMalloc(&cudaPhotonStack, depth * 2 * sizeof(KDTreeNode*) * width * height));
   HANDLE_ERROR(hipMalloc(&cudaPhotonStack, depth * 2 * sizeof(KDTreeNode*) * 640 * 640));
   //printf("ELEMENTS CPR LINDSEY STIRLING: %d\n", depth * 2 * width * height);
   KDTreeNode *node[10];
   int stuff[10];
   printf("Im running out of things to say: %p\n", cudaPhotonStack);
   printf("Im running out of things to say: %p\n", cudaPhotonStack + 1);
   printf("Im running out of things to sayN: %p\n", node);
   printf("Im running out of things to sayN: %p\n", node + 1);
   printf("Im running out of things to sayS: %p\n", stuff);
   printf("Im running out of things to sayS: %p\n", stuff + 1);
   printf("PTR SIZE %d\n", sizeof(KDTreeNode*));
   raytracer = new RayTracer(objArrD, objects.size(), gSize, cSize, reinterpret_cast<KDTreeNode*>(globalsD), reinterpret_cast<KDTreeNode*>(causticsD));
   raytracer->cudaStack = cudaPhotonStack;
   raytracer->stackPartition = depth * 2;
   printf("fam\n");
   
   HANDLE_ERROR(hipMalloc(&raytracerD, sizeof(RayTracer)));
   HANDLE_ERROR(hipMemcpy(raytracerD, raytracer, sizeof(RayTracer), hipMemcpyHostToDevice));
   
   //Make blocks of dimension 32x32, with 32x32 threads on them
   dimGrid = dim3(width / TILEWIDTH, height / TILEWIDTH);
   dimBlock = dim3(TILEWIDTH, TILEWIDTH);
   
   printf("Calling Kernel...\n");
   //Run the Kernel code
   //hipDeviceSetLimit(hipLimitMallocHeapSize, size_t(3000000000));
   
   size_t lim, freeM, totalM;
   cudaThreadGetLimit(&lim, hipLimitStackSize);
   printf("THREAD LIM: %d\n", lim);
   //cudaThreadSetLimit(hipLimitStackSize, 4096);
   //cudaThreadSetLimit(hipLimitStackSize, 6184);
   cudaThreadSetLimit(hipLimitStackSize, 8192);
   
   hipMemGetInfo(&freeM, &totalM); 
   printf("%lu KB free of total %lu KB\n",((freeM/1024)/2048)/16,((totalM/1024)/2048)/16);

   //cudaThreadSetLimit(hipLimitStackSize, 16384);
   cudaThreadGetLimit(&lim, hipLimitStackSize);
   printf("NEW THREAD LIM: %d\n", lim);
   
   //cudaThreadSetLimit(hipLimitStackSize, 16384);
   cudaThreadGetLimit(&lim, hipLimitMallocHeapSize);
   printf("THREAD HEAP LIM: %d\n", lim);
   cudaThreadSetLimit(hipLimitMallocHeapSize, lim*100);
   cudaThreadGetLimit(&lim, hipLimitMallocHeapSize);
   printf("NEW THREAD HEAP LIM: %d\n", lim);
   printf("TILE WIDTH: %d\n", TILEWIDTH);
   //hipLimitMallocHeapSize
   //BUG HAPPENS IN HERE
   time(startTime);
   //hipFuncSetCacheConfig(reinterpret_cast<const void*>(GIPhotonMapKernel), hipFuncCachePreferShared);
   //GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, sizesD, objects.size(), pixelsD, cameraD, width, height, raytracerD); //, globalsD, causticsD);

   GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, sizesD, objects.size(), pixelsD, cameraD, width, height, raytracerD); //, globalsD, causticsD);
   
   //GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, objects.size(), pixelsD, cameraD, width, height);
   HANDLE_ERROR(hipPeekAtLastError());
   HANDLE_ERROR(hipDeviceSynchronize());
   printf("SIZE GLM VEC: %lu\n", sizeof(glm::vec3));
   printf("SIZE PIXEL: %lu\n", sizeof(Pixel));
   //Copy modified pixel array from card back to host
   printf("Size parray: %d\n", pixelArrSize);
   HANDLE_ERROR(hipMemcpy(pixels, pixelsD, pixelArrSize, hipMemcpyDeviceToHost));
   
   //Free the data on the card
   printf("Freeing data on the card...\n");
   //Camera
   HANDLE_ERROR(hipFree(cameraD));
   //RT
   HANDLE_ERROR(hipFree(raytracerD));
   
   printf("it\n");
   //Objects
   for (uint i = 0; i < objects.size(); i++) {
   
      HANDLE_ERROR(hipFree(objArrH[i]));
   }
   printf("is\n");
   HANDLE_ERROR(hipFree(objArrD));
   HANDLE_ERROR(hipFree(sizesD));
   //Pixels
   HANDLE_ERROR(hipFree(pixelsD));
   //free(pixelsD);
   //KDTrees
   printf("where\n");
   if (globalsD) HANDLE_ERROR(hipFree(globalsD));
   if (causticsD) HANDLE_ERROR(hipFree(causticsD));
   HANDLE_ERROR(hipFree(cudaPhotonStack));
   
   //Free other data
   delete raytracer;
   free(sizesH);
   free(objArrH);
}
