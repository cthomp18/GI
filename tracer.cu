#include "hip/hip_runtime.h"
#include "tracer.h"
// http://stackoverflow.com/questions/3016077/how-to-spot-undefined-behavior
//Chris Lupo's error handling fuction/macro
static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void editFuncPtrs(SceneObject *object) {
   if (object->type == 0) {         //SPHERE
      object->checkCollision = &(checkSphereCollision);
      object->getNormal = &(getSphereNormal);
   } else if (object->type == 1) {  //PLANE
      object->checkCollision = &(checkPlaneCollision);
      object->getNormal = &(getPlaneNormal);
   } else if (object->type == 2) {  //TRIANGLE
      object->checkCollision = &(checkTriCollision);
      object->getNormal = &(getTriNormal);
   } else if (object->type == 3) {  //BOX
      object->checkCollision = &(checkBoxCollision);
      object->getNormal = &(getBoxNormal);
   } else if (object->type == 4) {  //CONE
      object->checkCollision = &(checkConeCollision);
      object->getNormal = &(getConeNormal);
   }
}

__global__ void toOctTree(Triangle *objectArray, int size, int gridDimension) {
   //printf("xd\n");
   Triangle* tempT;
   OctTreeNode* tempO;
   
   int threadInd = (blockIdx.y * (gridDimension * TILEWIDTH) * TILEWIDTH) + (blockIdx.x * TILEWIDTH) +
                   (threadIdx.y * (gridDimension * TILEWIDTH)) + threadIdx.x;//blockIdx.x;//*TILEWIDTH + threadIdx.x;
   int i;

   /*if (threadInd == 0) {
      printf("sup homie\n");
      tempO = new OctTreeNode((OctTreeNode*)(&(objectArray[0])));
      printf("suh\n");
      memcpy(objectArray, tempO, sizeof(OctTreeNode));
      printf(":)\n");
      delete(tempO);
   }*/
   //printf("%d\n", threadInd);
   //printf("xd\n");
   //if (threadInd == 0) {
      //printf("ARR LOC: %p\n", objectArray);}
      //printf(":(\n");
      //printf("SIZE ONODE: %d\n", sizeof(OctTreeNode));
      //printf("SIZE TOTS: %d\n", sizeof(OctTreeNode) * size);
   if (threadInd == 0) { printf(":() %d\n", size); 
      printf("%f %f %f\n", objectArray->boundingBox.minPt.x, objectArray->boundingBox.minPt.y, objectArray->boundingBox.minPt.z);
      printf("%f %f %f\n", objectArray->boundingBox.maxPt.x, objectArray->boundingBox.maxPt.y, objectArray->boundingBox.maxPt.z);
   }
   if (threadInd == size - 1) { printf("UH SUH DUDE %d\n", size); }
   //__syncthreads();
   if (threadInd < size && threadInd >= 0) {
      /*if (threadInd == size - 1) {
         printf("Thread Index: %d\n", threadInd);
         printf("Block? %d\n", blockIdx.x);
      }*/
      //printf("suh\n");
      if (objectArray[threadInd].type == 8) {
         tempO = reinterpret_cast<OctTreeNode*>(objectArray + threadInd);
         /*for (i = 0; i < 8; i++) {
            tempO->octants[i] = NULL;
         }*/
         tempO->checkCollision = &(checkOctTreeCollision);
         tempO->getNormal = &(getOctTreeNormal);
         //if (threadInd == 0) {
         //printf("B)\n");
         //tempO = (OctTreeNode*)(objectArray + threadInd);
         //printf("xd\n");
         //printf("fam\n");
         for (i = 0; i < 8; i++) {
            if (threadInd < 16) {
               //printf("Index: %d\n", tempO->indeces[i]);
            }
            if (tempO->indeces[i] != -1) {
               //printf("fuq\n");
               /*if (tempO->octants[i] == NULL) {
                  printf("Cool: %d\n", i);
               }*/
               
               tempO->octants[i] = reinterpret_cast<SceneObject*>(objectArray + tempO->indeces[i]);
               /*if (threadInd == 0) {printf("INDEX: %d\n", tempO->indeces[i]);
               printf("SO CAST: %p\n", reinterpret_cast<SceneObject*>(objectArray + tempO->indeces[i]));
               printf("REG: %p\n", objectArray + tempO->indeces[i]);}*/
               /*if (tempO->octants[i] == NULL) {
                  printf("Not Cool: %d\n", i);
               }*/
               
            } else {
               tempO->octants[i] = NULL;
            }
         }
         if (threadInd == 1) {
            //printf("Index: %d\n", tempO->indeces[i]);
            /*Index: 1
Index: 27182
Index: 54363
Index: 81544
Index: 108725
Index: 135906
Index: 163087
Index: 190268
*/
/*Index: 2
Index: 3399
Index: 6797
Index: 10194
Index: 13592
Index: 16989
Index: 20387
Index: 23784
*/
            //s.y = 1.0;
            //printf("%f %f %f\n", s.x, s.y, s.z);
//printf("%f %f %f\n", t.x, t.y, t.z);
            //tempO->checkCollision(s, t, 0.0f, (SceneObject**)&tempT);
         }
         //printf("diddly\n");
         //memcpy(&(objectArray[threadInd]), tempO, sizeof(OctTreeNode));
         //printf("ding\n");
         //for (i = 0; i < 8; i++) tempO->octants[i] = NULL;
         //delete tempO;
         //printf("dong\n");
         //}
      } else {
         //printf("Type: %d\n", objectArray[threadInd].type);
         /*printf("%d\n", threadInd);
         printf("%d\n", objectArray);
         printf("%d\n", sizeof(Triangle));
         printf("%d\n", sizeof(OctTreeNode));
         printf("%d\n", &(objectArray[threadInd]));*/
         //tempT = reinterpret_cast<Triangle*>(objectArray + threadInd);
         tempT = objectArray + threadInd;
         tempT->checkCollision = &(checkTriCollision);
         tempT->getNormal = &(getTriNormal);
         //memcpy(&(objectArray[threadInd]), tempO, sizeof(OctTreeNode));
         //memcpy(&(objectArray[threadInd]), tempT, sizeof(Triangle));
         //delete tempT;
      }
      //printf("damn\n");
   } else {
      //printf("WHAT\n");
      //printf("%d\n", threadInd);
      //printf("%d\n", size);
      //printf("COOL\n");
   }
   //printf("DAWG\n");
   //__syncthreads();
   //printf("%d\n", threadInd);
   //printf("WHAT\n", threadInd);
   if (threadInd == 1) {
      //printf("THE Fasdfadsf\n");
      //printf("OCTANT: %d\n", (&(objectArray[1]))->type);
      //printf("OCTANT: %d\n", ((OctTreeNode*)(&(objectArray[0])))->octants[0]->type);
      //tempO = (OctTreeNode*)(&(objectArray[0]));
      
      for (i = 0; i < 8; i++) {
         /*if (tempO->octants[i]) {
            printf("good %d\n", tempO->indeces[i]);
         } else {
            printf("bad %d\n", tempO->indeces[i]);
         }*/
      }
         //SceneObject **so;
         //tempO->checkCollision(s, t, 0.0f, so);
   }
   //}
   //memcpy(&(objectArray[threadInd]), tempO, sizeof(OctTreeNode));
   //printf("THE FUCK\n");
   //__syncthreads();
   __syncthreads();
   //printf("FUCK\n");
}

__global__ void toQuadTree(Triangle *objectArray, int size, int gridDimension) {
   Triangle* tempT;
   QuadTreeNode* tempO;
   
   int threadInd = (blockIdx.y * (gridDimension * TILEWIDTH) * TILEWIDTH) + (blockIdx.x * TILEWIDTH) +
                   (threadIdx.y * (gridDimension * TILEWIDTH)) + threadIdx.x;//blockIdx.x;//*TILEWIDTH + threadIdx.x;
   int i;

   /*if (threadInd == 0) { printf(":() %d\n", size); 
      printf("%f %f %f\n", objectArray->boundingBox.minPt.x, objectArray->boundingBox.minPt.y, objectArray->boundingBox.minPt.z);
      printf("%f %f %f\n", objectArray->boundingBox.maxPt.x, objectArray->boundingBox.maxPt.y, objectArray->boundingBox.maxPt.z);
   }*/
   //if (threadInd == size - 1) { printf("UH SUH DUDE %d\n", size); }
   
   if (threadInd < size && threadInd >= 0) {
      if (objectArray[threadInd].type == 7) {
         tempO = reinterpret_cast<QuadTreeNode*>(objectArray + threadInd);
         
         tempO->checkCollision = &(checkQuadTreeCollision);
         tempO->getNormal = &(getQuadTreeNormal);

         if (tempO->indeces[0] != -1) {
            tempO->q1 = reinterpret_cast<SceneObject*>(objectArray + tempO->indeces[0]);
         } else {
            tempO->q1 = NULL;
         }
         if (tempO->indeces[1] != -1) {
            tempO->q2 = reinterpret_cast<SceneObject*>(objectArray + tempO->indeces[1]);
         } else {
            tempO->q2 = NULL;
         }
         if (tempO->indeces[2] != -1) {
            tempO->q3 = reinterpret_cast<SceneObject*>(objectArray + tempO->indeces[2]);
         } else {
            tempO->q3 = NULL;
         }
         if (tempO->indeces[3] != -1) {
            tempO->q4 = reinterpret_cast<SceneObject*>(objectArray + tempO->indeces[3]);
         } else {
            tempO->q4 = NULL;
         }
      } else {
         tempT = objectArray + threadInd;
         
         tempT->checkCollision = &(checkTriCollision);
         tempT->getNormal = &(getTriNormal);
      }
   }
   __syncthreads();
}

__global__ void toKDTree(Photon *kdArray, int size, int gridDimension) {
   int row = blockIdx.y*TILEWIDTH + threadIdx.y;
   int col = blockIdx.x*TILEWIDTH + threadIdx.x;
   
   KDTreeNode* tempKD;
   
   int threadInd = (blockIdx.y * (gridDimension * TILEWIDTH) * TILEWIDTH) + (blockIdx.x * TILEWIDTH) +
                   (threadIdx.y * (gridDimension * TILEWIDTH)) + threadIdx.x;//blockIdx.x;//*TILEWIDTH + threadIdx.x;
   int i;

   if (threadInd == 0) { printf(":() %d\n", size); 
      //printf("%f %f %f\n", objectArray->boundingBox.minPt.x, objectArray->boundingBox.minPt.y, objectArray->boundingBox.minPt.z);
      //printf("%f %f %f\n", objectArray->boundingBox.maxPt.x, objectArray->boundingBox.maxPt.y, objectArray->boundingBox.maxPt.z);
   }
   if (threadInd == size - 1) { printf("UH SUH DUDE %d\n", size); }
   
   if (threadInd < size && threadInd >= 0) {
      tempKD = reinterpret_cast<KDTreeNode*>(kdArray + (threadInd * 2));
      //tempO->checkCollision = &(checkOctTreeCollision);
      //tempO->getNormal = &(getOctTreeNormal);

      tempKD->photon = kdArray + (threadInd * 2) + 1;
      
      if (tempKD->leftInd != -1) {
         tempKD->left = reinterpret_cast<KDTreeNode*>(kdArray + tempKD->leftInd);
      } else {
         tempKD->left = NULL;
      }
      
      if (tempKD->rightInd != -1) {
         tempKD->right = reinterpret_cast<KDTreeNode*>(kdArray + tempKD->rightInd);
      } else {
         tempKD->right = NULL;
      }
   } else {
      /*printf("WHAT\n");
      printf("%d\n", threadInd);
      printf("%d\n", size);*/
   }

   __syncthreads();
}

__global__ void GIPhotonMapKernel(SceneObject **objArr, int *objSizes, int objSize, Pixel *pixelsD, Camera *camera, int width, int height, RayTracer *raytracer) {//, KDTreeNode *globalPhotons, KDTreeNode *causticPhotons) {
   //__shared__ TYPE Mds[TILEWIDTH][TILEWIDTH];
   //__shared__ TYPE Nds[TILEWIDTH][TILEWIDTH];
   int row = blockIdx.y*TILEWIDTH + threadIdx.y;
   int col = blockIdx.x*TILEWIDTH + threadIdx.x;
   //if (row == 0 && col == 0) printf("cool\n");
   //if (row == 639 && col == 639) printf("cooler\n");
   //if (row > 616) {// && col < 240) {
   //int c = 0;
   //printf("yo\n");
   /*printf("REG MIN %f %f %f\n", objArr[0]->boundingBox.minPt.x, objArr[0]->boundingBox.minPt.y, objArr[0]->boundingBox.minPt.z);
   printf("REG MAX %f %f %f\n", objArr[0]->boundingBox.maxPt.x, objArr[0]->boundingBox.maxPt.y, objArr[0]->boundingBox.maxPt.z);
   printf("RAY MIN %f %f %f\n", raytracer->objects[0]->boundingBox.minPt.x, raytracer->objects[0]->boundingBox.minPt.y, raytracer->objects[0]->boundingBox.minPt.z);
   printf("RAY MAX %f %f %f\n", raytracer->objects[0]->boundingBox.maxPt.x, raytracer->objects[0]->boundingBox.maxPt.y, raytracer->objects[0]->boundingBox.maxPt.z);*/
   glm::vec3 cPos = camera->getPosition();
   //glm::vec3 cPos = glm::vec3(0.0, 0.0, 0.0);
   //RayTracer* raytrace = new RayTracer(objArr, 0, 0, 0, 0, 0);
   //delete(raytrace);
   int currentImgInd;
   
   //Collision* col = new Collision();
   //delete(col);
   //for (int i = 0; i < width / TILEWIDTH; i++) {
   //i * imgheight + j;
      //int threadIndex = row*width1 + (m*TILEWIDTH + threadIdx.x);
      //Nds[threadIdx.y][threadIdx.x] = Nd[col + (m*TILEWIDTH + threadIdx.y)*width2];
      //cout << "i: " << i << endl;
   Collision* collision;
   glm::vec3 ray, tempColor;
      //for (int j = 0; j < height; j++) {
   currentImgInd = col * height + row;
   ray = pixelsD[currentImgInd].pt;// - cPos;
   ray = glm::normalize(ray);

   /*if (currentImgInd == 0) {
      //printf("First Obj Type: %d\n", objArr[0]->type);
      OctTreeNode* o = (OctTreeNode*)objArr[0];
      
      printf("Huh %d\n", o->type);
      printf("ImgInd: %d\n", currentImgInd);
      
      for(int i = 0; i < 8; i++) {
         if (o->octants[i] != NULL) {
            printf("Oct Type: %d\n", o->octants[i]->type);
         } else {
            printf("Null Oct: %d\n", i);
         }
      }
      printf("Tree size: %d\n", o->treeLength());
   }*/
if (col < 100) {// && row == 0) {
   //printf("Im running out of things to say: %p\n", raytracer->cudaStack);
   //printf("Im running out of things to say: %p\n", raytracer->cudaStack + 1);
   //printf("ROW %d\n", row);
//if (currentImgInd < 0) {
   //printf("hello?\n");
    
   collision = raytracer->trace(cPos, ray, false);
    
   //printf("whats up?\n");
         //printf("Making sure ;)\n");
         //printf("HI\n");
   if (collision) {
      //printf("UH SUH\n");
   } else {
      printf(" WUT \n");
   }
   if (collision->time > TOLERANCE) {
      //printf("in here?\n");
      pixelsD[currentImgInd].clr = raytracer->calcRadiance(cPos, cPos + ray * collision->time, collision->object, false, 1.0f, 1.33f, 0.95f, currentImgInd, 0);//(threadIdx.y * TILEWIDTH) + threadIdx.x, 0); //Cam must start in air
      if (collision->object) {
         //printf("yo wasuu\n");
         //printf("%d\n", collision->object->type);
      } else {
         //printf("k\n");
      }
      //tempColor = collision->object->getNormal(collision->object, cPos + ray * collision->time, 2.0f);
      //printf(" WUT \n");
      //pixelsD[currentImgInd].clr = (tempColor * 0.5f) + 0.5f;
      //printf("Anything here?\n");
      //pixelsD[currentImgInd].clr = glm::vec3(1.0f, 1.0f, 1.0f);
   }
   else {
      //printf("should not be in here\n");
      pixelsD[currentImgInd].clr = glm::vec3(1.0f, 1.0f, 1.0f);
      //printf("Making sure\n");
   }
   //printf("WHAT\n");
   //printf("ROW END %d\n", row);
   delete(collision);
}
   //}
         //cout << "PIXCOL: " << pixels[i][j].clr.r << " " << pixels[i][j].clr.g << " " << pixels[i][j].clr.b << endl;
      //}
   //}
   __syncthreads();
   
   
   /*for (int i = 0; i < width * height; i++) {
      pixelsD[i].clr.x = pixelsD[i].clr.y = pixelsD[i].clr.z = 1.0;
   }*/
}

// Set the card up to run cuda
void RayTraceOnDevice(int width, int height, Pixel *pixels, std::vector<SceneObject*> objects, Camera *cam, KDTreeNode *globalPhotons, KDTreeNode *causticPhotons) {
   SceneObject** objArrD = NULL;
   SceneObject** objArrH = NULL;// = &objects[0];
   Photon* globalsD = NULL;
   Photon* causticsD = NULL;
   int gSize = 0;
   int cSize = 0;
   int* sizesH = NULL;
   int* sizesD = NULL;
   int gridDimension = 0;
   int depth;
      
   Triangle* tempOctTree = NULL;
   Photon* tempKDTree = NULL;
   Camera* cameraD = NULL;
   Pixel* pixelsD = NULL;
   RayTracer* raytracerD = NULL;
   RayTracer* raytracer = NULL;
   int tempSize = 0, tempInd = 0, pixelArrSize = 0;
   
   dim3 dimGrid = dim3(0,0);
   dim3 dimBlock = dim3(0,0);
   
   KDTreeNode **cudaPhotonStack;
   
   pixelArrSize = width * height * sizeof(Pixel);
   printf("~~~~~~~~~~~~~~FUNC LOC: %p\n", &checkOctTreeCollision);
   printf("Transferring Data...\n");
   printf("Size parray: %d\n", pixelArrSize);

   HANDLE_ERROR(hipMalloc(&pixelsD, pixelArrSize));
   HANDLE_ERROR(hipMemcpy(pixelsD, pixels, pixelArrSize, hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&cameraD, sizeof(Camera)));
   HANDLE_ERROR(hipMemcpy(cameraD, cam, sizeof(Camera), hipMemcpyHostToDevice));
   
   //Allocate memory for all needed object pointers and their relative sizes
   HANDLE_ERROR(hipMalloc(&objArrD, objects.size() * sizeof(SceneObject*)));
   HANDLE_ERROR(hipMalloc(&sizesD, objects.size() * sizeof(int)));
   
   //Allocate memory on host for device pointers and object sizes
   printf("suh\n");
   objArrH = (SceneObject**)malloc(objects.size() * sizeof(SceneObject*));
   printf("dud\n");
   sizesH = (int*)malloc(objects.size() * sizeof(int));
   printf("bruh\n");
   
   //printf("OBJ SIZE: %d\n", objects.size());
   printf("Objects...\n");
   for (uint i = 0; i < objects.size(); i++) {
      //printf("hello\n");
      if (objects[i]->type == 8) { //Oct tree handling
         //printf("yes\n");
         printf("OCT TREE START GPU\n");
         tempInd = 0; //yah
         //Get the proper size of the tree
         tempSize = static_cast<OctTreeNode*>(objects[i])->treeLength();
         sizesH[i] = tempSize;
         
         //Copy over array represenation of tree
         printf("TREE SIZE: %d\n", tempSize);
         
         
         tempOctTree = (Triangle*)calloc(tempSize, sizeof(Triangle));
         
         
         printf("ARR SIZE: %lu\n", tempSize * sizeof(Triangle));
         printf("ARR START POS: %p\n", tempOctTree);
         printf("frick\n");
         printf("SO SIZE %lu\n", sizeof(SceneObject));
         printf("TRI SIZE %lu\n", sizeof(Triangle));
         printf("OCT SIZE %lu\n", sizeof(OctTreeNode));
         
         
         reinterpret_cast<OctTreeNode*>(objects[i])->toSerialArray(tempOctTree, &tempInd);
         printf("BBS\n");
         printf("%f %f %f\n", objects[i]->boundingBox.minPt.x, objects[i]->boundingBox.minPt.y, objects[i]->boundingBox.minPt.z);
         printf("%f %f %f\n", objects[i]->boundingBox.maxPt.x, objects[i]->boundingBox.maxPt.y, objects[i]->boundingBox.maxPt.z);
         printf("%f %f %f\n", tempOctTree->boundingBox.minPt.x, tempOctTree->boundingBox.minPt.y, tempOctTree->boundingBox.minPt.z);
         printf("%f %f %f\n", tempOctTree->boundingBox.maxPt.x, tempOctTree->boundingBox.maxPt.y, tempOctTree->boundingBox.maxPt.z);
         
         printf("frack\n");
         
         HANDLE_ERROR(hipMalloc(&(objArrH[i]), tempSize * sizeof(Triangle)));
         
         printf("whats\n");
         
         HANDLE_ERROR(hipMemcpy(objArrH[i], tempOctTree, tempSize * sizeof(Triangle), hipMemcpyHostToDevice));
         
         printf("that\n");
         
         free(tempOctTree);
         
         printf("OH FUCK\n");
         
         //Rebuild tree on device
         gridDimension = int(ceil(sqrt((float(tempSize)) / (float(TILEWIDTH * TILEWIDTH)))));
         printf("GD: %d\n", gridDimension);
         dimGrid = dim3(gridDimension, gridDimension);//dim3((tempSize / TILEWIDTH) + 1, 1);
         dimBlock = dim3(TILEWIDTH,TILEWIDTH);//dim3(TILEWIDTH, 1);
         //printf("kk\n");
         
         
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
         
         
         toOctTree<<<dimGrid, dimBlock>>>((Triangle*)(objArrH[i]), tempSize, gridDimension);
         
         
         //printf("kkk\n");
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
   
      } else if (objects[i]->type == 7) { //Quad tree handling (Gerstner Wave Triangles)
         //printf("yes\n");
         printf("QUAD TREE START GPU\n");
         tempInd = 0; //yah
         //Get the proper size of the tree
         tempSize = static_cast<QuadTreeNode*>(objects[i])->treeLength();
         sizesH[i] = tempSize;
         
         //Copy over array represenation of tree
         printf("TREE SIZE: %d\n", tempSize);
         
         
         tempOctTree = (Triangle*)calloc(tempSize, sizeof(Triangle));
         
         
         printf("ARR SIZE: %lu\n", tempSize * sizeof(Triangle));
         printf("ARR START POS: %p\n", tempOctTree);
         printf("frick\n");
         printf("SO SIZE %lu\n", sizeof(SceneObject));
         printf("TRI SIZE %lu\n", sizeof(Triangle));
         printf("Quad SIZE %lu\n", sizeof(QuadTreeNode));
         
         
         reinterpret_cast<QuadTreeNode*>(objects[i])->toSerialArray(tempOctTree, &tempInd);
         printf("BBS\n");
         printf("%f %f %f\n", objects[i]->boundingBox.minPt.x, objects[i]->boundingBox.minPt.y, objects[i]->boundingBox.minPt.z);
         printf("%f %f %f\n", objects[i]->boundingBox.maxPt.x, objects[i]->boundingBox.maxPt.y, objects[i]->boundingBox.maxPt.z);
         printf("%f %f %f\n", tempOctTree->boundingBox.minPt.x, tempOctTree->boundingBox.minPt.y, tempOctTree->boundingBox.minPt.z);
         printf("%f %f %f\n", tempOctTree->boundingBox.maxPt.x, tempOctTree->boundingBox.maxPt.y, tempOctTree->boundingBox.maxPt.z);
         
         printf("frack\n");
         
         HANDLE_ERROR(hipMalloc(&(objArrH[i]), tempSize * sizeof(Triangle)));
         
         printf("whats\n");
         
         HANDLE_ERROR(hipMemcpy(objArrH[i], tempOctTree, tempSize * sizeof(Triangle), hipMemcpyHostToDevice));
         
         printf("that\n");
         
         free(tempOctTree);
         
         printf("OH FUCK\n");
         
         //Rebuild tree on device
         gridDimension = int(ceil(sqrt((float(tempSize)) / (float(TILEWIDTH * TILEWIDTH)))));
         printf("GD: %d\n", gridDimension);
         dimGrid = dim3(gridDimension, gridDimension);//dim3((tempSize / TILEWIDTH) + 1, 1);
         dimBlock = dim3(TILEWIDTH,TILEWIDTH);//dim3(TILEWIDTH, 1);
         //printf("kk\n");
         
         
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
         
         
         toQuadTree<<<dimGrid, dimBlock>>>((Triangle*)(objArrH[i]), tempSize, gridDimension);
         
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
      } else { //Other object handling (mostly, if not all, planes)
         if (objects[i]->type == 0) {         //SPHERE
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Sphere)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Sphere), hipMemcpyHostToDevice));
         } else if (objects[i]->type == 1) {  //PLANE
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Plane)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Plane), hipMemcpyHostToDevice));
         } else if (objects[i]->type == 2) {  //TRIANGLE
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Triangle)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Triangle), hipMemcpyHostToDevice));
         } else if (objects[i]->type == 3) {  //BOX
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Box)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Box), hipMemcpyHostToDevice));
         } else if (objects[i]->type == 4) {  //CONE
            HANDLE_ERROR(hipMalloc(&(objArrH[i]), sizeof(Cone)));
            HANDLE_ERROR(hipMemcpy(objArrH[i], objects[i], sizeof(Cone), hipMemcpyHostToDevice));
         }
         editFuncPtrs<<<1,1>>>(objArrH[i]);
         sizesH[i] = 1;
      }
   }
   
   printf("Photon maps...\n");
   printf("SIZE KD: %lu\n", sizeof(KDTreeNode));
   printf("SIZE PHOTON: %lu\n", sizeof(Photon));
   
   if (globalPhotons) {
      printf("Porting global photons...\n");
      gSize = 0; //yah
      tempInd = 0;
      depth = 1;
      //Get the proper size of the tree
      gSize = globalPhotons->Treesize();
      //Copy over array represenation of tree
      printf("TREE SIZE: %d\n", gSize);
      
      
      tempKDTree = (Photon*)calloc(gSize * 2, sizeof(Photon));
      
      printf("GSIZE: %d\n", gSize);
      printf("ARR SIZE: %lu\n", gSize * sizeof(Photon));
      printf("ARR START POS: %p\n", tempKDTree);
      printf("frick\n");
      printf("P SIZE %lu\n", sizeof(Photon));
      printf("KD SIZE %lu\n", sizeof(KDTreeNode));
      
      //if (threadNum == 0) {
         //printf("TREE START\n");
         //globalPhotons->printTree(globalPhotons);
         //printf("TREE END\n");
     // }   
      globalPhotons->toSerialArray(tempKDTree, &tempInd);
      
      /*printf("TREE ARR START\n");
      for (int i = 0; i < gSize * 2; i++) {
         if (i % 2 == 0) {
            printf("AXIS: %d\n", reinterpret_cast<KDTreeNode*>(tempKDTree + i)->axis);
            printf("LEFT IND: %d, RIGHT IND: %d\n", reinterpret_cast<KDTreeNode*>(tempKDTree + i)->leftInd, reinterpret_cast<KDTreeNode*>(tempKDTree + i)->rightInd);
         } else {
            printf("Pt: %f %f %f\n", tempKDTree[i].pt.x, tempKDTree[i].pt.y, tempKDTree[i].pt.z);
         }
      }*/
      //globalPhotons->printTree(globalPhotons);
      printf("TREE ARR END\n");
      
      printf("frack\n");
      
      HANDLE_ERROR(hipMalloc(&globalsD, gSize * 2 * sizeof(Photon)));
      
      printf("whats\n");
      
      HANDLE_ERROR(hipMemcpy(globalsD, tempKDTree, gSize * 2 * sizeof(Photon), hipMemcpyHostToDevice));
      
      printf("that\n");
      
      free(tempKDTree);
      
      printf("OH FUCK\n");
      
      //Rebuild tree on device
      gridDimension = int(ceil(sqrt((float(gSize)) / (float(TILEWIDTH * TILEWIDTH)))));
      printf("GD: %d\n", gridDimension);
      dimGrid = dim3(gridDimension, gridDimension);//dim3((tempSize / TILEWIDTH) + 1, 1);
      dimBlock = dim3(TILEWIDTH,TILEWIDTH);//dim3(TILEWIDTH, 1);
      //printf("kk\n");
      
      
      HANDLE_ERROR(hipPeekAtLastError());
      HANDLE_ERROR(hipDeviceSynchronize());
      
      
      toKDTree<<<dimGrid, dimBlock>>>(globalsD, gSize, gridDimension);
      
      //printf("TREE START\n");
      //globalPhotons->printTree(globalPhotons);
      //printf("TREE END\n");
         
      HANDLE_ERROR(hipPeekAtLastError());
      HANDLE_ERROR(hipDeviceSynchronize());
   }
   
   if (causticPhotons) {
      printf("Porting caustic photons...\n");
      cSize = 0;
   }
   
   //Copy device pointers from host array to device array and sizes of each obj pointer struct
   HANDLE_ERROR(hipMemcpy(objArrD, objArrH, objects.size() * sizeof(SceneObject*), hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMemcpy(sizesD, sizesH, objects.size() * sizeof(int), hipMemcpyHostToDevice));
   
   printf("hiya\n");
   printf("CSIZE: %d\n", cSize);
   
   //Make photon stack
   int treeLength = (gSize > cSize ? gSize : cSize);
   printf("GSIZE: %d\n", gSize);
   printf("CSIZE: %d\n", cSize);
   printf("TREELEN: %d\n", treeLength);
   while (treeLength != 0) {
      depth++;
      treeLength /= 2;
   }
   printf("DEPTH: %d\n", depth);
   
   HANDLE_ERROR(hipMalloc(&cudaPhotonStack, depth * 2 * sizeof(KDTreeNode*) * 640 * 640));
   
   KDTreeNode *node[10];
   int stuff[10];
   printf("Im running out of things to say: %p\n", cudaPhotonStack);
   printf("Im running out of things to say: %p\n", cudaPhotonStack + 1);
   printf("Im running out of things to sayN: %p\n", node);
   printf("Im running out of things to sayN: %p\n", node + 1);
   printf("Im running out of things to sayS: %p\n", stuff);
   printf("Im running out of things to sayS: %p\n", stuff + 1);
   printf("PTR SIZE %d\n", sizeof(KDTreeNode*));
   raytracer = new RayTracer(objArrD, objects.size(), gSize, cSize, reinterpret_cast<KDTreeNode*>(globalsD), reinterpret_cast<KDTreeNode*>(causticsD));
   raytracer->cudaStack = cudaPhotonStack;
   raytracer->stackPartition = depth * 2;
   printf("fam\n");
   
   HANDLE_ERROR(hipMalloc(&raytracerD, sizeof(RayTracer)));
   HANDLE_ERROR(hipMemcpy(raytracerD, raytracer, sizeof(RayTracer), hipMemcpyHostToDevice));
   
   //Make blocks of dimension 32x32, with 32x32 threads on them
   dimGrid = dim3(width / TILEWIDTH, height / TILEWIDTH);
   dimBlock = dim3(TILEWIDTH, TILEWIDTH);
   
   printf("Calling Kernel...\n");
   //Run the Kernel code
   //hipDeviceSetLimit(hipLimitMallocHeapSize, size_t(3000000000));
   
   size_t lim;
   cudaThreadGetLimit(&lim, hipLimitStackSize);
   printf("THREAD LIM: %d\n", lim);
   //cudaThreadSetLimit(hipLimitStackSize, 4096);
   cudaThreadSetLimit(hipLimitStackSize, 8192);
   //cudaThreadSetLimit(hipLimitStackSize, 16384);
   cudaThreadGetLimit(&lim, hipLimitStackSize);
   printf("NEW THREAD LIM: %d\n", lim);
   
   //BUG HAPPENS IN HERE
   GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, sizesD, objects.size(), pixelsD, cameraD, width, height, raytracerD); //, globalsD, causticsD);
   
   
   //GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, objects.size(), pixelsD, cameraD, width, height);
   HANDLE_ERROR(hipPeekAtLastError());
   HANDLE_ERROR(hipDeviceSynchronize());
   printf("SIZE GLM VEC: %lu\n", sizeof(glm::vec3));
   printf("SIZE PIXEL: %lu\n", sizeof(Pixel));
   //Copy modified pixel array from card back to host
   printf("Size parray: %d\n", pixelArrSize);
   HANDLE_ERROR(hipMemcpy(pixels, pixelsD, pixelArrSize, hipMemcpyDeviceToHost));
   
   //Free the data on the card
   printf("Freeing data on the card...\n");
   //Camera
   HANDLE_ERROR(hipFree(cameraD));
   //RT
   HANDLE_ERROR(hipFree(raytracerD));
   
   printf("it\n");
   //Objects
   for (uint i = 0; i < objects.size(); i++) {
      HANDLE_ERROR(hipFree(objArrH[i]));
   }
   printf("is\n");
   HANDLE_ERROR(hipFree(objArrD));
   HANDLE_ERROR(hipFree(sizesD));
   //Pixels
   HANDLE_ERROR(hipFree(pixelsD));
   //free(pixelsD);
   //KDTrees
   printf("where\n");
   if (globalsD) HANDLE_ERROR(hipFree(globalsD));
   if (causticsD) HANDLE_ERROR(hipFree(causticsD));
   HANDLE_ERROR(hipFree(cudaPhotonStack));
   
   //Free other data
   delete raytracer;
   free(sizesH);
   free(objArrH);
}
