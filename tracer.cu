#include "hip/hip_runtime.h"
#include "tracer.h"
// http://stackoverflow.com/questions/3016077/how-to-spot-undefined-behavior
//Chris Lupo's error handling fuction/macro
static void HandleError( hipError_t err,
    const char *file,
    int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void toOctTree(Triangle *objectArray, int size, int gridDimension) {
   //printf("xd\n");
   Triangle* tempT;
   OctTreeNode* tempO;
   
   int threadInd = (blockIdx.y * (gridDimension * TILEWIDTH) * TILEWIDTH) + (blockIdx.x * TILEWIDTH) +
                   (threadIdx.y * (gridDimension * TILEWIDTH)) + threadIdx.x;//blockIdx.x;//*TILEWIDTH + threadIdx.x;
   int i;

   /*if (threadInd == 0) {
      printf("sup homie\n");
      tempO = new OctTreeNode((OctTreeNode*)(&(objectArray[0])));
      printf("suh\n");
      memcpy(objectArray, tempO, sizeof(OctTreeNode));
      printf(":)\n");
      delete(tempO);
   }*/
   //printf("%d\n", threadInd);
   //printf("xd\n");
   if (threadInd < 7) {
      //printf(":(\n");
      //printf("SIZE ONODE: %d\n", sizeof(OctTreeNode));
      //printf("SIZE TOTS: %d\n", sizeof(OctTreeNode) * size);
   if (threadInd == 0) { printf(":() %d\n", size); }
   if (threadInd == size - 1) { printf("UH SUH DUDE %d\n", size); }
   //__syncthreads();
   if (threadInd < size && threadInd >= 0) {
      /*if (threadInd == size - 1) {
         printf("Thread Index: %d\n", threadInd);
         printf("Block? %d\n", blockIdx.x);
      }*/
      //printf("suh\n");
      if (objectArray[threadInd].type == 8) {
         tempO = new OctTreeNode((OctTreeNode*)(&(objectArray[threadInd])));
         /*for (i = 0; i < 8; i++) {
            tempO->octants[i] = NULL;
         }*/
         
         //if (threadInd == 0) {
         //printf("B)\n");
         //tempO = (OctTreeNode*)(objectArray + threadInd);
         //printf("xd\n");
         //printf("fam\n");
         for (i = 0; i < 8; i++) {
            /*if (threadInd < 16) {
               printf("Index: %d\n", tempO->indeces[i]);
            }*/
            if (tempO->indeces[i] != -1) {
               //printf("fuq\n");
               /*if (tempO->octants[i] == NULL) {
                  printf("Cool: %d\n", i);
               }*/
               
               tempO->octants[i] = (SceneObject*)(&(objectArray[tempO->indeces[i]]));
               /*if (tempO->octants[i] == NULL) {
                  printf("Not Cool: %d\n", i);
               }*/
               
            } else {
               tempO->octants[i] = NULL;
            }
         }
         if (threadInd == 1) {
            //printf("Index: %d\n", tempO->indeces[i]);
            /*Index: 1
Index: 27182
Index: 54363
Index: 81544
Index: 108725
Index: 135906
Index: 163087
Index: 190268
*/
/*Index: 2
Index: 3399
Index: 6797
Index: 10194
Index: 13592
Index: 16989
Index: 20387
Index: 23784
*/
            //s.y = 1.0;
            //printf("%f %f %f\n", s.x, s.y, s.z);
//printf("%f %f %f\n", t.x, t.y, t.z);
            //tempO->checkCollision(s, t, 0.0f, (SceneObject**)&tempT);
         }
         //printf("diddly\n");
         memcpy(&(objectArray[threadInd]), tempO, sizeof(OctTreeNode));
         //printf("ding\n");
         for (i = 0; i < 8; i++) tempO->octants[i] = NULL;
         delete tempO;
         //printf("dong\n");
         //}
      } else {
         //printf("Type: %d\n", objectArray[threadInd].type);
         printf("%d\n", threadInd);
         printf("%d\n", objectArray);
         printf("%d\n", sizeof(Triangle));
         printf("%d\n", sizeof(OctTreeNode));
         printf("%d\n", &(objectArray[threadInd]));
         tempT = new Triangle((Triangle*)(objectArray + threadInd));
         //tempT = objectArray + threadInd;
         //memcpy(&(objectArray[threadInd]), tempO, sizeof(OctTreeNode));
         //memcpy(&(objectArray[threadInd]), tempT, sizeof(Triangle));
         delete tempT;
      }
      //printf("damn\n");
   } else {
      //printf("WHAT\n");
      //printf("%d\n", threadInd);
      //printf("%d\n", size);
      //printf("COOL\n");
   }
   //printf("DAWG\n");
   //__syncthreads();
   //printf("%d\n", threadInd);
   //printf("WHAT\n", threadInd);
   if (threadInd == 1) {
      //printf("THE Fasdfadsf\n");
      //printf("OCTANT: %d\n", (&(objectArray[1]))->type);
      //printf("OCTANT: %d\n", ((OctTreeNode*)(&(objectArray[0])))->octants[0]->type);
      tempO = (OctTreeNode*)(&(objectArray[0]));
      
      for (i = 0; i < 8; i++) {
         /*if (tempO->octants[i]) {
            printf("good %d\n", tempO->indeces[i]);
         } else {
            printf("bad %d\n", tempO->indeces[i]);
         }*/
      }
         //SceneObject **so;
         //tempO->checkCollision(s, t, 0.0f, so);
   }
   }
   //memcpy(&(objectArray[threadInd]), tempO, sizeof(OctTreeNode));
   //printf("THE FUCK\n");
   //__syncthreads();
   __syncthreads();
   //printf("FUCK\n");
}

__global__ void GIPhotonMapKernel(SceneObject **objArr, int *objSizes, int objSize, Pixel *pixelsD, Camera *camera, int width, int height, RayTracer raytracer) {
   //__shared__ TYPE Mds[TILEWIDTH][TILEWIDTH];
   //__shared__ TYPE Nds[TILEWIDTH][TILEWIDTH];
   int row = blockIdx.y*TILEWIDTH + threadIdx.y;
   int col = blockIdx.x*TILEWIDTH + threadIdx.x;
   if (row == 320 && col == 320) {
   //int c = 0;
   //printf("yo\n");
   glm::vec3 cPos = camera->getPosition();
   //glm::vec3 cPos = glm::vec3(0.0, 0.0, 0.0);
   //RayTracer* raytrace = new RayTracer(objArr, 0, 0, 0, 0, 0);
   //delete(raytrace);
   int currentImgInd;
   
   //Collision* col = new Collision();
   //delete(col);
   //for (int i = 0; i < width / TILEWIDTH; i++) {
   //i * imgheight + j;
      //int threadIndex = row*width1 + (m*TILEWIDTH + threadIdx.x);
      //Nds[threadIdx.y][threadIdx.x] = Nd[col + (m*TILEWIDTH + threadIdx.y)*width2];
      //cout << "i: " << i << endl;
   Collision* collision;
   glm::vec3 ray, tempColor;
      //for (int j = 0; j < height; j++) {
   currentImgInd = col * height + row;
   ray = pixelsD[currentImgInd].pt;// - cPos;
   ray = glm::normalize(ray);

   /*if (currentImgInd == 0) {
      //printf("First Obj Type: %d\n", objArr[0]->type);
      OctTreeNode* o = (OctTreeNode*)objArr[0];
      
      printf("Huh %d\n", o->type);
      printf("ImgInd: %d\n", currentImgInd);
      
      for(int i = 0; i < 8; i++) {
         if (o->octants[i] != NULL) {
            printf("Oct Type: %d\n", o->octants[i]->type);
         } else {
            printf("Null Oct: %d\n", i);
         }
      }
      printf("Tree size: %d\n", o->treeLength());
   }*/
//if (row == 320 && col == 320) {
   printf("hello?\n");
   collision = raytracer.trace(cPos, ray, false);
   printf("whats up?\n");
         //printf("Making sure ;)\n");
   if (collision->time > TOLERANCE) {
      //pixels[i][j].clr = raytrace->calcRadiance(cPos, cPos + ray * col->time, col->object, unit, 1.0f, 1.33f, 0.95f, 5); //Cam must start in air
      tempColor = collision->object->getNormal(cPos + ray * collision->time, 2.0f);
      pixelsD[currentImgInd].clr = (tempColor * 0.5f) + 0.5f;
      //printf("Anything here?\n");
   }
   else {
      pixelsD[currentImgInd].clr = glm::vec3(1.0f, 1.0f, 1.0f);
      //printf("Making sure\n");
   }
   
   delete(collision);
}
   //}
         //cout << "PIXCOL: " << pixels[i][j].clr.r << " " << pixels[i][j].clr.g << " " << pixels[i][j].clr.b << endl;
      //}
   //}
   __syncthreads();
   
   
   /*for (int i = 0; i < width * height; i++) {
      pixelsD[i].clr.x = pixelsD[i].clr.y = pixelsD[i].clr.z = 1.0;
   }*/
}

// Set the card up to run cuda
void RayTraceOnDevice(int width, int height, Pixel *pixels, std::vector<SceneObject*> objects, Camera *cam) {
   SceneObject** objArrD = NULL;
   SceneObject** objArrH = NULL;// = &objects[0];
   int* sizesH = NULL;
   int* sizesD = NULL;
   int gridDimension = 0;
   
   Triangle* tempOctTree = NULL;
   Camera* cameraD = NULL;
   Pixel* pixelsD = NULL;
   RayTracer* raytracer = NULL;
   int tempSize = 0, tempInd = 0, pixelArrSize = 0;
   
   dim3 dimGrid = dim3(0,0);
   dim3 dimBlock = dim3(0,0);
   
   pixelArrSize = width * height * sizeof(Pixel);
   printf("Transferring Data...\n");
   printf("Size parray: %d\n", pixelArrSize);

   HANDLE_ERROR(hipMalloc(&pixelsD, pixelArrSize));
   HANDLE_ERROR(hipMemcpy(pixelsD, pixels, pixelArrSize, hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMalloc(&cameraD, sizeof(Camera)));
   HANDLE_ERROR(hipMemcpy(cameraD, cam, sizeof(Camera), hipMemcpyHostToDevice));
   
   //Allocate memory for all needed object pointers and their relative sizes
   HANDLE_ERROR(hipMalloc(&objArrD, objects.size() * sizeof(SceneObject*)));
   HANDLE_ERROR(hipMalloc(&sizesD, objects.size() * sizeof(int)));
   
   //Allocate memory on host for device pointers and object sizes
   printf("suh\n");
   objArrH = (SceneObject**)malloc(objects.size() * sizeof(SceneObject*));
   printf("dud\n");
   sizesH = (int*)malloc(objects.size() * sizeof(int));
   printf("bruh\n");
   
   //printf("OBJ SIZE: %d\n", objects.size());
   for (uint i = 0; i < objects.size(); i++) {
      //printf("hello\n");
      if (objects[i]->type == 8) { //Oct tree handling
         //printf("yes\n");
         tempInd = 0; //yah
         //Get the proper size of the tree
         tempSize = static_cast<OctTreeNode*>(objects[i])->treeLength();
         sizesH[i] = tempSize;
         
         //Copy over array represenation of tree
         printf("TREE SIZE: %d\n", tempSize);
         tempOctTree = (Triangle*)calloc(tempSize, sizeof(Triangle));
         printf("ARR SIZE: %lu\n", tempSize * sizeof(Triangle));
         printf("ARR START POS: %p\n", tempOctTree);
         printf("frick\n");
         printf("TRI SIZE %lu\n", sizeof(Triangle));
         printf("OCT SIZE %lu\n", sizeof(OctTreeNode));
         reinterpret_cast<OctTreeNode*>(objects[i])->toSerialArray(tempOctTree, &tempInd);
         printf("frack\n");
         HANDLE_ERROR(hipMalloc(&(objArrH[i]), tempSize * sizeof(Triangle)));
         printf("whats\n");
         HANDLE_ERROR(hipMemcpy(objArrH[i], tempOctTree, tempSize * sizeof(Triangle), hipMemcpyHostToDevice));
         printf("that\n");
         free(tempOctTree);
         printf("OH FUCK\n");
         
         //Rebuild tree on device
         gridDimension = int(ceil(sqrt((float(tempSize)) / (float(TILEWIDTH * TILEWIDTH)))));
         printf("GD: %d\n", gridDimension);
         dimGrid = dim3(gridDimension, gridDimension);//dim3((tempSize / TILEWIDTH) + 1, 1);
         dimBlock = dim3(TILEWIDTH,TILEWIDTH);//dim3(TILEWIDTH, 1);
         //printf("kk\n");
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
         toOctTree<<<dimGrid, dimBlock>>>((Triangle*)(objArrH[i]), tempSize, gridDimension);
         //printf("kkk\n");
         HANDLE_ERROR(hipPeekAtLastError());
         HANDLE_ERROR(hipDeviceSynchronize());
   
      } else if (objects[i]->type == 5) { //Quad tree handling (Gerstner Wave Triangles)
         sizesH[i] = 1;
      } else { //Other object handling (mostly, if not all, planes)
         sizesH[i] = 1;
      }
   }
   
   //Copy device pointers from host array to device array and sizes of each obj pointer struct
   HANDLE_ERROR(hipMemcpy(objArrD, objArrH, objects.size() * sizeof(SceneObject*), hipMemcpyHostToDevice));
   HANDLE_ERROR(hipMemcpy(sizesD, sizesH, objects.size() * sizeof(int), hipMemcpyHostToDevice));
   
   printf("hiya\n");
   raytracer = new RayTracer(objArrD, objects.size(), 0, 0, NULL, NULL);
   printf("fam\n");
   
   //Make blocks of dimension 32x32, with 32x32 threads on them
   dimGrid = dim3(width / TILEWIDTH, height / TILEWIDTH);
   dimBlock = dim3(TILEWIDTH, TILEWIDTH);
   
   printf("Calling Kernel...\n");
   //Run the Kernel code
   //hipDeviceSetLimit(hipLimitMallocHeapSize, size_t(3000000000));
   GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, sizesD, objects.size(), pixelsD, cameraD, width, height, *raytracer);
   //GIPhotonMapKernel<<<dimGrid, dimBlock>>>(objArrD, objects.size(), pixelsD, cameraD, width, height);
   HANDLE_ERROR(hipPeekAtLastError());
   HANDLE_ERROR(hipDeviceSynchronize());
   printf("SIZE GLM VEC: %lu\n", sizeof(glm::vec3));
   printf("SIZE PIXEL: %lu\n", sizeof(Pixel));
   //Copy modified pixel array from card back to host
   printf("Size parray: %d\n", pixelArrSize);
   HANDLE_ERROR(hipMemcpy(pixels, pixelsD, pixelArrSize, hipMemcpyDeviceToHost));
   
   //Free the data on the card
   printf("Freeing data on the card...\n");
   //Camera
   HANDLE_ERROR(hipFree(cameraD));
   printf("it\n");
   //Objects
   for (uint i = 0; i < objects.size(); i++) {
      HANDLE_ERROR(hipFree(objArrH[i]));
   }
   printf("is\n");
   HANDLE_ERROR(hipFree(objArrD));
   HANDLE_ERROR(hipFree(sizesD));
   //Pixels
   HANDLE_ERROR(hipFree(pixelsD));
   //free(pixelsD);
   //KDTrees
   printf("where\n");
   
   //Free other data
   delete raytracer;
   free(sizesH);
   free(objArrH);
}
