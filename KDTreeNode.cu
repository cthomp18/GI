#include "hip/hip_runtime.h"
/*
   Cody Thompson
   Photon Mapping
*/

//#include "KDTreeNode.h"
#include "KDTreeNode.cuh"

using namespace std;

KDTreeNode::KDTreeNode(KDTreeNode *l, KDTreeNode *r, Photon *p, int a) {
   left = l;
   right = r;
   photon = p;
   axis = a;
   
   leftInd = rightInd = -1;
}

KDTreeNode::KDTreeNode() {
   left = NULL;
   right = NULL;
   photon = NULL;
   axis = -1;
}

KDTreeNode::~KDTreeNode() {
   if (left) delete left;
   if (right) delete right;
   if (photon) delete photon;
}

KDTreeNode* KDTreeNode::buildKDTree(std::vector<Photon*> pmap, int lastAxis) {
   //Build KDTree
   KDTreeNode *node;// = new KDTreeNode(NULL, NULL, NULL, -1);
   if (pmap.empty()) return new KDTreeNode(NULL, NULL, NULL, -1);;

   std::vector<Photon*> subtreeL, subtreeR;
   Photon* p;
   int median, sortAxis;
   
   //Find Best Dimension
   float xMin, yMin, zMin, xMax, yMax, zMax;
   float dx = 0.0, dy = 0.0, dz = 0.0;
   if (lastAxis != 0) {
      xMin = findMin(pmap, 0);
      xMax = findMax(pmap, 0);
      dx = xMax - xMin;
   }
   if (lastAxis != 1) {
      yMin = findMin(pmap, 1);
      yMax = findMax(pmap, 1);
      dy = yMax - yMin;
   }
   if (lastAxis != 2) {
      zMin = findMin(pmap, 2);
      zMax = findMax(pmap, 2);
      dz = zMax - zMin;
   }

   if (dx >= dy && dx >= dz) {
      sortAxis = 0;
   } else if (dy >= dx && dy >= dz) {
      sortAxis = 1;
   } else {
      sortAxis = 2;
   }
   //cout << pmap.size() << endl;
   std::vector<Photon*> tempPMap;
   tempPMap.clear();
   for (uint i = 0; i < pmap.size(); i++) {
      pmap[i]->sortAxis = sortAxis;
      tempPMap.push_back(pmap[i]);
   }
   //Sort by photons by value of best dimension
   //cout << "Here?" << endl;
   std::sort(tempPMap.begin(), tempPMap.end(), compPhotons);
   //cout << "yeeeeeee" << endl;
   for (uint i = 0; i < pmap.size(); i++) {
      //p = new Photon(tempPMap[i].pt, tempPMap[i].incidence, tempPMap[i].intensity, tempPMap[i].type);
      pmap[i] = tempPMap[i];//p;
   }
   //Get median point index
   median = (pmap.size()-1) / 2;
   //Set initial node properties
   //cout << "Boo" << endl;
   //cout << "median: " << median << endl;
   //cout << pmap[median]->pt.x() << " " << pmap[median]->pt.y() << " " << pmap[median]->pt.z() << endl;
   node = new KDTreeNode(NULL, NULL, pmap[median], sortAxis);
   //cout << "yee" << endl;
   
   //Put points in corresponding subtrees by dim value (and delete previous PMap)
   for (int i = 0; i < median; i++) {
      subtreeL.push_back(pmap[i]);
   }
   for (uint i = median+1; i < pmap.size(); i++) {
      subtreeR.push_back(pmap[i]);
   }
   /*while(!pmap.empty()) {
		delete pmap.back();
		pmap.pop_back();
	}*/
	
   //Form node's children
   if (subtreeL.size() > 0) node->left = buildKDTree(subtreeL, sortAxis);
   else node->left = NULL;
   if (subtreeR.size() > 0) node->right = buildKDTree(subtreeR, sortAxis);
   else node->right = NULL;
   
   return node;
}

//recursive variant
/*void KDTreeNode::locatePhotons(int i, glm::vec3 pt, Photon** locateHeap, int *heapSize, float sampleDistSqrd, float *newRadSqrd, glm::mat3 mInv, int numPhotons) {
   glm::vec3 rayBetween = pt - photon->pt;
   float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
   
   if (2*i + 1 < numPhotons) {
      float distToPlane = 0.0f;
      //Find distance to plane (difference WRT splitting axis)
      if (axis == 0) distToPlane = pt.x - photon->pt.x;
      else if (axis == 1) distToPlane = pt.y - photon->pt.y;
      else if (axis == 2) distToPlane = pt.z - photon->pt.z;
      
      //Point is on the 'left' of the plane
      if (distToPlane < 0.0) {
         //Search on left child
         if (left != NULL) left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         //If distance to plane is less than the sample distance radius, then
         //sample sphere intersects plane, so check right child as well
         if (distToPlane*distToPlane < sampleDistSqrd) {
            if (right != NULL) right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         }
      } else {
         //Point on 'right' of plane
         //Search on right child
         if (right != NULL) right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         //If distance to plane is less than the sample distance radius, then
         //sample sphere intersects plane, so check left child as well
         if (distToPlane*distToPlane < sampleDistSqrd) {
            if (left != NULL) left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         }
      }
   }
   //Check if photon is close enough to the point
   if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
      glm::vec3 originLoc;
      originLoc = glm::vec3(photon->pt[0] - pt[0], photon->pt[1] - pt[1], photon->pt[2] - pt[2]);
      float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
      if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
         originLoc = originLoc * mInv;
      }
      if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0) {
         locateHeap[*heapSize] = photon;
         *heapSize += 1;
         //cout << "INCIDENCE: " << -node->photon->incidence[0] << " " << -node->photon->incidence[1] << " " << -node->photon->incidence[2] << endl;
         //cout << "INTENSITY: " << node->photon->intensity[0] << " " << node->photon->intensity[1] << " " << node->photon->intensity[2] << endl;
         if (*heapSize == CUTOFF_HEAP_SIZE) {
            *newRadSqrd = distToPhotonSqrd;
         }
      }
   }
}*/

//iterative variant
/*void KDTreeNode::locatePhotons(glm::vec3 pt, Photon** locateHeap, int *heapSize, float sampleDistSqrd, float *newRadSqrd, glm::mat3 mInv, int numPhotons, KDTreeNode** stack) {
   
   glm::vec3 rayBetween;// = pt - photon->pt;
   float distToPhotonSqrd;// = glm::length(rayBetween) * glm::length(rayBetween);
   //float rad;
   //KDTreeNode **stack;
   //glm::vec3 originLoc;
   
   int added = 0;
   int stackMade = 0;
   
   
   //float distToPlane;
   
   
   
      int depth = 1;
   
   KDTreeNode *currentNode;// = this;
   KDTreeNode *previousNode = this;
   int currentStackSpot = 0;
   //depth *= 2;
   //KDTreeNode **stack;
   // Allocate space for the stack of nodes for the iterative solution
   //printf("prob this then\n");
   if (stack == NULL) {
      int treeLength = numPhotons;
      // Find depth of tree (should be balanced, but depth will be one more just to be very safe)
      while (treeLength != 0) {
         depth++;
         treeLength /= 2;
      }
      //printf("HSADFAHSF\n");
      //KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      stackMade = 1;
   }
   //printf("where\n");
   //printf("confirming\n");
   //printf("HI\n");
   stack[currentStackSpot] = this;
   
   //printf("here?\n");
   //printf("DEPTH %d\n", depth);
   
   
   KDTreeNode *currentNode;// = this;
   KDTreeNode *previousNode = this;
   while (currentStackSpot >= 0) {
      //printf("%d\n", currentStackSpot);
      //printf("uhh\n");
      currentNode = stack[currentStackSpot];
      //printf("idk\n");
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //if (currentNode->left != NULL || currentNode->right != NULL) printf("k\n");
         //currentStackSpot++;
         //stack[currentStackSpot] = currentNode;
         //if (2*i + 1 < numPhotons) {
         float distToPlane = 0.0f;

         if (currentNode->axis == 0) distToPlane = pt.x - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) distToPlane = pt.y - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) distToPlane = pt.z - currentNode->photon->pt.z;
         
         if (distToPlane < 0.0) {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->right != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->right;
                  added = 1;
               }
            }
            if (currentNode->left != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->left;
               added = 1;
            }
         } else {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->left != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->left;
                  added = 1;
               }
            }
            if (currentNode->right != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->right;
               added = 1;
            }
         }
         
         // Add the current node back into the stack if it had children added
         // Otherwise modify the current stack spot
         if (!added) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            added = 0;
         }
         //}
      } else {
         //glm::vec3 rayBetween = pt - currentNode->photon->pt;
         //float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         //if (currentNode == NULL) printf("FUUUUUUUUUUCCCCCCCCCCCCCCCCCCCCKKKKKKKKKKKKKKKKKKKKKKKKKK\n");
         rayBetween = pt - currentNode->photon->pt;
         distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
            //printf("WWWOOOOOOOOOOOOOOOOOOOOOOOOO\n");
            glm::vec3 originLoc;
            //originLoc = currentNode->photon->pt - pt;
            
            originLoc = glm::vec3(currentNode->photon->pt[0] - pt[0], currentNode->photon->pt[1] - pt[1], currentNode->photon->pt[2] - pt[2]);
            float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            //rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               originLoc = originLoc * mInv;
            }
            //if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               //originLoc = originLoc * mInv;
            //   originLoc[0] = (originLoc[0] * mInv[0][0]) + (originLoc[0] * mInv[0][1]) + (originLoc[0] * mInv[0][2]);
            //   originLoc[1] = (originLoc[0] * mInv[1][0]) + (originLoc[1] * mInv[1][1]) + (originLoc[1] * mInv[1][2]);
            //   originLoc[2] = (originLoc[0] * mInv[2][0]) + (originLoc[2] * mInv[2][1]) + (originLoc[2] * mInv[2][2]);
            //}
            //originLoc[0] = mInv[0][0] + mInv[0][1] + mInv[0][2];
            //originLoc[1] = mInv[1][0] + mInv[1][1] + mInv[1][2];
            //originLoc[2] = mInv[2][0] + mInv[2][1] + mInv[2][2];
            //float oneoversampleDistsqrd = 1.0f / sampleDistSqrd;
            //rad = rad* rad);
            //float conditional = originLoc[0]*originLoc[0];
            //conditional = conditional + originLoc[1]*originLoc[1];
            //conditional = conditional / sampleDistSqrd;
            //rad = rad * rad;
            //originLoc[0] = originLoc[0] * originLoc[0];
            //originLoc[0] = originLoc[0] / sampleDistSqrd;
            //originLoc[1] = originLoc[1] * originLoc[1];
            //originLoc[1] = originLoc[1] / sampleDistSqrd;
            //originLoc[2] = originLoc[2] * originLoc[2];
            //originLoc[2] = originLoc[2] / rad;
            //conditional = conditional * rad;
            //conditional = conditional + originLoc[2]*originLoc[2];
            //conditional = conditional / rad;
            if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])*oneoversampleDistsqrd) + ((originLoc[1]*originLoc[1])*oneoversampleDistsqrd) + ((originLoc[2]*originLoc[2])*oneoverradsqrd) < 1.0) {
            //if (conditional < 1.0f) {
               //printf("YOOOOOOOOOOOOOOOOOOOOOOOOO: %d\n", *heapSize);
               locateHeap[*heapSize] = currentNode->photon;
               *heapSize += 1;
               if (*heapSize == CUTOFF_HEAP_SIZE) {
                  *newRadSqrd = distToPhotonSqrd;
                  return;
               }
            }
         }
         previousNode = currentNode;
         currentStackSpot--;
      }
   }
   if (stackMade) {
     printf("FREEING STACK\n");   
      free(stack);
   }
}*/

__device__ __noinline__ void KDTreeNode::locatePhotons(glm::vec3 pt, Photon** locateHeap, volatile float * volatile mInv, int numPhotons, float *shF, int *shI) {
   //F09-F11: temporary vector
   //F12: temporary local
   //F13: temporary local
   //F14: sample squared distance
   //F15: new squared radius if needed

   //I13: temporary local
   //I14: current stack spot
   //I15: size of the photon heap
   
   //NOTE: PLEASE SEE THE BELOW COMMENTED FUNCTION FOR BETTER VARIABLE USE AS COMPARISON
   
   volatile int depth = 1;
   while (numPhotons != 0) {
      depth++;
      numPhotons /= 2;
   }
   KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));

   //int added = 0;
   shI[0] = 0;
   
   volatile KDTreeNode *currentNode;// = this;
   volatile KDTreeNode *previousNode = this;
   //volatile int currentStackSpot = 0;
   //stack[currentStackSpot] = this;
   shI[1] = 0;
   stack[shI[1]] = this;
   while (shI[1] >= 0) {
   //while(currentStackSpot >= 0) {
      currentNode = stack[shI[1]];
      //currentNode = stack[currentStackSpot];
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //shF[6] = 0.0f;

         if (currentNode->axis == 0) shF[6] = pt.x - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) shF[6] = pt.y - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) shF[6] = pt.z - currentNode->photon->pt.z;
         
         if (shF[6] < 0.0) {
            if (shF[6]*shF[6] < shF[7]) {
               if (currentNode->right != NULL) {
                  shI[1] += 1;
                  stack[shI[1]] = currentNode->right;
                  //currentStackSpot++;
                  //stack[currentStackSpot] = currentNode->right;
                  //added = 1;
                  shI[0] = 1;
               }
            }
            if (currentNode->left != NULL) {
               shI[1] += 1;
               stack[shI[1]] = currentNode->left;
               //currentStackSpot++;
               //stack[currentStackSpot] = currentNode->left;
               //added = 1;
               shI[0] = 1;
            }
         } else {
            if (shF[6]*shF[6] < shF[7]) {
               if (currentNode->left != NULL) {
                  shI[1] += 1;
                  stack[shI[1]] = currentNode->left;
                  //currentStackSpot++;
                  //stack[currentStackSpot] = currentNode->left;
                  //added = 1;
                  shI[0] = 1;
               }
            }
            if (currentNode->right != NULL) {
               shI[1] += 1;
               stack[shI[1]] = currentNode->right;
               //currentStackSpot++;
               //stack[currentStackSpot] = currentNode->right;
               //added = 1;
               shI[0] = 1;
            }
         }
         
         // Add the current node back into the stack if it had children added
         // Otherwise modify the current stack spot
         if (!(shI[0])) {
         //if (!added) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            shI[0] = 0;
            //added = 0;
         }
         //}
         //shI[1] = -1;
      } else {
         glm::vec3 rayBetween = pt - currentNode->photon->pt;
         //float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         shF[6] = glm::length(rayBetween) * glm::length(rayBetween);
         //distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         if (shF[6] <= shF[7] && shI[2] < CUTOFF_HEAP_SIZE) {
         //if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
            //glm::vec3 originLoc = glm::vec3(currentNode->photon->pt[0] - pt[0], currentNode->photon->pt[1] - pt[1], currentNode->photon->pt[2] - pt[2]);
            //shF[2] = currentNode->photon->pt[0] - pt[0];'
            volatile float f1 = currentNode->photon->pt[0] - pt[0];
            shF[3] = currentNode->photon->pt[1] - pt[1];
            shF[4] = currentNode->photon->pt[2] - pt[2];
            
            //shF[5] = sqrt(shF[7]) * ELLIPSOID_SCALE;
            //rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            //if (fabs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               //originLoc = originLoc * mInv;
            //}
            //if (fabs(ELLIPSOID_SCALE - 1.0f) > TOLERANCE) {
               //originLoc = originLoc * mInv;
               //shF[2] = (shF[0] * mInv[0][0]) + (shF[1] * mInv[0][1]) + (shF[4] * mInv[0][2]);
               //shF[3] = (shF[0] * mInv[1][0]) + (shF[1] * mInv[1][1]) + (shF[4] * mInv[1][2]);
               //shF[4] = (shF[0] * mInv[2][0]) + (shF[1] * mInv[2][1]) + (shF[4] * mInv[2][2]);
               //shF[2] = (shF[0] * mInv.value[0].x) + (shF[1] * mInv.value[0].y) + (shF[4] * mInv.value[0].z);
               //shF[3] = (shF[0] * mInv.value[1].x) + (shF[1] * mInv.value[1].y) + (shF[4] * mInv.value[1].z);
               //shF[4] = (shF[0] * mInv.value[2].x) + (shF[1] * mInv.value[2].y) + (shF[4] * mInv.value[2].z);
               //volatile float f1 = ((currentNode->photon->pt[0] - pt[0]) * mInv[0]) + (shF[3] * mInv[1]) + (shF[4] * mInv[2]);//, 
               //float f1 = (shF[2] * mInv[3]) + (shF[3] * mInv[4]) + (shF[4] * mInv[5]);
               shF[5] = (f1 * mInv[3]) + (shF[3] * mInv[4]) + (shF[4] * mInv[5]);
               //shF[2] = (shF[0] * mInv[0]) + (shF[1] * mInv[1]) + (shF[4] * mInv[2]);
               //f1 = (shF[2] * mInv[0]) + (shF[3] * mInv[1]) + (shF[4] * mInv[2]);
               shF[5] = shF[5] * shF[5];
               //f1 = ((currentNode->photon->pt[0] - pt[0]) * mInv[3]) + (shF[3] * mInv[4]) + (shF[4] * mInv[5]);
               //f1 = (shF[2] * mInv[3]) + (shF[3] * mInv[4]) + (shF[4] * mInv[5]);
               volatile float f2 = (f1 * mInv[3]) + (shF[3] * mInv[4]) + (shF[4] * mInv[5]);
               shF[5] += f2 * f2;
               //f1 = f1 * f1 + shF[5];
               //shF[4] = ((currentNode->photon->pt[0] - pt[0]) * mInv[6]) + (shF[3] * mInv[7]) + (shF[4] * mInv[8]);
               shF[4] = (f1 * mInv[6]) + (shF[3] * mInv[7]) + (shF[4] * mInv[8]);
               //shF[2] = f1;
               //shF[3] = f2;
            //}
            
            f1 = shF[7] * ELLIPSOID_SCALE * ELLIPSOID_SCALE;
            //if (((shF[2]*shF[2])/shF[7]) + ((shF[3]*shF[3])/shF[7]) + ((shF[4]*shF[4])/(shF[5]*shF[5])) < 1.0f) {
            //if ((f1 / shF[7]) + ((shF[4]*shF[4])/shF[5]) < 1.0f) {
            if ((shF[5] / shF[7]) + ((shF[4]*shF[4])/f1) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])*oneoversampleDistsqrd) + ((originLoc[1]*originLoc[1])*oneoversampleDistsqrd) + ((originLoc[2]*originLoc[2])*oneoverradsqrd) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])/shF[7]) + ((originLoc[1]*originLoc[1])/shF[7]) + ((originLoc[2]*originLoc[2])/(shF[5]*shF[5])) < 1.0f) {
               locateHeap[shI[2]] = currentNode->photon;
               //locateHeap[*heapSize] = currentNode->photon;
               shI[2] += 1;
               //*heapSize += 1;
               if (shI[2] == CUTOFF_HEAP_SIZE) {
               //if (*heapSize == CUTOFF_HEAP_SIZE) {
                  shF[8] = shF[6];
                  return;
               }
            }
         }
         previousNode = currentNode;
         shI[1] -= 1;
         //currentStackSpot--;
      }
   }
   
   free(stack);
}

/*void KDTreeNode::locatePhotons(glm::vec3 pt, int ts, Photon** locateHeap, float sampleDistSqrd, float *newRadSqrd, int numPhotons, float *sh) {
      //glm::vec3 rayBetween;// = pt - photon->pt;
   //float distToPhotonSqrd;// = glm::length(rayBetween) * glm::length(rayBetween);
   
   //KDTreeNode **stack;
    
   int added = 0;
   //int stackMade = 0;

   //depth *= 2;
   //KDTreeNode **stack;
   // Allocate space for the stack of nodes for the iterative solution
   //printf("prob this then\n");
   //if (stack == NULL) {
      volatile int treeLength = numPhotons;
      volatile int depth = 1;
      // Find depth of tree (should be balanced, but depth will be one more just to be very safe)
      while (treeLength != 0) {
         depth++;
         treeLength /= 2;
      }
      //printf("HSADFAHSF\n");
      KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      //stackMade = 1;
   //}
   //printf("where\n");
   //printf("confirming\n");
   //printf("HI\n");
   stack[0] = this;
   
   //printf("here?\n");
   //printf("DEPTH %d\n", depth);
   
   volatile KDTreeNode *currentNode;// = this;
   volatile KDTreeNode *previousNode = this;
   volatile int currentStackSpot = 0;
   
   while (currentStackSpot >= 0) {
      //printf("%d\n", currentStackSpot);
      //printf("uhh\n");
      currentNode = stack[currentStackSpot];
      //printf("idk\n");
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //if (currentNode->left != NULL || currentNode->right != NULL) printf("k\n");
         //currentStackSpot++;
         //stack[currentStackSpot] = currentNode;
         //if (2*i + 1 < numPhotons) {
         float distToPlane = 0.0f;

         if (currentNode->axis == 0) distToPlane = pt.x - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) distToPlane = pt.y - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) distToPlane = pt.z - currentNode->photon->pt.z;
         
         if (distToPlane < 0.0) {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->right != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->right;
                  added = 1;
               }
            }
            if (currentNode->left != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->left;
               added = 1;
            }
         } else {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->left != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->left;
                  added = 1;
               }
            }
            if (currentNode->right != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->right;
               added = 1;
            }
         }
         
         // Add the current node back into the stack if it had children added
         // Otherwise modify the current stack spot
         if (!added) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            added = 0;
         }
         //}
      } else {
         glm::vec3 rayBetween = pt - currentNode->photon->pt;
         volatile float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         if (distToPhotonSqrd <= sampleDistSqrd && (int)shF[ts] < CUTOFF_HEAP_SIZE) {
         //if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
            glm::vec3 originLoc = currentNode->photon->pt - pt;
            //glm::vec3 originLoc(currentNode->photon->pt.x - shF[pt], currentNode->photon->pt.y - shF[pt+1], currentNode->photon->pt.z - shF[pt+2]);
            //originLoc = glm::vec3(currentNode->photon->pt[0] - pt[0], currentNode->photon->pt[1] - pt[1], currentNode->photon->pt[2] - pt[2]);
            volatile float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            //if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               //originLoc = originLoc * mInv;
               //originLoc[0] = (originLoc[0] * mInv[0][0]) + (originLoc[0] * mInv[0][1]) + (originLoc[0] * mInv[0][2]);
               //originLoc[1] = (originLoc[0] * mInv[1][0]) + (originLoc[1] * mInv[1][1]) + (originLoc[1] * mInv[1][2]);
               //originLoc[2] = (originLoc[0] * mInv[2][0]) + (originLoc[2] * mInv[2][1]) + (originLoc[2] * mInv[2][2]);
            //}

            if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])*oneoversampleDistsqrd) + ((originLoc[1]*originLoc[1])*oneoversampleDistsqrd) + ((originLoc[2]*originLoc[2])*oneoverradsqrd) < 1.0) {
            //if (conditional < 1.0f) {
               locateHeap[(int)shF[ts]] = currentNode->photon;
               shF[ts] += 1;
               if ((int)shF[ts] == CUTOFF_HEAP_SIZE) {
                  *newRadSqrd = distToPhotonSqrd;
                  return;
               }
            }
            //if (((shF[pt+4]*shF[pt+4])/shF[pt+9]) + ((shF[pt+5]*shF[pt+5])/shF[pt+9]) + ((shF[pt+6]*shF[pt+6])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
               //locateHeap[(int)shF[pt+10]] = currentNode->photon;
               //shF[pt+10] += 1;
               //if ((int)shF[pt+10] == CUTOFF_HEAP_SIZE) {
               //   *nrs = shF[pt+7];
               //   return;
               //}
            //}
         }
         previousNode = currentNode;
         currentStackSpot--;
      }
   }
   //if (stackMade) {
   //  printf("FREEING STACK\n");   
   //   free(stack);
   //}
}*/


/*void KDTreeNode::locatePhotons(int pt, volatile Photon** locateHeap, float sampleDistSqrd, float *newRadSqrd, int numPhotons, float *sh) {
   //extern __shared__ float shF[];
   //volatile int added = 0;
   shF[pt+5] = 0.001;

      volatile int treeLength = numPhotons;
      volatile int depth = 1;
      while (treeLength != 0) {
         depth++;
         treeLength /= 2;
      }
      KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      printf("DEPTH:%d\n", depth);
   stack[0] = this;
   
   volatile KDTreeNode *currentNode;// = this;
   volatile KDTreeNode *previousNode = this;
   shF[pt+8] = 0.1;
   volatile KDTreeNode *what;
   shF[pt+9] = sampleDistSqrd;
   volatile float * nrs = newRadSqrd;
   
   while (shF[pt+8] >= 0) {
      currentNode = stack[(int)shF[pt+8]];
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //volatile float distToPlane = 0.0f;
         shF[pt+7] = 0.0f;
         //volatile glm::vec3
         if (currentNode->axis == 0) shF[pt+7] = shF[pt] - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) shF[pt+7] = shF[pt+1] - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) shF[pt+7] = shF[pt+2] - currentNode->photon->pt.z;
         
         if (shF[pt+7] < 0.0) {
            if (shF[pt+7]*shF[pt+7] < shF[pt+9]) {
               if (currentNode->right != NULL) {
                  shF[pt+8]+=1;
                  stack[(int)shF[pt+8]] = currentNode->right;
                  //what = currentNode->right;
                  //stack[(int)shF[pt+8]] = what;
                  shF[pt+5] = 1.1;
               }
            }
            if (currentNode->left != NULL) {
               shF[pt+8]+=1;
               stack[(int)shF[pt+8]] = currentNode->left;
               //what = currentNode->left;
               //stack[(int)shF[pt+8]] = what;
               shF[pt+5] = 1.1;
            }
         } else {
            if (shF[pt+7]*shF[pt+7] < shF[pt+9]) {
               if (currentNode->left != NULL) {
                  shF[pt+8]+=1;
                  stack[(int)shF[pt+8]] = currentNode->left;
                  //what = currentNode->left;
                  //stack[(int)shF[pt+8]] = what;
                  shF[pt+5] = 1.1;
               }
            }
            if (currentNode->right != NULL) {
               shF[pt+8]+=1;
               stack[(int)shF[pt+8]] = currentNode->right;
               //what = currentNode->right;
               //stack[(int)shF[pt+8]] = what;
               shF[pt+5] = 1.1;
            }
         }
         
         if (!((int)shF[pt+5])) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            shF[pt+5] = 0.1;
         }
      } else {
         //glm::vec3 rayBetween;
         //rayBetween.x = shF[pt] - currentNode->photon->pt.x;
         //rayBetween.y = shF[pt+1] - currentNode->photon->pt.y;
         //rayBetween.z = shF[pt+2] - currentNode->photon->pt.z;
         //shF[pt+4] = shF[pt] - currentNode->photon->pt.x;
         //shF[pt+5] = shF[pt+1] - currentNode->photon->pt.y;
         //shF[pt+6] = shF[pt+2] - currentNode->photon->pt.z;
         //volatile glm::vec3 rayBetween = glm::vec3(shF[pt] - currentNode->photon->pt.x, shF[pt+1] - currentNode->photon->pt.y, shF[pt+2] - currentNode->photon->pt.z);
         glm::vec3 rayBetween = glm::vec3(shF[pt] - currentNode->photon->pt.x, shF[pt+1] - currentNode->photon->pt.y, shF[pt+2] - currentNode->photon->pt.z);
         //volatile float sum = shF[pt+4] * shF[pt+4];
         //sum += shF[pt+5] * shF[pt+5];
         //sum += shF[pt+6] * shF[pt+6];
         //volatile float rb = sqrt(rayBetween.x * rayBetween.x + rayBetween.y * rayBetween.y + rayBetween.z * rayBetween.z);
         //volatile float rb = sqrt(sum);
         //volatile float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         shF[pt+7] = glm::length(rayBetween) * glm::length(rayBetween);
         if (shF[pt+7] <= shF[pt+9] && shF[pt+10] < CUTOFF_HEAP_SIZE) {
            glm::vec3 originLoc(currentNode->photon->pt.x - shF[pt], currentNode->photon->pt.y - shF[pt+1], currentNode->photon->pt.z - shF[pt+2]);
            //shF[pt+4] = currentNode->photon->pt.x - shF[pt];
            //shF[pt+5] = currentNode->photon->pt.y - shF[pt+1];
            //shF[pt+6] = currentNode->photon->pt.z - shF[pt+2];
            
            volatile float rad = sqrt(shF[pt+9]) * ELLIPSOID_SCALE;
            //shF[pt+6] = sqrt(shF[pt+9]) * ELLIPSOID_SCALE;
            //volatile float sds = sampleDistSqrd;
            //if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
            //   shF[pt+4] = (shF[pt+4] * mInv[0][0]) + (shF[pt+4] * mInv[0][1]) + (shF[pt+4] * mInv[0][2]);
            //   shF[pt+5] = (shF[pt+5] * mInv[1][0]) + (shF[pt+5] * mInv[1][1]) + (shF[pt+5] * mInv[1][2]);
            //   shF[pt+6] = (shF[pt+6] * mInv[2][0]) + (shF[pt+6] * mInv[2][1]) + (shF[pt+6] * mInv[2][2]);
            //}
            if (((shF[pt+4]*shF[pt+4])/shF[pt+9]) + ((shF[pt+5]*shF[pt+5])/shF[pt+9]) + ((shF[pt+6]*shF[pt+6])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
               locateHeap[(int)shF[pt+10]] = currentNode->photon;
               shF[pt+10] += 1;
               if ((int)shF[pt+10] == CUTOFF_HEAP_SIZE) {
                  *nrs = shF[pt+7];
                  return;
               }
            }
         }
         previousNode = currentNode;
         shF[pt+8]-=1;
      }
   }
}*/

int KDTreeNode::Treesize() {
   int num = 1;
   if (this->left != NULL) num += this->left->Treesize();
   if (this->right != NULL) num += this->right->Treesize();
   return num;
}
void KDTreeNode::printTree(KDTreeNode *node) {
   //std::cout << "Pt: " << glm::to_string(node->photon->pt) << std::endl;
   printf("Pt: %f %f %f\n", node->photon->pt.x, node->photon->pt.y, node->photon->pt.z);
   printf("Axis Split: %d\n", node->axis);
   if (node->left != NULL) {
      printf("LEFT\n");
      printTree(node->left);
      printf("BACK\n");
   }
   if (node->right != NULL) {
      printf("RIGHT\n");
      printTree(node->right);
      printf("BACK\n");
   }
}

float KDTreeNode::findMin(std::vector<Photon*> pmap, int axis) {
   float min = 0.0f;
   for (uint i = 0; i < pmap.size(); i++) {
      if (i == 0 || pmap[i]->pt[axis] < min) {
         min = pmap[i]->pt[axis];
      }
   }
   return min;
}

float KDTreeNode::findMax(std::vector<Photon*> pmap, int axis) {
   float max = 0.0f;
   for (uint i = 0; i < pmap.size(); i++) {
      if (i == 0 || pmap[i]->pt[axis] > max) {
         max = pmap[i]->pt[axis];
      }
   }
   
   /* CODY, THIS IS WRONG, YOU SHOULD KNOW THAT.
   YOU WANT TO RETURN MIN */
   return max;
}

void KDTreeNode::toSerialArray(Photon *objectArray, int *currentIndex) {
   int i;
   int thisInd = *currentIndex;
   *currentIndex += 2;
   
   /*memcpy(objectArray + (*currentIndex), this, sizeof(KDTreeNode));
   *currentIndex += 1;
   memcpy(objectArray + (*currentIndex), photon, sizeof(Photon));
   *currentIndex += 1;*/
   
   if (left) {
      leftInd = *currentIndex;
      left->toSerialArray(objectArray, currentIndex);
   }
   if (right) {
      rightInd = *currentIndex;
      right->toSerialArray(objectArray, currentIndex);
   }
   
   memcpy(objectArray + thisInd, this, sizeof(KDTreeNode));
   memcpy(objectArray + thisInd + 1, photon, sizeof(Photon));
}

bool compPhotons(Photon* p1, Photon* p2) { 
   if (p1->sortAxis == 0) {
      return p1->pt.x < p2->pt.x;
   } else if (p1->sortAxis == 1) {
      return p1->pt.y < p2->pt.y;
   } else if (p1->sortAxis == 2) {
      return p1->pt.z < p2->pt.z;
   } else {
      return false; //fuck
   }
}
