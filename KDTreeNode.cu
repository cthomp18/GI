#include "hip/hip_runtime.h"
/*
   Cody Thompson
   Photon Mapping
*/

#include "KDTreeNode.h"

using namespace std;

KDTreeNode::KDTreeNode(KDTreeNode *l, KDTreeNode *r, Photon *p, int a) {
   left = l;
   right = r;
   photon = p;
   axis = a;
   
   leftInd = rightInd = -1;
}

KDTreeNode::KDTreeNode() {
   left = NULL;
   right = NULL;
   photon = NULL;
   axis = -1;
}

KDTreeNode::~KDTreeNode() {
   if (left) delete left;
   if (right) delete right;
   if (photon) delete photon;
}

KDTreeNode* KDTreeNode::buildKDTree(std::vector<Photon*> pmap, int lastAxis) {
   //Build KDTree
   KDTreeNode *node;// = new KDTreeNode(NULL, NULL, NULL, -1);
   if (pmap.empty()) return new KDTreeNode(NULL, NULL, NULL, -1);;

   std::vector<Photon*> subtreeL, subtreeR;
   Photon* p;
   int median, sortAxis;
   
   //Find Best Dimension
   float xMin, yMin, zMin, xMax, yMax, zMax;
   float dx = 0.0, dy = 0.0, dz = 0.0;
   if (lastAxis != 0) {
      xMin = findMin(pmap, 0);
      xMax = findMax(pmap, 0);
      dx = xMax - xMin;
   }
   if (lastAxis != 1) {
      yMin = findMin(pmap, 1);
      yMax = findMax(pmap, 1);
      dy = yMax - yMin;
   }
   if (lastAxis != 2) {
      zMin = findMin(pmap, 2);
      zMax = findMax(pmap, 2);
      dz = zMax - zMin;
   }

   if (dx >= dy && dx >= dz) {
      sortAxis = 0;
   } else if (dy >= dx && dy >= dz) {
      sortAxis = 1;
   } else {
      sortAxis = 2;
   }
   //cout << pmap.size() << endl;
   std::vector<Photon*> tempPMap;
   tempPMap.clear();
   for (uint i = 0; i < pmap.size(); i++) {
      pmap[i]->sortAxis = sortAxis;
      tempPMap.push_back(pmap[i]);
   }
   //Sort by photons by value of best dimension
   //cout << "Here?" << endl;
   std::sort(tempPMap.begin(), tempPMap.end(), compPhotons);
   //cout << "yeeeeeee" << endl;
   for (uint i = 0; i < pmap.size(); i++) {
      //p = new Photon(tempPMap[i].pt, tempPMap[i].incidence, tempPMap[i].intensity, tempPMap[i].type);
      pmap[i] = tempPMap[i];//p;
   }
   //Get median point index
   median = (pmap.size()-1) / 2;
   //Set initial node properties
   //cout << "Boo" << endl;
   //cout << "median: " << median << endl;
   //cout << pmap[median]->pt.x() << " " << pmap[median]->pt.y() << " " << pmap[median]->pt.z() << endl;
   node = new KDTreeNode(NULL, NULL, pmap[median], sortAxis);
   //cout << "yee" << endl;
   
   //Put points in corresponding subtrees by dim value (and delete previous PMap)
   for (int i = 0; i < median; i++) {
      subtreeL.push_back(pmap[i]);
   }
   for (uint i = median+1; i < pmap.size(); i++) {
      subtreeR.push_back(pmap[i]);
   }
   /*while(!pmap.empty()) {
		delete pmap.back();
		pmap.pop_back();
	}*/
	
   //Form node's children
   if (subtreeL.size() > 0) node->left = buildKDTree(subtreeL, sortAxis);
   else node->left = NULL;
   if (subtreeR.size() > 0) node->right = buildKDTree(subtreeR, sortAxis);
   else node->right = NULL;
   
   return node;
}

//recursive variant
/*void KDTreeNode::locatePhotons(int i, glm::vec3 pt, Photon** locateHeap, int *heapSize, float sampleDistSqrd, float *newRadSqrd, glm::mat3 mInv, int numPhotons) {
   glm::vec3 rayBetween = pt - photon->pt;
   float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
   
   if (2*i + 1 < numPhotons) {
      float distToPlane = 0.0f;
      //Find distance to plane (difference WRT splitting axis)
      if (axis == 0) distToPlane = pt.x - photon->pt.x;
      else if (axis == 1) distToPlane = pt.y - photon->pt.y;
      else if (axis == 2) distToPlane = pt.z - photon->pt.z;
      
      //Point is on the 'left' of the plane
      if (distToPlane < 0.0) {
         //Search on left child
         if (left != NULL) left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         //If distance to plane is less than the sample distance radius, then
         //sample sphere intersects plane, so check right child as well
         if (distToPlane*distToPlane < sampleDistSqrd) {
            if (right != NULL) right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         }
      } else {
         //Point on 'right' of plane
         //Search on right child
         if (right != NULL) right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         //If distance to plane is less than the sample distance radius, then
         //sample sphere intersects plane, so check left child as well
         if (distToPlane*distToPlane < sampleDistSqrd) {
            if (left != NULL) left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         }
      }
   }
   //Check if photon is close enough to the point
   if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
      glm::vec3 originLoc;
      originLoc = glm::vec3(photon->pt[0] - pt[0], photon->pt[1] - pt[1], photon->pt[2] - pt[2]);
      float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
      if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
         originLoc = originLoc * mInv;
      }
      if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0) {
         locateHeap[*heapSize] = photon;
         *heapSize += 1;
         //cout << "INCIDENCE: " << -node->photon->incidence[0] << " " << -node->photon->incidence[1] << " " << -node->photon->incidence[2] << endl;
         //cout << "INTENSITY: " << node->photon->intensity[0] << " " << node->photon->intensity[1] << " " << node->photon->intensity[2] << endl;
         if (*heapSize == CUTOFF_HEAP_SIZE) {
            *newRadSqrd = distToPhotonSqrd;
         }
      }
   }
}*/

//iterative variant
void KDTreeNode::locatePhotons(int i, glm::vec3 pt, Photon** locateHeap, int *heapSize, float sampleDistSqrd, float *newRadSqrd, glm::mat3 mInv, int numPhotons, KDTreeNode **stack) {
   
   glm::vec3 rayBetween;// = pt - photon->pt;
   float distToPhotonSqrd;// = glm::length(rayBetween) * glm::length(rayBetween);
   
   //KDTreeNode **stack;
   
   
   int depth = 1;
   int added = 0;
   int stackMade = 0;
   
   KDTreeNode *currentNode;// = this;
   KDTreeNode *previousNode = this;
   
   int currentStackSpot = 0;
   
   
   
   //depth *= 2;
   //KDTreeNode *stack[30];
   // Allocate space for the stack of nodes for the iterative solution
   //printf("prob this then\n");
   if (stack == NULL) {
      int treeLength = numPhotons;
      
      // Find depth of tree (should be balanced, but depth will be one more just to be very safe)
      while (treeLength != 0) {
         depth++;
         treeLength /= 2;
      }
      printf("HSADFAHSF\n");
      stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      stackMade = 1;
   }
   //printf("where\n");
   //printf("confirming\n");
   //printf("HI\n");
   stack[currentStackSpot] = this;
   
   //printf("here?\n");
   //printf("DEPTH %d\n", depth);
   while (currentStackSpot >= 0) {
      //printf("%d\n", currentStackSpot);
      //printf("uhh\n");
      currentNode = stack[currentStackSpot];
      //printf("idk\n");
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //if (currentNode->left != NULL || currentNode->right != NULL) printf("k\n");
         //currentStackSpot++;
         //stack[currentStackSpot] = currentNode;
         //if (2*i + 1 < numPhotons) {
         float distToPlane = 0.0f;

         if (currentNode->axis == 0) distToPlane = pt.x - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) distToPlane = pt.y - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) distToPlane = pt.z - currentNode->photon->pt.z;
         
         if (distToPlane < 0.0) {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->right != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->right;
                  added = 1;
                  //right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
               }
            }
            if (currentNode->left != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->left;
               added = 1;
               //left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
            }
         } else {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->left != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->left;
                  added = 1;
                  //left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
               }
            }
            if (currentNode->right != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->right;
               added = 1;
               //right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
            }
         }
         
         // Add the current node back into the stack if it had children added
         // Otherwise modify the current stack spot
         if (!added) {
            //currentStackSpot--;
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            added = 0;
         }
         //}
      } else {
         rayBetween = pt - currentNode->photon->pt;
         distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
            glm::vec3 originLoc;
            originLoc = glm::vec3(currentNode->photon->pt[0] - pt[0], currentNode->photon->pt[1] - pt[1], currentNode->photon->pt[2] - pt[2]);
            float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               originLoc = originLoc * mInv;
            }
            if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0) {
               locateHeap[*heapSize] = currentNode->photon;
               *heapSize += 1;
               if (*heapSize == CUTOFF_HEAP_SIZE) {
                  *newRadSqrd = distToPhotonSqrd;
                  return;
               }
            }
         }
         previousNode = currentNode;
         currentStackSpot--;
      }
   }
   if (stackMade) {
      printf("FREEING STACK\n");   
      free(stack);
   }
}

int KDTreeNode::Treesize() {
   int num = 1;
   if (this->left != NULL) num += this->left->Treesize();
   if (this->right != NULL) num += this->right->Treesize();
   return num;
}
void KDTreeNode::printTree(KDTreeNode *node) {
   //std::cout << "Pt: " << glm::to_string(node->photon->pt) << std::endl;
   printf("Pt: %f %f %f\n", node->photon->pt.x, node->photon->pt.y, node->photon->pt.z);
   printf("Axis Split: %d\n", node->axis);
   if (node->left != NULL) {
      printf("LEFT\n");
      printTree(node->left);
      printf("BACK\n");
   }
   if (node->right != NULL) {
      printf("RIGHT\n");
      printTree(node->right);
      printf("BACK\n");
   }
}

float KDTreeNode::findMin(std::vector<Photon*> pmap, int axis) {
   float min = 0.0f;
   for (uint i = 0; i < pmap.size(); i++) {
      if (i == 0 || pmap[i]->pt[axis] < min) {
         min = pmap[i]->pt[axis];
      }
   }
   return min;
}

float KDTreeNode::findMax(std::vector<Photon*> pmap, int axis) {
   float max = 0.0f;
   for (uint i = 0; i < pmap.size(); i++) {
      if (i == 0 || pmap[i]->pt[axis] > max) {
         max = pmap[i]->pt[axis];
      }
   }
   
   /* CODY, THIS IS WRONG, YOU SHOULD KNOW THAT.
   YOU WANT TO RETURN MIN */
   return max;
}

void KDTreeNode::toSerialArray(Photon *objectArray, int *currentIndex) {
   int i;
   int thisInd = *currentIndex;
   *currentIndex += 2;
   
   /*memcpy(objectArray + (*currentIndex), this, sizeof(KDTreeNode));
   *currentIndex += 1;
   memcpy(objectArray + (*currentIndex), photon, sizeof(Photon));
   *currentIndex += 1;*/
   
   if (left) {
      leftInd = *currentIndex;
      left->toSerialArray(objectArray, currentIndex);
   }
   if (right) {
      rightInd = *currentIndex;
      right->toSerialArray(objectArray, currentIndex);
   }
   
   memcpy(objectArray + thisInd, this, sizeof(KDTreeNode));
   memcpy(objectArray + thisInd + 1, photon, sizeof(Photon));
}

bool compPhotons(Photon* p1, Photon* p2) { 
   if (p1->sortAxis == 0) {
      return p1->pt.x < p2->pt.x;
   } else if (p1->sortAxis == 1) {
      return p1->pt.y < p2->pt.y;
   } else if (p1->sortAxis == 2) {
      return p1->pt.z < p2->pt.z;
   } else {
      return false; //fuck
   }
}
