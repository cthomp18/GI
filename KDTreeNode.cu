#include "hip/hip_runtime.h"
/*
   Cody Thompson
   Photon Mapping
*/

//#include "KDTreeNode.h"
#include "KDTreeNode.cuh"

using namespace std;

KDTreeNode::KDTreeNode(KDTreeNode *l, KDTreeNode *r, Photon *p, int a) {
   left = l;
   right = r;
   photon = p;
   axis = a;
   
   leftInd = rightInd = -1;
}

KDTreeNode::KDTreeNode() {
   left = NULL;
   right = NULL;
   photon = NULL;
   axis = -1;
}

KDTreeNode::~KDTreeNode() {
   if (left) delete left;
   if (right) delete right;
   if (photon) delete photon;
}

KDTreeNode* KDTreeNode::buildKDTree(std::vector<Photon*> pmap, int lastAxis) {
   //Build KDTree
   KDTreeNode *node;// = new KDTreeNode(NULL, NULL, NULL, -1);
   if (pmap.empty()) return new KDTreeNode(NULL, NULL, NULL, -1);;

   std::vector<Photon*> subtreeL, subtreeR;
   Photon* p;
   int median, sortAxis;
   
   //Find Best Dimension
   float xMin, yMin, zMin, xMax, yMax, zMax;
   float dx = 0.0, dy = 0.0, dz = 0.0;
   if (lastAxis != 0) {
      xMin = findMin(pmap, 0);
      xMax = findMax(pmap, 0);
      dx = xMax - xMin;
   }
   if (lastAxis != 1) {
      yMin = findMin(pmap, 1);
      yMax = findMax(pmap, 1);
      dy = yMax - yMin;
   }
   if (lastAxis != 2) {
      zMin = findMin(pmap, 2);
      zMax = findMax(pmap, 2);
      dz = zMax - zMin;
   }

   if (dx >= dy && dx >= dz) {
      sortAxis = 0;
   } else if (dy >= dx && dy >= dz) {
      sortAxis = 1;
   } else {
      sortAxis = 2;
   }
   //cout << pmap.size() << endl;
   std::vector<Photon*> tempPMap;
   tempPMap.clear();
   for (uint i = 0; i < pmap.size(); i++) {
      pmap[i]->sortAxis = sortAxis;
      tempPMap.push_back(pmap[i]);
   }
   //Sort by photons by value of best dimension
   std::sort(tempPMap.begin(), tempPMap.end(), compPhotons);
   for (uint i = 0; i < pmap.size(); i++) {
      //p = new Photon(tempPMap[i].pt, tempPMap[i].incidence, tempPMap[i].intensity, tempPMap[i].type);
      pmap[i] = tempPMap[i];//p;
   }
   //Get median point index
   median = (pmap.size()-1) / 2;
   //Set initial node properties
   node = new KDTreeNode(NULL, NULL, pmap[median], sortAxis);
   
   //Put points in corresponding subtrees by dim value (and delete previous PMap)
   for (int i = 0; i < median; i++) {
      subtreeL.push_back(pmap[i]);
   }
   for (uint i = median+1; i < pmap.size(); i++) {
      subtreeR.push_back(pmap[i]);
   }
	
   //Form node's children
   if (subtreeL.size() > 0) node->left = buildKDTree(subtreeL, sortAxis);
   else node->left = NULL;
   if (subtreeR.size() > 0) node->right = buildKDTree(subtreeR, sortAxis);
   else node->right = NULL;
   
   return node;
}

//recursive variant
/*void KDTreeNode::locatePhotons(int i, glm::vec3 pt, Photon** locateHeap, int *heapSize, float sampleDistSqrd, float *newRadSqrd, glm::mat3 mInv, int numPhotons) {
   glm::vec3 rayBetween = pt - photon->pt;
   float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
   
   if (2*i + 1 < numPhotons) {
      float distToPlane = 0.0f;
      //Find distance to plane (difference WRT splitting axis)
      if (axis == 0) distToPlane = pt.x - photon->pt.x;
      else if (axis == 1) distToPlane = pt.y - photon->pt.y;
      else if (axis == 2) distToPlane = pt.z - photon->pt.z;
      
      //Point is on the 'left' of the plane
      if (distToPlane < 0.0) {
         //Search on left child
         if (left != NULL) left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         //If distance to plane is less than the sample distance radius, then
         //sample sphere intersects plane, so check right child as well
         if (distToPlane*distToPlane < sampleDistSqrd) {
            if (right != NULL) right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         }
      } else {
         //Point on 'right' of plane
         //Search on right child
         if (right != NULL) right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         //If distance to plane is less than the sample distance radius, then
         //sample sphere intersects plane, so check left child as well
         if (distToPlane*distToPlane < sampleDistSqrd) {
            if (left != NULL) left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         }
      }
   }
   //Check if photon is close enough to the point
   if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
      glm::vec3 originLoc;
      originLoc = glm::vec3(photon->pt[0] - pt[0], photon->pt[1] - pt[1], photon->pt[2] - pt[2]);
      float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
      if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
         originLoc = originLoc * mInv;
      }
      if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0) {
         locateHeap[*heapSize] = photon;
         *heapSize += 1;
         //cout << "INCIDENCE: " << -node->photon->incidence[0] << " " << -node->photon->incidence[1] << " " << -node->photon->incidence[2] << endl;
         //cout << "INTENSITY: " << node->photon->intensity[0] << " " << node->photon->intensity[1] << " " << node->photon->intensity[2] << endl;
         if (*heapSize == CUTOFF_HEAP_SIZE) {
            *newRadSqrd = distToPhotonSqrd;
         }
      }
   }
}*/

//iterative variant
/*void KDTreeNode::locatePhotons(glm::vec3 pt, Photon** locateHeap, int *heapSize, float sampleDistSqrd, float *newRadSqrd, glm::mat3 mInv, int numPhotons, KDTreeNode** stack) {
   
   glm::vec3 rayBetween;// = pt - photon->pt;
   float distToPhotonSqrd;// = glm::length(rayBetween) * glm::length(rayBetween);
   //float rad;
   //KDTreeNode **stack;
   //glm::vec3 originLoc;
   
   int added = 0;
   int stackMade = 0;
   
   
   //float distToPlane;
   
   
   
      int depth = 1;
   
   KDTreeNode *currentNode;// = this;
   KDTreeNode *previousNode = this;
   int currentStackSpot = 0;
   //depth *= 2;
   //KDTreeNode **stack;
   // Allocate space for the stack of nodes for the iterative solution
   if (stack == NULL) {
      int treeLength = numPhotons;
      // Find depth of tree (should be balanced, but depth will be one more just to be very safe)
      while (treeLength != 0) {
         depth++;
         treeLength /= 2;
      }
      //KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      stackMade = 1;
   }
   stack[currentStackSpot] = this;
   
   KDTreeNode *currentNode;// = this;
   KDTreeNode *previousNode = this;
   while (currentStackSpot >= 0) {
      currentNode = stack[currentStackSpot];
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //if (currentNode->left != NULL || currentNode->right != NULL) printf("k\n");
         //currentStackSpot++;
         //stack[currentStackSpot] = currentNode;
         //if (2*i + 1 < numPhotons) {
         float distToPlane = 0.0f;

         if (currentNode->axis == 0) distToPlane = pt.x - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) distToPlane = pt.y - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) distToPlane = pt.z - currentNode->photon->pt.z;
         
         if (distToPlane < 0.0) {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->right != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->right;
                  added = 1;
               }
            }
            if (currentNode->left != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->left;
               added = 1;
            }
         } else {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->left != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->left;
                  added = 1;
               }
            }
            if (currentNode->right != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->right;
               added = 1;
            }
         }
         
         // Add the current node back into the stack if it had children added
         // Otherwise modify the current stack spot
         if (!added) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            added = 0;
         }
         //}
      } else {
         //glm::vec3 rayBetween = pt - currentNode->photon->pt;
         //float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         rayBetween = pt - currentNode->photon->pt;
         distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
            glm::vec3 originLoc;
            //originLoc = currentNode->photon->pt - pt;
            
            originLoc = glm::vec3(currentNode->photon->pt[0] - pt[0], currentNode->photon->pt[1] - pt[1], currentNode->photon->pt[2] - pt[2]);
            float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            //rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               originLoc = originLoc * mInv;
            }
            //if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               //originLoc = originLoc * mInv;
            //   originLoc[0] = (originLoc[0] * mInv[0][0]) + (originLoc[0] * mInv[0][1]) + (originLoc[0] * mInv[0][2]);
            //   originLoc[1] = (originLoc[0] * mInv[1][0]) + (originLoc[1] * mInv[1][1]) + (originLoc[1] * mInv[1][2]);
            //   originLoc[2] = (originLoc[0] * mInv[2][0]) + (originLoc[2] * mInv[2][1]) + (originLoc[2] * mInv[2][2]);
            //}
            //originLoc[0] = mInv[0][0] + mInv[0][1] + mInv[0][2];
            //originLoc[1] = mInv[1][0] + mInv[1][1] + mInv[1][2];
            //originLoc[2] = mInv[2][0] + mInv[2][1] + mInv[2][2];
            //float oneoversampleDistsqrd = 1.0f / sampleDistSqrd;
            //rad = rad* rad);
            //float conditional = originLoc[0]*originLoc[0];
            //conditional = conditional + originLoc[1]*originLoc[1];
            //conditional = conditional / sampleDistSqrd;
            //rad = rad * rad;
            //originLoc[0] = originLoc[0] * originLoc[0];
            //originLoc[0] = originLoc[0] / sampleDistSqrd;
            //originLoc[1] = originLoc[1] * originLoc[1];
            //originLoc[1] = originLoc[1] / sampleDistSqrd;
            //originLoc[2] = originLoc[2] * originLoc[2];
            //originLoc[2] = originLoc[2] / rad;
            //conditional = conditional * rad;
            //conditional = conditional + originLoc[2]*originLoc[2];
            //conditional = conditional / rad;
            if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])*oneoversampleDistsqrd) + ((originLoc[1]*originLoc[1])*oneoversampleDistsqrd) + ((originLoc[2]*originLoc[2])*oneoverradsqrd) < 1.0) {
            //if (conditional < 1.0f) {
               locateHeap[*heapSize] = currentNode->photon;
               *heapSize += 1;
               if (*heapSize == CUTOFF_HEAP_SIZE) {
                  *newRadSqrd = distToPhotonSqrd;
                  return;
               }
            }
         }
         previousNode = currentNode;
         currentStackSpot--;
      }
   }
   if (stackMade) {
      free(stack);
   }
}*/

__device__ __noinline__ void KDTreeNode::locatePhotons(glm::vec3 pt, Photon** locateHeap, volatile float * volatile mInv, int numPhotons, float *shF, int *shI) {
   //F09-F11: temporary vector
   //F12: temporary local
   //F13: temporary local
   //F14: sample squared distance
   //F15: new squared radius if needed

   //I13: temporary local
   //I14: current stack spot
   //I15: size of the photon heap
   
   //NOTE: PLEASE SEE THE BELOW COMMENTED FUNCTION FOR BETTER VARIABLE USE AS COMPARISON
   
   volatile int depth = 1;
   while (numPhotons != 0) {
      depth++;
      numPhotons /= 2;
   }
   KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));

   shI[0] = 0;
   
   volatile KDTreeNode *currentNode;// = this;
   volatile KDTreeNode *previousNode = this;
   shI[1] = 0;
   stack[shI[1]] = this;
   while (shI[1] >= 0) {
      currentNode = stack[shI[1]];
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {

         if (currentNode->axis == 0) shF[6] = pt.x - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) shF[6] = pt.y - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) shF[6] = pt.z - currentNode->photon->pt.z;
         
         if (shF[6] < 0.0) {
            if (shF[6]*shF[6] < shF[7]) {
               if (currentNode->right != NULL) {
                  shI[1] += 1;
                  stack[shI[1]] = currentNode->right;
                  shI[0] = 1;
               }
            }
            if (currentNode->left != NULL) {
               shI[1] += 1;
               stack[shI[1]] = currentNode->left;
               shI[0] = 1;
            }
         } else {
            if (shF[6]*shF[6] < shF[7]) {
               if (currentNode->left != NULL) {
                  shI[1] += 1;
                  stack[shI[1]] = currentNode->left;
                  shI[0] = 1;
               }
            }
            if (currentNode->right != NULL) {
               shI[1] += 1;
               stack[shI[1]] = currentNode->right;
               shI[0] = 1;
            }
         }
         
         // Add the current node back into the stack if it had children added
         // Otherwise modify the current stack spot
         if (!(shI[0])) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            shI[0] = 0;
         }
      } else {
         glm::vec3 rayBetween = pt - currentNode->photon->pt;
         shF[6] = glm::length(rayBetween) * glm::length(rayBetween);
         if (shF[6] <= shF[7] && shI[2] < CUTOFF_HEAP_SIZE) {
            volatile float f1 = currentNode->photon->pt[0] - pt[0];
            shF[3] = currentNode->photon->pt[1] - pt[1];
            shF[4] = currentNode->photon->pt[2] - pt[2];
            
               shF[5] = (f1 * mInv[3]) + (shF[3] * mInv[4]) + (shF[4] * mInv[5]);
               shF[5] = shF[5] * shF[5];
               volatile float f2 = (f1 * mInv[3]) + (shF[3] * mInv[4]) + (shF[4] * mInv[5]);
               shF[5] += f2 * f2;
               shF[4] = (f1 * mInv[6]) + (shF[3] * mInv[7]) + (shF[4] * mInv[8]);
            
            f1 = shF[7] * ELLIPSOID_SCALE * ELLIPSOID_SCALE;
            if ((shF[5] / shF[7]) + ((shF[4]*shF[4])/f1) < 1.0f) {
               locateHeap[shI[2]] = currentNode->photon;
               shI[2] += 1;
               if (shI[2] == CUTOFF_HEAP_SIZE) {
                  shF[8] = shF[6];
                  return;
               }
            }
         }
         previousNode = currentNode;
         shI[1] -= 1;
      }
   }
   
   free(stack);
}

int KDTreeNode::Treesize() {
   int num = 1;
   if (this->left != NULL) num += this->left->Treesize();
   if (this->right != NULL) num += this->right->Treesize();
   return num;
}

void KDTreeNode::printTree(KDTreeNode *node) {
   //std::cout << "Pt: " << glm::to_string(node->photon->pt) << std::endl;
   printf("Pt: %f %f %f\n", node->photon->pt.x, node->photon->pt.y, node->photon->pt.z);
   printf("Axis Split: %d\n", node->axis);
   if (node->left != NULL) {
      printf("LEFT\n");
      printTree(node->left);
      printf("BACK\n");
   }
   if (node->right != NULL) {
      printf("RIGHT\n");
      printTree(node->right);
      printf("BACK\n");
   }
}

float KDTreeNode::findMin(std::vector<Photon*> pmap, int axis) {
   float min = 0.0f;
   for (uint i = 0; i < pmap.size(); i++) {
      if (i == 0 || pmap[i]->pt[axis] < min) {
         min = pmap[i]->pt[axis];
      }
   }
   return min;
}

float KDTreeNode::findMax(std::vector<Photon*> pmap, int axis) {
   float max = 0.0f;
   for (uint i = 0; i < pmap.size(); i++) {
      if (i == 0 || pmap[i]->pt[axis] > max) {
         max = pmap[i]->pt[axis];
      }
   }
   
   /* CODY, THIS IS WRONG, YOU SHOULD KNOW THAT.
   YOU WANT TO RETURN MIN */
   return max;
}

void KDTreeNode::toSerialArray(Photon *objectArray, int *currentIndex) {
   int i;
   int thisInd = *currentIndex;
   *currentIndex += 2;
   
   if (left) {
      leftInd = *currentIndex;
      left->toSerialArray(objectArray, currentIndex);
   }
   if (right) {
      rightInd = *currentIndex;
      right->toSerialArray(objectArray, currentIndex);
   }
   
   memcpy(objectArray + thisInd, this, sizeof(KDTreeNode));
   memcpy(objectArray + thisInd + 1, photon, sizeof(Photon));
}

bool compPhotons(Photon* p1, Photon* p2) { 
   if (p1->sortAxis == 0) {
      return p1->pt.x < p2->pt.x;
   } else if (p1->sortAxis == 1) {
      return p1->pt.y < p2->pt.y;
   } else if (p1->sortAxis == 2) {
      return p1->pt.z < p2->pt.z;
   } else {
      return false;
   }
}
