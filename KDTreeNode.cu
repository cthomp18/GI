#include "hip/hip_runtime.h"
/*
   Cody Thompson
   Photon Mapping
*/

#include "KDTreeNode.h"

using namespace std;

KDTreeNode::KDTreeNode(KDTreeNode *l, KDTreeNode *r, Photon *p, int a) {
   left = l;
   right = r;
   photon = p;
   axis = a;
   
   leftInd = rightInd = -1;
}

KDTreeNode::KDTreeNode() {
   left = NULL;
   right = NULL;
   photon = NULL;
   axis = -1;
}

KDTreeNode::~KDTreeNode() {
   if (left) delete left;
   if (right) delete right;
   if (photon) delete photon;
}

KDTreeNode* KDTreeNode::buildKDTree(std::vector<Photon*> pmap, int lastAxis) {
   //Build KDTree
   KDTreeNode *node;// = new KDTreeNode(NULL, NULL, NULL, -1);
   if (pmap.empty()) return new KDTreeNode(NULL, NULL, NULL, -1);;

   std::vector<Photon*> subtreeL, subtreeR;
   Photon* p;
   int median, sortAxis;
   
   //Find Best Dimension
   float xMin, yMin, zMin, xMax, yMax, zMax;
   float dx = 0.0, dy = 0.0, dz = 0.0;
   if (lastAxis != 0) {
      xMin = findMin(pmap, 0);
      xMax = findMax(pmap, 0);
      dx = xMax - xMin;
   }
   if (lastAxis != 1) {
      yMin = findMin(pmap, 1);
      yMax = findMax(pmap, 1);
      dy = yMax - yMin;
   }
   if (lastAxis != 2) {
      zMin = findMin(pmap, 2);
      zMax = findMax(pmap, 2);
      dz = zMax - zMin;
   }

   if (dx >= dy && dx >= dz) {
      sortAxis = 0;
   } else if (dy >= dx && dy >= dz) {
      sortAxis = 1;
   } else {
      sortAxis = 2;
   }
   //cout << pmap.size() << endl;
   std::vector<Photon*> tempPMap;
   tempPMap.clear();
   for (uint i = 0; i < pmap.size(); i++) {
      pmap[i]->sortAxis = sortAxis;
      tempPMap.push_back(pmap[i]);
   }
   //Sort by photons by value of best dimension
   //cout << "Here?" << endl;
   std::sort(tempPMap.begin(), tempPMap.end(), compPhotons);
   //cout << "yeeeeeee" << endl;
   for (uint i = 0; i < pmap.size(); i++) {
      //p = new Photon(tempPMap[i].pt, tempPMap[i].incidence, tempPMap[i].intensity, tempPMap[i].type);
      pmap[i] = tempPMap[i];//p;
   }
   //Get median point index
   median = (pmap.size()-1) / 2;
   //Set initial node properties
   //cout << "Boo" << endl;
   //cout << "median: " << median << endl;
   //cout << pmap[median]->pt.x() << " " << pmap[median]->pt.y() << " " << pmap[median]->pt.z() << endl;
   node = new KDTreeNode(NULL, NULL, pmap[median], sortAxis);
   //cout << "yee" << endl;
   
   //Put points in corresponding subtrees by dim value (and delete previous PMap)
   for (int i = 0; i < median; i++) {
      subtreeL.push_back(pmap[i]);
   }
   for (uint i = median+1; i < pmap.size(); i++) {
      subtreeR.push_back(pmap[i]);
   }
   /*while(!pmap.empty()) {
		delete pmap.back();
		pmap.pop_back();
	}*/
	
   //Form node's children
   if (subtreeL.size() > 0) node->left = buildKDTree(subtreeL, sortAxis);
   else node->left = NULL;
   if (subtreeR.size() > 0) node->right = buildKDTree(subtreeR, sortAxis);
   else node->right = NULL;
   
   return node;
}

//recursive variant
/*void KDTreeNode::locatePhotons(int i, glm::vec3 pt, Photon** locateHeap, int *heapSize, float sampleDistSqrd, float *newRadSqrd, glm::mat3 mInv, int numPhotons) {
   glm::vec3 rayBetween = pt - photon->pt;
   float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
   
   if (2*i + 1 < numPhotons) {
      float distToPlane = 0.0f;
      //Find distance to plane (difference WRT splitting axis)
      if (axis == 0) distToPlane = pt.x - photon->pt.x;
      else if (axis == 1) distToPlane = pt.y - photon->pt.y;
      else if (axis == 2) distToPlane = pt.z - photon->pt.z;
      
      //Point is on the 'left' of the plane
      if (distToPlane < 0.0) {
         //Search on left child
         if (left != NULL) left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         //If distance to plane is less than the sample distance radius, then
         //sample sphere intersects plane, so check right child as well
         if (distToPlane*distToPlane < sampleDistSqrd) {
            if (right != NULL) right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         }
      } else {
         //Point on 'right' of plane
         //Search on right child
         if (right != NULL) right->locatePhotons(2*i + 1, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         //If distance to plane is less than the sample distance radius, then
         //sample sphere intersects plane, so check left child as well
         if (distToPlane*distToPlane < sampleDistSqrd) {
            if (left != NULL) left->locatePhotons(2*i, pt, locateHeap, heapSize, sampleDistSqrd, newRadSqrd, mInv, numPhotons);
         }
      }
   }
   //Check if photon is close enough to the point
   if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
      glm::vec3 originLoc;
      originLoc = glm::vec3(photon->pt[0] - pt[0], photon->pt[1] - pt[1], photon->pt[2] - pt[2]);
      float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
      if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
         originLoc = originLoc * mInv;
      }
      if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0) {
         locateHeap[*heapSize] = photon;
         *heapSize += 1;
         //cout << "INCIDENCE: " << -node->photon->incidence[0] << " " << -node->photon->incidence[1] << " " << -node->photon->incidence[2] << endl;
         //cout << "INTENSITY: " << node->photon->intensity[0] << " " << node->photon->intensity[1] << " " << node->photon->intensity[2] << endl;
         if (*heapSize == CUTOFF_HEAP_SIZE) {
            *newRadSqrd = distToPhotonSqrd;
         }
      }
   }
}*/

//iterative variant
void KDTreeNode::locatePhotons(glm::vec3 pt, Photon** locateHeap, int *heapSize, float sampleDistSqrd, float *newRadSqrd, glm::mat3 mInv, int numPhotons, KDTreeNode** stack) {
   
   glm::vec3 rayBetween;// = pt - photon->pt;
   float distToPhotonSqrd;// = glm::length(rayBetween) * glm::length(rayBetween);
   //float rad;
   //KDTreeNode **stack;
   //glm::vec3 originLoc;
   
   int added = 0;
   int stackMade = 0;
   
   
   //float distToPlane;
   
   
   
      int depth = 1;
   
   KDTreeNode *currentNode;// = this;
   KDTreeNode *previousNode = this;
   int currentStackSpot = 0;
   //depth *= 2;
   //KDTreeNode **stack;
   // Allocate space for the stack of nodes for the iterative solution
   //printf("prob this then\n");
   if (stack == NULL) {
      int treeLength = numPhotons;
      // Find depth of tree (should be balanced, but depth will be one more just to be very safe)
      while (treeLength != 0) {
         depth++;
         treeLength /= 2;
      }
      //printf("HSADFAHSF\n");
      //KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      stackMade = 1;
   }
   //printf("where\n");
   //printf("confirming\n");
   //printf("HI\n");
   stack[currentStackSpot] = this;
   
   //printf("here?\n");
   //printf("DEPTH %d\n", depth);
   
   
   
   while (currentStackSpot >= 0) {
      //printf("%d\n", currentStackSpot);
      //printf("uhh\n");
      currentNode = stack[currentStackSpot];
      //printf("idk\n");
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //if (currentNode->left != NULL || currentNode->right != NULL) printf("k\n");
         //currentStackSpot++;
         //stack[currentStackSpot] = currentNode;
         //if (2*i + 1 < numPhotons) {
         float distToPlane = 0.0f;

         if (currentNode->axis == 0) distToPlane = pt.x - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) distToPlane = pt.y - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) distToPlane = pt.z - currentNode->photon->pt.z;
         
         if (distToPlane < 0.0) {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->right != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->right;
                  added = 1;
               }
            }
            if (currentNode->left != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->left;
               added = 1;
            }
         } else {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->left != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->left;
                  added = 1;
               }
            }
            if (currentNode->right != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->right;
               added = 1;
            }
         }
         
         // Add the current node back into the stack if it had children added
         // Otherwise modify the current stack spot
         if (!added) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            added = 0;
         }
         //}
      } else {
         //glm::vec3 rayBetween = pt - currentNode->photon->pt;
         //float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         //if (currentNode == NULL) printf("FUUUUUUUUUUCCCCCCCCCCCCCCCCCCCCKKKKKKKKKKKKKKKKKKKKKKKKKK\n");
         rayBetween = pt - currentNode->photon->pt;
         distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
            //printf("WWWOOOOOOOOOOOOOOOOOOOOOOOOO\n");
            glm::vec3 originLoc;
            //originLoc = currentNode->photon->pt - pt;
            
            originLoc = glm::vec3(currentNode->photon->pt[0] - pt[0], currentNode->photon->pt[1] - pt[1], currentNode->photon->pt[2] - pt[2]);
            float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            //rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               originLoc = originLoc * mInv;
            }
            /*if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               //originLoc = originLoc * mInv;
               originLoc[0] = (originLoc[0] * mInv[0][0]) + (originLoc[0] * mInv[0][1]) + (originLoc[0] * mInv[0][2]);
               originLoc[1] = (originLoc[0] * mInv[1][0]) + (originLoc[1] * mInv[1][1]) + (originLoc[1] * mInv[1][2]);
               originLoc[2] = (originLoc[0] * mInv[2][0]) + (originLoc[2] * mInv[2][1]) + (originLoc[2] * mInv[2][2]);
            }*/
            //originLoc[0] = mInv[0][0] + mInv[0][1] + mInv[0][2];
            //originLoc[1] = mInv[1][0] + mInv[1][1] + mInv[1][2];
            //originLoc[2] = mInv[2][0] + mInv[2][1] + mInv[2][2];
            //float oneoversampleDistsqrd = 1.0f / sampleDistSqrd;
            //rad = rad* rad);
            //float conditional = originLoc[0]*originLoc[0];
            //conditional = conditional + originLoc[1]*originLoc[1];
            //conditional = conditional / sampleDistSqrd;
            //rad = rad * rad;
            //originLoc[0] = originLoc[0] * originLoc[0];
            //originLoc[0] = originLoc[0] / sampleDistSqrd;
            //originLoc[1] = originLoc[1] * originLoc[1];
            //originLoc[1] = originLoc[1] / sampleDistSqrd;
            //originLoc[2] = originLoc[2] * originLoc[2];
            //originLoc[2] = originLoc[2] / rad;
            //conditional = conditional * rad;
            //conditional = conditional + originLoc[2]*originLoc[2];
            //conditional = conditional / rad;
            if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])*oneoversampleDistsqrd) + ((originLoc[1]*originLoc[1])*oneoversampleDistsqrd) + ((originLoc[2]*originLoc[2])*oneoverradsqrd) < 1.0) {
            //if (conditional < 1.0f) {
               //printf("YOOOOOOOOOOOOOOOOOOOOOOOOO: %d\n", *heapSize);
               locateHeap[*heapSize] = currentNode->photon;
               *heapSize += 1;
               if (*heapSize == CUTOFF_HEAP_SIZE) {
                  *newRadSqrd = distToPhotonSqrd;
                  return;
               }
            }
         }
         previousNode = currentNode;
         currentStackSpot--;
      }
   }
   if (stackMade) {
     printf("FREEING STACK\n");   
      free(stack);
   }
}


/*void KDTreeNode::locatePhotons(glm::vec3 pt, int ts, Photon** locateHeap, float sampleDistSqrd, float *newRadSqrd, int numPhotons, float *sh) {
      //glm::vec3 rayBetween;// = pt - photon->pt;
   //float distToPhotonSqrd;// = glm::length(rayBetween) * glm::length(rayBetween);
   
   //KDTreeNode **stack;
    
   int added = 0;
   //int stackMade = 0;

   //depth *= 2;
   //KDTreeNode **stack;
   // Allocate space for the stack of nodes for the iterative solution
   //printf("prob this then\n");
   //if (stack == NULL) {
      volatile int treeLength = numPhotons;
      volatile int depth = 1;
      // Find depth of tree (should be balanced, but depth will be one more just to be very safe)
      while (treeLength != 0) {
         depth++;
         treeLength /= 2;
      }
      //printf("HSADFAHSF\n");
      KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      //stackMade = 1;
   //}
   //printf("where\n");
   //printf("confirming\n");
   //printf("HI\n");
   stack[0] = this;
   
   //printf("here?\n");
   //printf("DEPTH %d\n", depth);
   
   volatile KDTreeNode *currentNode;// = this;
   volatile KDTreeNode *previousNode = this;
   volatile int currentStackSpot = 0;
   
   while (currentStackSpot >= 0) {
      //printf("%d\n", currentStackSpot);
      //printf("uhh\n");
      currentNode = stack[currentStackSpot];
      //printf("idk\n");
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //if (currentNode->left != NULL || currentNode->right != NULL) printf("k\n");
         //currentStackSpot++;
         //stack[currentStackSpot] = currentNode;
         //if (2*i + 1 < numPhotons) {
         float distToPlane = 0.0f;

         if (currentNode->axis == 0) distToPlane = pt.x - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) distToPlane = pt.y - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) distToPlane = pt.z - currentNode->photon->pt.z;
         
         if (distToPlane < 0.0) {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->right != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->right;
                  added = 1;
               }
            }
            if (currentNode->left != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->left;
               added = 1;
            }
         } else {
            if (distToPlane*distToPlane < sampleDistSqrd) {
               if (currentNode->left != NULL) {
                  currentStackSpot++;
                  stack[currentStackSpot] = currentNode->left;
                  added = 1;
               }
            }
            if (currentNode->right != NULL) {
               currentStackSpot++;
               stack[currentStackSpot] = currentNode->right;
               added = 1;
            }
         }
         
         // Add the current node back into the stack if it had children added
         // Otherwise modify the current stack spot
         if (!added) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            added = 0;
         }
         //}
      } else {
         glm::vec3 rayBetween = pt - currentNode->photon->pt;
         volatile float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         if (distToPhotonSqrd <= sampleDistSqrd && (int)sh[ts] < CUTOFF_HEAP_SIZE) {
         //if (distToPhotonSqrd <= sampleDistSqrd && *heapSize < CUTOFF_HEAP_SIZE) {
            glm::vec3 originLoc = currentNode->photon->pt - pt;
            //glm::vec3 originLoc(currentNode->photon->pt.x - sh[pt], currentNode->photon->pt.y - sh[pt+1], currentNode->photon->pt.z - sh[pt+2]);
            //originLoc = glm::vec3(currentNode->photon->pt[0] - pt[0], currentNode->photon->pt[1] - pt[1], currentNode->photon->pt[2] - pt[2]);
            volatile float rad = sqrt(sampleDistSqrd) * ELLIPSOID_SCALE;
            //if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
               //originLoc = originLoc * mInv;
               //originLoc[0] = (originLoc[0] * mInv[0][0]) + (originLoc[0] * mInv[0][1]) + (originLoc[0] * mInv[0][2]);
               //originLoc[1] = (originLoc[0] * mInv[1][0]) + (originLoc[1] * mInv[1][1]) + (originLoc[1] * mInv[1][2]);
               //originLoc[2] = (originLoc[0] * mInv[2][0]) + (originLoc[2] * mInv[2][1]) + (originLoc[2] * mInv[2][2]);
            //}

            if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])*oneoversampleDistsqrd) + ((originLoc[1]*originLoc[1])*oneoversampleDistsqrd) + ((originLoc[2]*originLoc[2])*oneoverradsqrd) < 1.0) {
            //if (conditional < 1.0f) {
               locateHeap[(int)sh[ts]] = currentNode->photon;
               sh[ts] += 1;
               if ((int)sh[ts] == CUTOFF_HEAP_SIZE) {
                  *newRadSqrd = distToPhotonSqrd;
                  return;
               }
            }
            //if (((sh[pt+4]*sh[pt+4])/sh[pt+9]) + ((sh[pt+5]*sh[pt+5])/sh[pt+9]) + ((sh[pt+6]*sh[pt+6])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
               //locateHeap[(int)sh[pt+10]] = currentNode->photon;
               //sh[pt+10] += 1;
               //if ((int)sh[pt+10] == CUTOFF_HEAP_SIZE) {
               //   *nrs = sh[pt+7];
               //   return;
               //}
            //}
         }
         previousNode = currentNode;
         currentStackSpot--;
      }
   }
   //if (stackMade) {
   //  printf("FREEING STACK\n");   
   //   free(stack);
   //}
}*/


/*void KDTreeNode::locatePhotons(int pt, volatile Photon** locateHeap, float sampleDistSqrd, float *newRadSqrd, int numPhotons, float *sh) {
   //extern __shared__ float sh[];
   //volatile int added = 0;
   sh[pt+5] = 0.001;

      volatile int treeLength = numPhotons;
      volatile int depth = 1;
      while (treeLength != 0) {
         depth++;
         treeLength /= 2;
      }
      KDTreeNode **stack = (KDTreeNode **)malloc(depth * 2 * sizeof(KDTreeNode*));
      printf("DEPTH:%d\n", depth);
   stack[0] = this;
   
   volatile KDTreeNode *currentNode;// = this;
   volatile KDTreeNode *previousNode = this;
   sh[pt+8] = 0.1;
   volatile KDTreeNode *what;
   sh[pt+9] = sampleDistSqrd;
   volatile float * nrs = newRadSqrd;
   
   while (sh[pt+8] >= 0) {
      currentNode = stack[(int)sh[pt+8]];
      
      if (currentNode->left != previousNode && currentNode->right != previousNode &&
          (currentNode->left != NULL || currentNode->right != NULL)) {
         //volatile float distToPlane = 0.0f;
         sh[pt+7] = 0.0f;
         //volatile glm::vec3
         if (currentNode->axis == 0) sh[pt+7] = sh[pt] - currentNode->photon->pt.x;
         else if (currentNode->axis == 1) sh[pt+7] = sh[pt+1] - currentNode->photon->pt.y;
         else if (currentNode->axis == 2) sh[pt+7] = sh[pt+2] - currentNode->photon->pt.z;
         
         if (sh[pt+7] < 0.0) {
            if (sh[pt+7]*sh[pt+7] < sh[pt+9]) {
               if (currentNode->right != NULL) {
                  sh[pt+8]+=1;
                  stack[(int)sh[pt+8]] = currentNode->right;
                  //what = currentNode->right;
                  //stack[(int)sh[pt+8]] = what;
                  sh[pt+5] = 1.1;
               }
            }
            if (currentNode->left != NULL) {
               sh[pt+8]+=1;
               stack[(int)sh[pt+8]] = currentNode->left;
               //what = currentNode->left;
               //stack[(int)sh[pt+8]] = what;
               sh[pt+5] = 1.1;
            }
         } else {
            if (sh[pt+7]*sh[pt+7] < sh[pt+9]) {
               if (currentNode->left != NULL) {
                  sh[pt+8]+=1;
                  stack[(int)sh[pt+8]] = currentNode->left;
                  //what = currentNode->left;
                  //stack[(int)sh[pt+8]] = what;
                  sh[pt+5] = 1.1;
               }
            }
            if (currentNode->right != NULL) {
               sh[pt+8]+=1;
               stack[(int)sh[pt+8]] = currentNode->right;
               //what = currentNode->right;
               //stack[(int)sh[pt+8]] = what;
               sh[pt+5] = 1.1;
            }
         }
         
         if (!((int)sh[pt+5])) {
            if (currentNode->left == NULL) {
               previousNode = currentNode->right;
            } else {
               previousNode = currentNode->left;
            }
         } else {
            sh[pt+5] = 0.1;
         }
      } else {
         //glm::vec3 rayBetween;
         //rayBetween.x = sh[pt] - currentNode->photon->pt.x;
         //rayBetween.y = sh[pt+1] - currentNode->photon->pt.y;
         //rayBetween.z = sh[pt+2] - currentNode->photon->pt.z;
         //sh[pt+4] = sh[pt] - currentNode->photon->pt.x;
         //sh[pt+5] = sh[pt+1] - currentNode->photon->pt.y;
         //sh[pt+6] = sh[pt+2] - currentNode->photon->pt.z;
         //volatile glm::vec3 rayBetween = glm::vec3(sh[pt] - currentNode->photon->pt.x, sh[pt+1] - currentNode->photon->pt.y, sh[pt+2] - currentNode->photon->pt.z);
         glm::vec3 rayBetween = glm::vec3(sh[pt] - currentNode->photon->pt.x, sh[pt+1] - currentNode->photon->pt.y, sh[pt+2] - currentNode->photon->pt.z);
         //volatile float sum = sh[pt+4] * sh[pt+4];
         //sum += sh[pt+5] * sh[pt+5];
         //sum += sh[pt+6] * sh[pt+6];
         //volatile float rb = sqrt(rayBetween.x * rayBetween.x + rayBetween.y * rayBetween.y + rayBetween.z * rayBetween.z);
         //volatile float rb = sqrt(sum);
         //volatile float distToPhotonSqrd = glm::length(rayBetween) * glm::length(rayBetween);
         sh[pt+7] = glm::length(rayBetween) * glm::length(rayBetween);
         if (sh[pt+7] <= sh[pt+9] && sh[pt+10] < CUTOFF_HEAP_SIZE) {
            glm::vec3 originLoc(currentNode->photon->pt.x - sh[pt], currentNode->photon->pt.y - sh[pt+1], currentNode->photon->pt.z - sh[pt+2]);
            //sh[pt+4] = currentNode->photon->pt.x - sh[pt];
            //sh[pt+5] = currentNode->photon->pt.y - sh[pt+1];
            //sh[pt+6] = currentNode->photon->pt.z - sh[pt+2];
            
            volatile float rad = sqrt(sh[pt+9]) * ELLIPSOID_SCALE;
            //sh[pt+6] = sqrt(sh[pt+9]) * ELLIPSOID_SCALE;
            //volatile float sds = sampleDistSqrd;
            //if (abs(ELLIPSOID_SCALE - 1.0) > TOLERANCE) {
            //   sh[pt+4] = (sh[pt+4] * mInv[0][0]) + (sh[pt+4] * mInv[0][1]) + (sh[pt+4] * mInv[0][2]);
            //   sh[pt+5] = (sh[pt+5] * mInv[1][0]) + (sh[pt+5] * mInv[1][1]) + (sh[pt+5] * mInv[1][2]);
            //   sh[pt+6] = (sh[pt+6] * mInv[2][0]) + (sh[pt+6] * mInv[2][1]) + (sh[pt+6] * mInv[2][2]);
            //}
            if (((sh[pt+4]*sh[pt+4])/sh[pt+9]) + ((sh[pt+5]*sh[pt+5])/sh[pt+9]) + ((sh[pt+6]*sh[pt+6])/(rad*rad)) < 1.0f) {
            //if (((originLoc[0]*originLoc[0])/sampleDistSqrd) + ((originLoc[1]*originLoc[1])/sampleDistSqrd) + ((originLoc[2]*originLoc[2])/(rad*rad)) < 1.0f) {
               locateHeap[(int)sh[pt+10]] = currentNode->photon;
               sh[pt+10] += 1;
               if ((int)sh[pt+10] == CUTOFF_HEAP_SIZE) {
                  *nrs = sh[pt+7];
                  return;
               }
            }
         }
         previousNode = currentNode;
         sh[pt+8]-=1;
      }
   }
}*/

int KDTreeNode::Treesize() {
   int num = 1;
   if (this->left != NULL) num += this->left->Treesize();
   if (this->right != NULL) num += this->right->Treesize();
   return num;
}
void KDTreeNode::printTree(KDTreeNode *node) {
   //std::cout << "Pt: " << glm::to_string(node->photon->pt) << std::endl;
   printf("Pt: %f %f %f\n", node->photon->pt.x, node->photon->pt.y, node->photon->pt.z);
   printf("Axis Split: %d\n", node->axis);
   if (node->left != NULL) {
      printf("LEFT\n");
      printTree(node->left);
      printf("BACK\n");
   }
   if (node->right != NULL) {
      printf("RIGHT\n");
      printTree(node->right);
      printf("BACK\n");
   }
}

float KDTreeNode::findMin(std::vector<Photon*> pmap, int axis) {
   float min = 0.0f;
   for (uint i = 0; i < pmap.size(); i++) {
      if (i == 0 || pmap[i]->pt[axis] < min) {
         min = pmap[i]->pt[axis];
      }
   }
   return min;
}

float KDTreeNode::findMax(std::vector<Photon*> pmap, int axis) {
   float max = 0.0f;
   for (uint i = 0; i < pmap.size(); i++) {
      if (i == 0 || pmap[i]->pt[axis] > max) {
         max = pmap[i]->pt[axis];
      }
   }
   
   /* CODY, THIS IS WRONG, YOU SHOULD KNOW THAT.
   YOU WANT TO RETURN MIN */
   return max;
}

void KDTreeNode::toSerialArray(Photon *objectArray, int *currentIndex) {
   int i;
   int thisInd = *currentIndex;
   *currentIndex += 2;
   
   /*memcpy(objectArray + (*currentIndex), this, sizeof(KDTreeNode));
   *currentIndex += 1;
   memcpy(objectArray + (*currentIndex), photon, sizeof(Photon));
   *currentIndex += 1;*/
   
   if (left) {
      leftInd = *currentIndex;
      left->toSerialArray(objectArray, currentIndex);
   }
   if (right) {
      rightInd = *currentIndex;
      right->toSerialArray(objectArray, currentIndex);
   }
   
   memcpy(objectArray + thisInd, this, sizeof(KDTreeNode));
   memcpy(objectArray + thisInd + 1, photon, sizeof(Photon));
}

bool compPhotons(Photon* p1, Photon* p2) { 
   if (p1->sortAxis == 0) {
      return p1->pt.x < p2->pt.x;
   } else if (p1->sortAxis == 1) {
      return p1->pt.y < p2->pt.y;
   } else if (p1->sortAxis == 2) {
      return p1->pt.z < p2->pt.z;
   } else {
      return false; //fuck
   }
}
