#include "hip/hip_runtime.h"
#include "collisionFuncs.h"

float checkOctTreeCollision(SceneObject *obj, glm::vec3 ray, SceneObject** object, int *shI, float *shF) {
   //start
   //F6-F8: start vector

   glm::vec3 start(shF[6], shF[7], shF[8]);
   float t, tempT;
   SceneObject* tempObj;
   t = tempT = -1.0f;
   OctTreeNode *thisObj = reinterpret_cast<OctTreeNode*>(obj);
   SceneObject **octants = thisObj->octants;
   
   if (thisObj->boundingBox.checkCollision(ray, 0.0f, shI, shF) < TOLERANCE) { 
      return -1.0f; 
   }
   
   for (int i = 0; i < 8; i++) {
      if (octants[i]) {
         if (t >= 0.001) {
            tempT = octants[i]->boundingBox.checkCollision(ray, 0.0f, shI, shF);
            if (tempT >= TOLERANCE && tempT < t) {
               tempT = octants[i]->checkCollision(octants[i], ray, &tempObj, shI, shF);
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  *object = tempObj;
               }
            }
         } else {
            t = octants[i]->checkCollision(octants[i], ray, &tempObj, shI, shF);
            *object = tempObj;
         }
      }
   }
   if (t < TOLERANCE) {
      *object = NULL;
   }
   
   return t;
}

float checkTriCollision(SceneObject *obj, glm::vec3 ray, SceneObject** object, int *shI, float *shF) {
   //start
   //F6-F8: start vector
   //
   //mid
   //F5: time
   //F6: beta
   //F7: gamma
   //F8: determinant of original A matrix

   volatile glm::vec3 start(shF[6], shF[7], shF[8]);
   glm::mat3 A;
   volatile Triangle *thisObj = reinterpret_cast<volatile Triangle*>(obj);
   *object = obj;
   
   A = glm::mat3(thisObj->a.x - thisObj->b.x, thisObj->a.x - thisObj->c.x, ray.x,
                 thisObj->a.y - thisObj->b.y, thisObj->a.y - thisObj->c.y, ray.y,
                 thisObj->a.z - thisObj->b.z, thisObj->a.z - thisObj->c.z, ray.z);
   shF[8] = glm::determinant(A);
   
   A[0][2] = thisObj->a.x - start.x; A[1][2] = thisObj->a.y - start.y; A[2][2] = thisObj->a.z - start.z;
   shF[5] = glm::determinant(A) / shF[8];
   A[0][2] = ray.x; A[1][2] = ray.y; A[2][2] = ray.z;
   if (shF[5] > TOLERANCE) {
      A[0][0] = thisObj->a.x - start.x; A[1][0] = thisObj->a.y - start.y; A[2][0] = thisObj->a.z - start.z;
      shF[7] = glm::determinant(A) / shF[8];
      A[0][0] = thisObj->a.x - thisObj->b.x; A[1][0] = thisObj->a.y - thisObj->b.y; A[2][0] = thisObj->a.z - thisObj->b.z;
      if (shF[7] >= 0.0f && shF[7] <= 1.0f) {
         A[0][1] = thisObj->a.x - start.x; A[1][1] = thisObj->a.y - start.y; A[2][1] = thisObj->a.z - start.z;
         shF[6] = glm::determinant(A) / shF[8];
         if (shF[6] >= 0.0f && shF[6] + shF[7] <= 1.0f) {
            shF[6] = start.x;
            shF[7] = start.y;
            shF[8] = start.z;
            
            return shF[5];
         }
      }
   }
   
   shF[6] = start.x;
   shF[7] = start.y;
   shF[8] = start.z;
               
   return -1.0f;
}

//See more detailed function below
float checkPlaneCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   //F5: time
   //F6-F8: start vector
   
   *object = obj;
   shF[5] = -1.0f;
   volatile Plane *thisObj = reinterpret_cast<volatile Plane*>(obj);
   volatile float dotProd = ray.x * thisObj->normal.x + ray.y * thisObj->normal.y + ray.z * thisObj->normal.z;

   volatile glm::vec3 newpt;
   newpt.x = thisObj->planePt.x - shF[6];
   newpt.y = thisObj->planePt.y - shF[7];
   newpt.z = thisObj->planePt.z - shF[8];
   
   if (dotProd != 0.0f) {
      shF[5] = (newpt.x * thisObj->normal.x + newpt.y * thisObj->normal.y + newpt.z * thisObj->normal.z) / dotProd;
   }
   
   return shF[5];
}


float checkBiTreeCollision(SceneObject *obj, glm::vec3 ray, SceneObject** object, int *shI, float *shF) {
   glm::vec4 startTransform;
   glm::vec3 start(shF[6], shF[7], shF[8]);
   float t, tLeft, tRight;
   SceneObject *lObj, *rObj;
   t = tLeft = tRight = -1.0f;
   BiTreeNode *thisObj = reinterpret_cast<BiTreeNode*>(obj);
   
   startTransform = glm::vec4(start, 1.0f);
   if (thisObj->boundingBox.checkCollision(start, ray, 0.0f) < TOLERANCE) return -1.0f;
   
   tLeft = thisObj->left->checkCollision(thisObj->left, ray, &lObj, shI, shF);
   if (thisObj->right) {
      if (tLeft >= TOLERANCE) {
         tRight = thisObj->right->boundingBox.checkCollision(start, ray, 0.0f);
         if (tRight >= TOLERANCE && tRight < tLeft) {
            tRight = thisObj->right->checkCollision(thisObj->right, ray, &rObj, shI, shF);
         } else {
            tRight = -1.0f;
         }
      } else {
         tRight = thisObj->right->checkCollision(thisObj->right, ray, &rObj, shI, shF);
      }
   }
   
   if (tLeft < TOLERANCE) {
      if (tRight >= TOLERANCE) {
         t = tRight;
         *object = rObj;
      }
   } else {
      if (tRight < TOLERANCE) {
         t = tLeft;
         *object = lObj;
      } else {
         if (tLeft < tRight) {
            t = tLeft;
            *object = lObj;
         } else {
            t = tRight;
            *object = rObj;
         }
      }
   }
   
   *object = obj;
   return t;
}

float checkBoxCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   glm::vec3 start(shF[6], shF[7], shF[8]);
   float tgmin = FLT_MIN, tgmax = FLT_MAX, t1, t2, temp, t = -1.0f;
   Box *thisObj = reinterpret_cast<Box*>(obj);
   glm::vec3 maxPt = thisObj->maxPt;
   glm::vec3 minPt = thisObj->minPt;
   
   for (int i = 0; i < 3; i++) {
      temp = start[i];
      
      if (fabs(ray[i]) < TOLERANCE) { // Ray along 2D Plane checks
         if (temp > maxPt[i] || temp < minPt[i]) return -1.0f;
      }
      
      t1 = (minPt[i] - temp) / ray[i];
      t2 = (maxPt[i] - temp) / ray[i];
      if (t2 < t1) {
         temp = t2;
         t2 = t1;
         t1 = temp;
      }
      if (t1 > tgmin) tgmin = t1;
      if (t2 < tgmax) tgmax = t2;
   }
   
   *object = obj;
   
   if (tgmin > tgmax) return -1.0f;
   if (tgmin < TOLERANCE) return tgmax;
   return tgmin;
}

float checkQuadTreeCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   //start
   //F6-F8: start vector
   
   float t, tempT;
   SceneObject *tempObj;
   t = tempT = -1.0f;
   QuadTreeNode *thisObj = reinterpret_cast<QuadTreeNode*>(obj);
   SceneObject **quadrants = thisObj->quadrants;

   if (thisObj->boundingBox.checkCollision(ray, 0.0f, shI, shF) < TOLERANCE) {
      return -1.0f;
   }
   
   for (int i = 0; i < 4; i++) {
      if (quadrants[i]) {
         if (t >= TOLERANCE) {
            tempT = quadrants[i]->boundingBox.checkCollision(ray, 0.0f, shI, shF);
            if (tempT >= TOLERANCE && tempT < t) {
               tempT = quadrants[i]->checkCollision(quadrants[i], ray, &tempObj, shI, shF);
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  *object = tempObj;
               }
            }
         } else {
            t = quadrants[i]->checkCollision(quadrants[i], ray, &tempObj, shI, shF);
            *object = tempObj;
         }
      }
   }
   
   if (t < TOLERANCE) {
      *object = NULL;
   }
   
   return t;
}

float checkSphereCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   //start
   //F6-F8: start vector
   //
   //mid
   //F5: inner root
   //F6: A
   //F7: B
   //F8: C
   
   glm::vec3 start(shF[6], shF[7], shF[8]);
   volatile float t0 = -1.0f, t1 = -1.0f;
   
   Sphere *thisObj = reinterpret_cast<Sphere*>(obj);
   volatile float radius = thisObj->radius;
   glm::vec3 position = thisObj->position;
   
   shF[6] = glm::dot(ray, ray);
   shF[7] = 2.0f * glm::dot(start - position, ray);
   shF[8] = glm::dot(start - position, start - position) - (radius * radius);
   shF[5] = (shF[7] * shF[7]) - (4.0f * shF[6] * shF[8]);
   
   if (shF[5] >= 0.0f) {
      t0 = (-shF[7] - sqrt(shF[5])) / (2.0f * shF[6]);
      t1 = (-shF[7] + sqrt(shF[5])) / (2.0f * shF[6]);
      if (t1 >= TOLERANCE && t1 < t0) {
         t0 = t1;
      }
   }
   
   shF[6] = start.x;
   shF[7] = start.y;
   shF[8] = start.z;
   
   *object = obj;
   return t0;
}

//float checkGWCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
float checkGWCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   printf("GW Collision\n");
   *object = obj;
   return -1.0f;
}

//float checkConeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
float checkConeCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   printf("Cone Collision\n");
   *object = obj;
   return -1.0f;
}
