#include "hip/hip_runtime.h"
#include "collisionFuncs.h"

float checkOctTreeCollision(SceneObject *obj, glm::vec3 ray, SceneObject** object, int *shI, float *shF) {
   //start
   //F6-F8: start vector

   glm::vec3 start(shF[6], shF[7], shF[8]);
   float t, tempT;
   SceneObject* tempObj;
   t = tempT = -1.0f;
   OctTreeNode *thisObj = reinterpret_cast<OctTreeNode*>(obj);
   SceneObject **octants = thisObj->octants;
   
   if (thisObj->boundingBox.checkCollision(ray, 0.0f, shI, shF) < TOLERANCE) { 
      return -1.0f; 
   }
   
   for (int i = 0; i < 8; i++) {
      if (octants[i]) {
         if (t >= 0.001) {
            tempT = octants[i]->boundingBox.checkCollision(ray, 0.0f, shI, shF);
            if (tempT >= TOLERANCE && tempT < t) {
               tempT = octants[i]->checkCollision(octants[i], ray, &tempObj, shI, shF);
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  *object = tempObj;
               }
            }
         } else {
            t = octants[i]->checkCollision(octants[i], ray, &tempObj, shI, shF);
            *object = tempObj;
         }
      }
   }
   if (t < TOLERANCE) {
      *object = NULL;
   }
   
   return t;
}

/*float checkOctTreeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
   //printf("%f %f %f\n", start.x, start.y, start.z);
   //printf("%f %f %f\n", ray.x, ray.y, ray.z);
   //printf("otree collision\n");
   //printf("PARAM MEM: %p\n", obj);
   //return -1.0f;
   //printf("pls\n");
   glm::vec3 start(shF[6], shF[7], shF[8]);
   glm::vec4 startTransform;
   float t, tempT;
   //int i;
   SceneObject* tempObj;
   t = tempT = -1.0f;
   //obj = (SceneObject *)0xb0b8e4628;
   OctTreeNode *thisObj = reinterpret_cast<OctTreeNode*>(obj);
   //printf("Wanted to make sure\n");
   SceneObject **octants = thisObj->octants;
   int *indeces = thisObj->indeces;
   //if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   
   //printf("CURRENT MEM: %p\n", thisObj);
   //int pf = printf("Type? %d\n", thisObj->type);
   //printf("%d\n", pf);
   //printf("blah? %d\n", thisObj->blahblah);
   //printf("amb? %f\n", thisObj->ambient);
   //printf("is it here?\n");
   
   if (thisObj->boundingBox.checkCollision(start, ray, 0.0f) < TOLERANCE) { 
      //printf("%f %f %f\n", thisObj->boundingBox.minPt.x, thisObj->boundingBox.minPt.y, thisObj->boundingBox.minPt.z);
      //printf("%f %f %f\n", thisObj->boundingBox.maxPt.x, thisObj->boundingBox.maxPt.y, thisObj->boundingBox.maxPt.z);
      //printf("bb not hit\n");
      return -1.0f; 
   }
   //printf("nope\n");
   //if (octants[0] == NULL) {
   //   printf("roll up in the club\n");
   //} else {
   //   printf("yoyoyo\n");
   //}
   
   if (octants[0]) {
      //printf("weird\n");
      if (octants[0]->transformed) {
         //printf("where is it\n");
         //t = octants[0]->checkCollision(octants[0], glm::vec3(octants[0]->transform * startTransform), glm::mat3(thisObj->octants[0]->transform) * ray, time, &tempObj, shI, shF);
      } else {
         
         //printf("ok...\n");
         
         //pf = printf("Type octant? %d\n", octants[0]->type);
         //printf("%d\n", pf);
         //printf("BLAH octant? %d\n", octants[0]->blahblah);
         //printf("AMBi octant? %f\n", octants[0]->ambient);
         
         //octants[0]->type = 8;
         //octants[0]->blahblah = 1;
         //octants[0]->ambient = 4.0f;

         //printf("OCTANT MEM: %p\n", octants[0]);
         //printf("MORE OCTANT MEM: %p\n", thisObj + 1);
         //printf("THIS COLL MEM: %p\n", thisObj->checkCollision);
         //printf("OCTANT COLL MEM: %p\n", octants[0]->checkCollision);
         //printf("OCT COLL MEM: %p\n", &(checkOctTreeCollision));
         //printf("TRI COLL MEM: %p\n", &(checkTriCollision));
         
         //for (int i = 0; i < 8; i++) {
         //   printf("INDEX %d: %d\n", i, indeces[i]);
         //}
         
         //SETTING COLLISION FUNC PTR TO EQUIVALENT FUNCTION
         
         //octants[0]->checkCollision = &(checkOctTreeCollision2);
         
         //printf("OCTANT COLL MEM: %p\n", octants[0]->checkCollision);
         //printf("RANDOM\n");
         //printf("FILLER\n");
         //printf("STUFF\n");
         //printf("TO\n");
         //printf("SEE\n");
         //printf("IF\n");
         //printf("ITS\n");
         //printf("BAD\n");
         //printf("PARAM MEM AGAIN: %p\n", obj);
         
         
         // CALLING SECOND HERE
         // AFTER THIS IT WILL CRASH
         
         //octants[0] = (SceneObject*)0xb085e09a8;
         //printf("NEW TYPE: %d\n", octants[0]->type);
         t = octants[0]->checkCollision(octants[0], ray, &tempObj, shI, shF);
      }
   }
   *object = tempObj;
   //printf("t: %f\n", t);
   //printf("Type? %d\n", thisObj->type);
   //if (octants[1]) {
   //   printf("anything, really\n");
   //} else {
   //   printf("pls\n");
   //}
   //printf("looking for something\n");
   
   int i = 0;
   
   // SAME AS ABOVE, BUT FOR THE OTHER 7 OCTANTS
   
   for (i = 1; i < 8; i++) {
      //printf("please\n");
      if (octants[i]) {
      //printf("yo\n");
         if (t >= 0.001) {
            //printf("hi\n");
            tempT = octants[i]->boundingBox.checkCollision(start, ray, 0.0f);
            if (tempT >= TOLERANCE && tempT < t) {
               if (octants[i]->transformed) {
                  //tempT = octants[i]->checkCollision(octants[i], glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj, shI, shF);
               } else {
                  //printf("yo dawg\n");
                  if (octants[i]->type != 8) {
                     //printf("triangle starting\n");
                  }
                  
                  
                  tempT = octants[i]->checkCollision(octants[i], ray, &tempObj, shI, shF);
                  //printf("TTEMP : %f\n", tempT);
                  
                  //printf("k\n");
                  //if (octants[i]->type != 8) {
                  //   printf("triangle ending\n");
                  //}
               }
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  //printf("T : %f\n", t);
                  *object = tempObj;
               }
            }
         } else {
            //octants[i] = (SceneObject*)0xb085e09a8;
            //t = octants[i]->checkCollision(octants[i], glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj, shI, shF);
            //printf("T : %f\n", t);
            //t = octants[i]->checkCollision(octants[i], start, ray, time, &tempObj);
            *object = tempObj;
         }
      } //else break();
      //printf("sup fam\n");
   }
   //printf("whats happening\n");
   if (t < TOLERANCE) {
      *object = NULL;
   }
   //printf("END T : %f\n", t);
   return t;
}*/

//See more detailed function below
float checkTriCollision(SceneObject *obj, glm::vec3 ray, SceneObject** object, int *shI, float *shF) {
   //start
   //F6-F8: start vector
   //
   //mid
   //F5: time
   //F6: beta
   //F7: gamma
   //F8: determinant of original A matrix

   volatile glm::vec3 start(shF[6], shF[7], shF[8]);
   glm::mat3 A;
   volatile Triangle *thisObj = reinterpret_cast<volatile Triangle*>(obj);
   *object = obj;
   
   A = glm::mat3(thisObj->a.x - thisObj->b.x, thisObj->a.x - thisObj->c.x, ray.x,
                 thisObj->a.y - thisObj->b.y, thisObj->a.y - thisObj->c.y, ray.y,
                 thisObj->a.z - thisObj->b.z, thisObj->a.z - thisObj->c.z, ray.z);
   shF[8] = glm::determinant(A);
   
   A[0][2] = thisObj->a.x - start.x; A[1][2] = thisObj->a.y - start.y; A[2][2] = thisObj->a.z - start.z;
   shF[5] = glm::determinant(A) / shF[8];
   A[0][2] = ray.x; A[1][2] = ray.y; A[2][2] = ray.z;
   if (shF[5] > TOLERANCE) {
      A[0][0] = thisObj->a.x - start.x; A[1][0] = thisObj->a.y - start.y; A[2][0] = thisObj->a.z - start.z;
      shF[7] = glm::determinant(A) / shF[8];
      A[0][0] = thisObj->a.x - thisObj->b.x; A[1][0] = thisObj->a.y - thisObj->b.y; A[2][0] = thisObj->a.z - thisObj->b.z;
      if (shF[7] >= 0.0f && shF[7] <= 1.0f) {
         A[0][1] = thisObj->a.x - start.x; A[1][1] = thisObj->a.y - start.y; A[2][1] = thisObj->a.z - start.z;
         shF[6] = glm::determinant(A) / shF[8];
         if (shF[6] >= 0.0f && shF[6] + shF[7] <= 1.0f) {
            shF[6] = start.x;
            shF[7] = start.y;
            shF[8] = start.z;
            
            return shF[5];
         }
      }
   }
   
   shF[6] = start.x;
   shF[7] = start.y;
   shF[8] = start.z;
               
   return -1.0f;
}

/*float checkTriCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
   //printf("Triangle Collision\n");
   glm::mat3 A, Ai;
   double detA, t, beta, gamma;
   Triangle *thisObj = reinterpret_cast<Triangle*>(obj);
   *object = obj;
   glm::vec3 a = thisObj->a;
   glm::vec3 b = thisObj->b;
   glm::vec3 c = thisObj->c;
   
   A = glm::mat3(a.x - b.x, a.x - c.x, ray.x,
                 a.y - b.y, a.y - c.y, ray.y,
                 a.z - b.z, a.z - c.z, ray.z);

   detA = glm::determinant(A);
   
   Ai = A;
   Ai[0][2] = thisObj->a.x - start.x; Ai[1][2] = thisObj->a.y - start.y; Ai[2][2] = thisObj->a.z - start.z;
   t = glm::determinant(Ai) / detA;
   if (t > TOLERANCE) {
      Ai = A;
      Ai[0][0] = thisObj->a.x - start.x; Ai[1][0] = thisObj->a.y - start.y; Ai[2][0] = thisObj->a.z - start.z;
      gamma = glm::determinant(Ai) / detA;
      if (gamma >= 0.0f && gamma <= 1.0f) {
         Ai = A;
         Ai[0][1] = thisObj->a.x - start.x; Ai[1][1] = thisObj->a.y - start.y; Ai[2][1] = thisObj->a.z - start.z;
         beta = glm::determinant(Ai) / detA;
         if (beta >= 0.0f && beta + gamma <= 1.0f) {
            *object = obj;            
            return t;
         }
      }
   }
   
   return -1.0f;
}*/

//See more detailed function below
float checkPlaneCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   //F5: time
   //F6-F8: start vector
   
   *object = obj;
   shF[5] = -1.0f;
   volatile Plane *thisObj = reinterpret_cast<volatile Plane*>(obj);
   volatile float dotProd = ray.x * thisObj->normal.x + ray.y * thisObj->normal.y + ray.z * thisObj->normal.z;

   volatile glm::vec3 newpt;
   newpt.x = thisObj->planePt.x - shF[6];
   newpt.y = thisObj->planePt.y - shF[7];
   newpt.z = thisObj->planePt.z - shF[8];
   
   if (dotProd != 0.0f) {
      shF[5] = (newpt.x * thisObj->normal.x + newpt.y * thisObj->normal.y + newpt.z * thisObj->normal.z) / dotProd;
   }
   
   return shF[5];
}

/*float checkPlaneCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
   float t = -1.0f, dotProd;
   Plane *thisObj = reinterpret_cast<Plane*>(obj);

   dotProd = glm::dot(ray, thisObj->normal);
   
   if (dotProd != 0.0f) {
      t = glm::dot(thisObj->planePt - start, thisObj->normal) / glm::dot(ray, thisObj->normal);
   }
   
   *object = obj;
   return t;
}*/

//float checkBiTreeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
float checkBiTreeCollision(SceneObject *obj, glm::vec3 ray, SceneObject** object, int *shI, float *shF) {
   glm::vec4 startTransform;
   glm::vec3 start(shF[6], shF[7], shF[8]);
   float t, tLeft, tRight;
   SceneObject *lObj, *rObj;
   t = tLeft = tRight = -1.0f;
   BiTreeNode *thisObj = reinterpret_cast<BiTreeNode*>(obj);
   
  // if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   if (thisObj->boundingBox.checkCollision(start, ray, 0.0f) < TOLERANCE) return -1.0f;
   //if (left) {
   if (thisObj->left->transformed) {
      //tLeft = thisObj->left->checkCollision(thisObj->left, glm::vec3(thisObj->left->transform * startTransform), glm::mat3(thisObj->left->transform) * ray, time, &lObj, shI, shF);
   } else {
      tLeft = thisObj->left->checkCollision(thisObj->left, ray, &lObj, shI, shF);
   }
   if (thisObj->right) {
      if (tLeft >= TOLERANCE) {
         tRight = thisObj->right->boundingBox.checkCollision(start, ray, 0.0f);
         if (tRight >= TOLERANCE && tRight < tLeft) {
            if (thisObj->right->transformed) {
               //tRight = thisObj->right->checkCollision(thisObj->right, glm::vec3(thisObj->right->transform * startTransform), glm::mat3(thisObj->right->transform) * ray, time, &rObj, shI, shF);
            } else {
               tRight = thisObj->right->checkCollision(thisObj->right, ray, &rObj, shI, shF);
            }
         } else {
            tRight = -1.0f;
         }
      } else {
         //tRight = thisObj->right->checkCollision(thisObj->right, glm::vec3(thisObj->right->transform * startTransform), glm::mat3(thisObj->right->transform) * ray, time, &rObj, shI, shF);
         tRight = thisObj->right->checkCollision(thisObj->right, ray, &rObj, shI, shF);
      }
   }
   //}
   
   if (tLeft < TOLERANCE) {
      if (tRight >= TOLERANCE) {
         t = tRight;
         *object = rObj;
      }
   } else {
      if (tRight < TOLERANCE) {
         t = tLeft;
         *object = lObj;
      } else {
         if (tLeft < tRight) {
            t = tLeft;
            *object = lObj;
         } else {
            t = tRight;
            *object = rObj;
         }
      }
   }
   
   *object = obj;
   return t;
}

//float checkBoxCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
float checkBoxCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   //std::cout << "Box Collision" << std::endl;
   glm::vec3 start(shF[6], shF[7], shF[8]);
   float tgmin = FLT_MIN, tgmax = FLT_MAX, t1, t2, temp, t = -1.0f;
   Box *thisObj = reinterpret_cast<Box*>(obj);
   glm::vec3 maxPt = thisObj->maxPt;
   glm::vec3 minPt = thisObj->minPt;
   
   for (int i = 0; i < 3; i++) {
      temp = start[i];
      
      if (fabs(ray[i]) < TOLERANCE) { // Ray along 2D Plane checks
         if (temp > maxPt[i] || temp < minPt[i]) return -1.0f;
      }
      
      t1 = (minPt[i] - temp) / ray[i];
      t2 = (maxPt[i] - temp) / ray[i];
      if (t2 < t1) {
         temp = t2;
         t2 = t1;
         t1 = temp;
      }
      if (t1 > tgmin) tgmin = t1;
      if (t2 < tgmax) tgmax = t2;
   }
   
   /*if (start.x() >= minPt.x() && start.x() <= maxPt.x() &&
       start.y() >= minPt.y() && start.y() <= maxPt.y() && 
       start.z() >= minPt.z() && start.z() <= maxPt.z()) {
       t = 10.0f;
       
       std::cout << "Inside!!!" << std::endl;
   }*/
   //if (tgmin < TOLERANCE) return new Collision(tgmin, this);
   //if (tgmin > tgmax || tgmax < 0.001f) return new Collision(t, this);
   
   *object = obj;
   
   if (tgmin > tgmax) return -1.0f;
   if (tgmin < TOLERANCE) return tgmax;
   return tgmin;
}

//float checkQuadTreeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
float checkQuadTreeCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   //start
   //F6-F8: start vector
   
   float t, tempT;
   SceneObject *tempObj;
   t = tempT = -1.0f;
   QuadTreeNode *thisObj = reinterpret_cast<QuadTreeNode*>(obj);
   SceneObject **quadrants = thisObj->quadrants;

   if (thisObj->boundingBox.checkCollision(ray, 0.0f, shI, shF) < TOLERANCE) {
      return -1.0f;
   }
   
   for (int i = 0; i < 4; i++) {
      if (quadrants[i]) {
         if (t >= TOLERANCE) {
            tempT = quadrants[i]->boundingBox.checkCollision(ray, 0.0f, shI, shF);
            if (tempT >= TOLERANCE && tempT < t) {
               tempT = quadrants[i]->checkCollision(quadrants[i], ray, &tempObj, shI, shF);
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  *object = tempObj;
               }
            }
         } else {
            t = quadrants[i]->checkCollision(quadrants[i], ray, &tempObj, shI, shF);
            *object = tempObj;
         }
      }
   }
   
   if (t < TOLERANCE) {
      *object = NULL;
   }
   
   return t;
}

//See original below
float checkSphereCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   //start
   //F6-F8: start vector
   //
   //mid
   //F5: inner root
   //F6: A
   //F7: B
   //F8: C
   
   glm::vec3 start(shF[6], shF[7], shF[8]);
   volatile float t0 = -1.0f, t1 = -1.0f;
   
   Sphere *thisObj = reinterpret_cast<Sphere*>(obj);
   volatile float radius = thisObj->radius;
   glm::vec3 position = thisObj->position;
   
   shF[6] = glm::dot(ray, ray);
   shF[7] = 2.0f * glm::dot(start - position, ray);
   shF[8] = glm::dot(start - position, start - position) - (radius * radius);
   shF[5] = (shF[7] * shF[7]) - (4.0f * shF[6] * shF[8]);
   
   if (shF[5] >= 0.0f) {
      t0 = (-shF[7] - sqrt(shF[5])) / (2.0f * shF[6]);
      t1 = (-shF[7] + sqrt(shF[5])) / (2.0f * shF[6]);
      if (t1 >= TOLERANCE && t1 < t0) {
         t0 = t1;
      }
   }
   
   shF[6] = start.x;
   shF[7] = start.y;
   shF[8] = start.z;
   
   *object = obj;
   return t0;
}

/*float checkSphereCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
   volatile float t, t0 = -1.0f, t1 = -1.0f, A, B, C;
   
   Sphere *thisObj = reinterpret_cast<Sphere*>(obj);
   volatile float radius = thisObj->radius;
   glm::vec3 position = thisObj->position;
   float A, B, C;
   
   A = glm::dot(ray, ray);
   B = 2.0f * glm::dot(start - position, ray);
   C = glm::dot(start - position, start - position) - (radius * radius);
   innerRoot = (B * B) - (4.0f * A * C);
   
   if (innerRoot >= 0.0f) {
      t0 = (-B - sqrt(innerRoot)) / (2.0f * A);
      t1 = (-B + sqrt(innerRoot)) / (2.0f * A);
      if (t0 >= TOLERANCE && t0 < t1) {
         t = t0;
      } else if (t1 >= TOLERANCE) {
         t = t1;
      }
   }
   
   *object = obj;
   return t0;
}*/

//float checkGWCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
float checkGWCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   printf("GW Collision\n");
   *object = obj;
   return -1.0f;
}

//float checkConeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
float checkConeCollision(SceneObject *obj, glm::vec3 ray, SceneObject **object, int *shI, float *shF) {
   printf("Cone Collision\n");
   *object = obj;
   return -1.0f;
}
