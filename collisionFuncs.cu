#include "hip/hip_runtime.h"
#include "collisionFuncs.h"

float checkOctTreeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
/*printf("%f %f %f\n", start.x, start.y, start.z);
printf("%f %f %f\n", ray.x, ray.y, ray.z);
printf("otree collision\n");*/
//printf("PARAM MEM: %p\n", obj);
//return -1.0f;
   //printf("pls\n");
   glm::vec4 startTransform;
   float t, tempT;
   //int i;
   SceneObject* tempObj;
   t = tempT = -1.0f;
   //obj = (SceneObject *)0xb0b8e4628;
   OctTreeNode *thisObj = reinterpret_cast<OctTreeNode*>(obj);
   //printf("Wanted to make sure\n");
   SceneObject **octants = thisObj->octants;
   int *indeces = thisObj->indeces;
   //if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   
   
   /*printf("CURRENT MEM: %p\n", thisObj);
   int pf = printf("Type? %d\n", thisObj->type);
   printf("%d\n", pf);
   printf("blah? %d\n", thisObj->blahblah);
   printf("amb? %f\n", thisObj->ambient);
   printf("is it here?\n");*/
   
   
   if (thisObj->boundingBox.checkCollision(start, ray, time) < TOLERANCE) { 
      /*printf("%f %f %f\n", thisObj->boundingBox.minPt.x, thisObj->boundingBox.minPt.y, thisObj->boundingBox.minPt.z);
      printf("%f %f %f\n", thisObj->boundingBox.maxPt.x, thisObj->boundingBox.maxPt.y, thisObj->boundingBox.maxPt.z);
      printf("bb not hit\n"); */
      return -1.0f; 
   }
   /*printf("nope\n");
   if (octants[0] == NULL) {
      printf("roll up in the club\n");
   } else {
      printf("yoyoyo\n");
   }*/
   
   
   if (octants[0]) {
      //printf("weird\n");
      if (octants[0]->transformed) {
         //printf("where is it\n");
         t = octants[0]->checkCollision(octants[0], glm::vec3(octants[0]->transform * startTransform), glm::mat3(thisObj->octants[0]->transform) * ray, time, &tempObj);
      } else {
         
         /*printf("ok...\n");
         
         pf = printf("Type octant? %d\n", octants[0]->type);
         printf("%d\n", pf);
         printf("BLAH octant? %d\n", octants[0]->blahblah);
         printf("AMBi octant? %f\n", octants[0]->ambient);
         
         
         octants[0]->type = 8;
         octants[0]->blahblah = 1;
         octants[0]->ambient = 4.0f;

         
         printf("OCTANT MEM: %p\n", octants[0]);
         printf("MORE OCTANT MEM: %p\n", thisObj + 1);
         printf("THIS COLL MEM: %p\n", thisObj->checkCollision);
         printf("OCTANT COLL MEM: %p\n", octants[0]->checkCollision);
         printf("OCT COLL MEM: %p\n", &(checkOctTreeCollision));
         printf("TRI COLL MEM: %p\n", &(checkTriCollision));
         
         
         for (int i = 0; i < 8; i++) {
            printf("INDEX %d: %d\n", i, indeces[i]);
         }
         
         
         //SETTING COLLISION FUNC PTR TO EQUIVALENT FUNCTION
         
         //octants[0]->checkCollision = &(checkOctTreeCollision2);
         
         
         printf("OCTANT COLL MEM: %p\n", octants[0]->checkCollision);
         printf("RANDOM\n");
         printf("FILLER\n");
         printf("STUFF\n");
         printf("TO\n");
         printf("SEE\n");
         printf("IF\n");
         printf("ITS\n");
         printf("BAD\n");
         printf("PARAM MEM AGAIN: %p\n", obj);
         
         
         // CALLING SECOND HERE
         // AFTER THIS IT WILL CRASH
         
         //octants[0] = (SceneObject*)0xb085e09a8;
         printf("NEW TYPE: %d\n", octants[0]->type);*/
         t = octants[0]->checkCollision(octants[0], start, ray, time, &tempObj);
      }
   }
   *object = tempObj;
   /*printf("t: %f\n", t);
   printf("Type? %d\n", thisObj->type);
   if (octants[1]) {
      printf("anything, really\n");
   } else {
      printf("pls\n");
   }
   printf("looking for something\n");*/
   
   int i = 0;
   
   // SAME AS ABOVE, BUT FOR THE OTHER 7 OCTANTS
   
   for (i = 1; i < 8; i++) {
      //printf("please\n");
      if (octants[i]) {
      //printf("yo\n");
         if (t >= 0.001) {
            //printf("hi\n");
            tempT = octants[i]->boundingBox.checkCollision(start, ray, time);
            if (tempT >= TOLERANCE && tempT < t) {
               if (octants[i]->transformed) {
                  tempT = octants[i]->checkCollision(octants[i], glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj);
               } else {
                  //printf("yo dawg\n");
                  if (octants[i]->type != 8) {
                     //printf("triangle starting\n");
                  }
                  
                  
                  tempT = octants[i]->checkCollision(octants[i], start, ray, time, &tempObj);
                  //printf("TTEMP : %f\n", tempT);
                  
                  /*printf("k\n");
                  if (octants[i]->type != 8) {
                     printf("triangle ending\n");
                  }*/
               }
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  //printf("T : %f\n", t);
                  *object = tempObj;
               }
            }
         } else {
            //octants[i] = (SceneObject*)0xb085e09a8;
            t = octants[i]->checkCollision(octants[i], glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj);
            //printf("T : %f\n", t);
            //t = octants[i]->checkCollision(octants[i], start, ray, time, &tempObj);
            *object = tempObj;
         }
      } //else break();
      //printf("sup fam\n");
   }
   //printf("whats happening\n");
   if (t < TOLERANCE) {
      *object = NULL;
   }
   //printf("END T : %f\n", t);
   return t;
}

float checkOctTreeCollision2(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
//printf("%f %f %f\n", start.x, start.y, start.z);
//printf("%f %f %f\n", ray.x, ray.y, ray.z);
printf("otree collision 2\n");
printf("PARAM MEM: %p\n", obj);
//return -1.0f;
   //printf("pls\n");
   glm::vec4 startTransform;
   float t, tempT;
   //int i;
   SceneObject* tempObj;
   t = tempT = -1.0f;
   OctTreeNode *thisObj = reinterpret_cast<OctTreeNode*>(obj);
   //printf("Wanted to make sure\n");
   SceneObject **octants = thisObj->octants;
   int *indeces = thisObj->indeces;
   //if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   //if (this == NULL) printf("Seriously what the fuck\n");
   printf("CURRENT MEM: %p\n", thisObj);
   int pf = printf("Type? %d\n", thisObj->type);
   printf("%d\n", pf);
   printf("blah? %d\n", thisObj->blahblah);
   printf("amb? %f\n", thisObj->ambient);
   printf("is it here?\n");
   if (thisObj->boundingBox.checkCollision(start, ray, time) < TOLERANCE) { printf("bb not hit\n"); return -1.0f; }
   printf("nope\n");
   if (octants[0] == NULL) {
      printf("roll up in the club\n");
   } else {
      printf("yoyoyo\n");//printf("like i got a fat...\n");
   }
   if (octants[0]) {
   if (octants[0]->transformed) {
      t = octants[0]->checkCollision(octants[0], glm::vec3(octants[0]->transform * startTransform), glm::mat3(thisObj->octants[0]->transform) * ray, time, &tempObj);
   } else {
      pf = printf("Type octant? %d\n", octants[0]->type);
      printf("%d\n", pf);
      printf("BLAH octant? %d\n", octants[0]->blahblah);
      printf("AMBi octant? %f\n", octants[0]->ambient);
      
      
      octants[0]->type = 8;
      octants[0]->blahblah = 1;
      octants[0]->ambient = 4.0f;

      printf("OCTANT MEM: %p\n", octants[0]);
      printf("MORE OCTANT MEM: %p\n", thisObj + 1);
      printf("THIS COLL MEM: %p\n", thisObj->checkCollision);
      printf("OCTANT COLL MEM: %p\n", octants[0]->checkCollision);
      printf("OCT COLL MEM: %p\n", &(checkOctTreeCollision));
      printf("TRI COLL MEM: %p\n", &(checkTriCollision));
      for (int i = 0; i < 8; i++) {
         printf("INDEX %d: %d\n", i, indeces[i]);
      }
      
      t = octants[0]->checkCollision(octants[0], start, ray, time, &tempObj);
   }
   }
   *object = tempObj;
   printf("t: %f\n", t);
   printf("Type? %d\n", thisObj->type);
   if (octants[1]) {
      printf("anything, really\n");
   } else {
      printf("pls\n");
   }
   printf("looking for something\n");
   
   int i = 0;
   for (i = 1; i < 8; i++) {
      printf("please\n");
      if (octants[i]) {
      printf("yo\n");
         if (t >= 0.001) {
            printf("hi\n");
            tempT = octants[i]->boundingBox.checkCollision(start, ray, time);
            if (tempT >= TOLERANCE && tempT < t) {
               if (octants[i]->transformed) {
                  tempT = octants[i]->checkCollision(octants[i], glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj);
               } else {
                  printf("yo dawg\n");
                  if (octants[i]->type != 8) {
                     printf("triangle starting\n");
                  }
                  tempT = octants[i]->checkCollision(octants[i], start, ray, time, &tempObj);
                  printf("k\n");
                  if (octants[i]->type != 8) {
                     printf("triangle ending\n");
                  }
               }
               if (tempT >= TOLERANCE && tempT < t) {
                  t = tempT;
                  *object = tempObj;
               }
            }
         } else {
            t = octants[i]->checkCollision(octants[i], glm::vec3(octants[i]->transform * startTransform), glm::mat3(octants[i]->transform) * ray, time, &tempObj);
            *object = tempObj;
         }
      } //else break();
      printf("sup fam\n");
   }
   printf("whats happening\n");
   if (t < TOLERANCE) {
      *object = NULL;
   }
   
   return t;
}

float checkTriCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
   //printf("Triangle Collision\n");
   glm::mat3 A, Ai;
   double detA, t, beta, gamma;
   Triangle *thisObj = reinterpret_cast<Triangle*>(obj);
   
   glm::vec3 a = thisObj->a;
   glm::vec3 b = thisObj->b;
   glm::vec3 c = thisObj->c;
   
   A = glm::mat3(a.x - b.x, a.x - c.x, ray.x,
                 a.y - b.y, a.y - c.y, ray.y,
                 a.z - b.z, a.z - c.z, ray.z);
   detA = glm::determinant(A);
   //std::cout << "making sure lol" << std::endl;
   //std::cout << detA << std::endl;
   //if (std::fabs(detA) > 0.0f) {
      //std::cout << "hi?" << std::endl;
      Ai = A;
      Ai[0][2] = a.x - start.x; Ai[1][2] = a.y - start.y; Ai[2][2] = a.z - start.z;
      t = glm::determinant(Ai) / detA;
      if (t > TOLERANCE) {
         //std::cout << "hiya" << std::endl;
         Ai = A;
         Ai[0][0] = a.x - start.x; Ai[1][0] = a.y - start.y; Ai[2][0] = a.z - start.z;
         gamma = glm::determinant(Ai) / detA;
         if (gamma >= 0.0f && gamma <= 1.0f) {
            //std::cout << "hello" << std::endl;
            Ai = A;
            Ai[0][1] = a.x - start.x; Ai[1][1] = a.y - start.y; Ai[2][1] = a.z - start.z;
            beta = glm::determinant(Ai) / detA;
            //std::cout << "Beta: " << beta << " Gamma: " << gamma << std::endl;
            if (beta >= 0.0f && beta + gamma <= 1.0f) {
               //std::cout << t << std::endl;
               //printf("obj type: %d\n", obj->type);
               *object = obj;
               return t;
            }
         }
      }
   //}
   //if (time < 3.5) obj->checkCollision(obj, start, ray, time + 1, object);
   return -1.0f;
}

float checkPlaneCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
   float t = -1.0f;
   Plane *thisObj = reinterpret_cast<Plane*>(obj);
   //glm::vec3 origin = glm::vec3(0.0f, 0.0f, 0.0f);
   
   if (glm::dot(ray, thisObj->normal) != 0.0f) {
      t = glm::dot(thisObj->planePt - start, thisObj->normal) / glm::dot(ray, thisObj->normal);
   }
   
   *object = obj;
   return t;
}

float checkBiTreeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject** object) {
   glm::vec4 startTransform;
   float t, tLeft, tRight;
   SceneObject *lObj, *rObj;
   t = tLeft = tRight = -1.0f;
   BiTreeNode *thisObj = reinterpret_cast<BiTreeNode*>(obj);
   
  // if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   if (thisObj->boundingBox.checkCollision(start, ray, time) < TOLERANCE) return -1.0f;
   //if (left) {
   if (thisObj->left->transformed) {
      tLeft = thisObj->left->checkCollision(thisObj->left, glm::vec3(thisObj->left->transform * startTransform), glm::mat3(thisObj->left->transform) * ray, time, &lObj);
   } else {
      tLeft = thisObj->left->checkCollision(thisObj->left, start, ray, time, &lObj);
   }
   if (thisObj->right) {
      if (tLeft >= TOLERANCE) {
         tRight = thisObj->right->boundingBox.checkCollision(start, ray, time);
         if (tRight >= TOLERANCE && tRight < tLeft) {
            if (thisObj->right->transformed) {
               tRight = thisObj->right->checkCollision(thisObj->right, glm::vec3(thisObj->right->transform * startTransform), glm::mat3(thisObj->right->transform) * ray, time, &rObj);
            } else {
               tRight = thisObj->right->checkCollision(thisObj->right, start, ray, time, &rObj);
            }
         } else {
            tRight = -1.0f;
         }
      } else {
         tRight = thisObj->right->checkCollision(thisObj->right, glm::vec3(thisObj->right->transform * startTransform), glm::mat3(thisObj->right->transform) * ray, time, &rObj);
      }
   }
   //}
   
   if (tLeft < TOLERANCE) {
      if (tRight >= TOLERANCE) {
         t = tRight;
         *object = rObj;
      }
   } else {
      if (tRight < TOLERANCE) {
         t = tLeft;
         *object = lObj;
      } else {
         if (tLeft < tRight) {
            t = tLeft;
            *object = lObj;
         } else {
            t = tRight;
            *object = rObj;
         }
      }
   }
   
   *object = obj;
   return t;
}

float checkBoxCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
   //std::cout << "Box Collision" << std::endl;
   float tgmin = FLT_MIN, tgmax = FLT_MAX, t1, t2, temp, t = -1.0f;
   Box *thisObj = reinterpret_cast<Box*>(obj);
   glm::vec3 maxPt = thisObj->maxPt;
   glm::vec3 minPt = thisObj->minPt;
   
   for (int i = 0; i < 3; i++) {
      temp = start[i];
      
      if (fabs(ray[i]) < TOLERANCE) { // Ray along 2D Plane checks
         if (temp > maxPt[i] || temp < minPt[i]) return -1.0f;
      }
      
      t1 = (minPt[i] - temp) / ray[i];
      t2 = (maxPt[i] - temp) / ray[i];
      if (t2 < t1) {
         temp = t2;
         t2 = t1;
         t1 = temp;
      }
      if (t1 > tgmin) tgmin = t1;
      if (t2 < tgmax) tgmax = t2;
   }
   
   /*if (start.x() >= minPt.x() && start.x() <= maxPt.x() &&
       start.y() >= minPt.y() && start.y() <= maxPt.y() && 
       start.z() >= minPt.z() && start.z() <= maxPt.z()) {
       t = 10.0f;
       
       std::cout << "Inside!!!" << std::endl;
   }*/
   //if (tgmin < TOLERANCE) return new Collision(tgmin, this);
   //if (tgmin > tgmax || tgmax < 0.001f) return new Collision(t, this);
   
   *object = obj;
   
   if (tgmin > tgmax) return -1.0f;
   if (tgmin < TOLERANCE) return tgmax;
   return tgmin;
}

float checkQuadTreeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
//printf("qtree COLLISION\n");
   glm::vec4 startTransform;
   float t, tempT;
   SceneObject *obj1, *obj2, *obj3, *obj4;
   t = tempT = -1.0f;
   QuadTreeNode *thisObj = reinterpret_cast<QuadTreeNode*>(obj);
   SceneObject *q1 = thisObj->q1;
   SceneObject *q2 = thisObj->q2;
   SceneObject *q3 = thisObj->q3;
   SceneObject *q4 = thisObj->q4;
  // if (boundingBox == NULL) std::cout << "it's null!" << std::endl;
   startTransform = glm::vec4(start, 1.0f);
   if (thisObj->boundingBox.checkCollision(start, ray, time) < TOLERANCE) return -1.0f;
   

   if (q1->transformed) {
      t = q1->checkCollision(q1, glm::vec3(q1->transform * startTransform), glm::mat3(q1->transform) * ray, time, &obj1);
   } else {
      t = q1->checkCollision(q1, start, ray, time, &obj1);
   }
   *object = obj1;
//printf("1\n");
   if (q2) {
      if (t >= TOLERANCE) {
         tempT = q2->boundingBox.checkCollision(start, ray, time);
         if (tempT >= TOLERANCE && tempT < t) {
            if (q2->transformed) {
               tempT = q2->checkCollision(q2, glm::vec3(q2->transform * startTransform), glm::mat3(q2->transform) * ray, time, &obj2);
            } else {
               tempT = q2->checkCollision(q2, start, ray, time, &obj2);
            }
            if (tempT >= TOLERANCE && tempT < t) {
               t = tempT;
               *object = obj2;
            }
         }
      } else {
         t = q2->checkCollision(q2, glm::vec3(q2->transform * startTransform), glm::mat3(q2->transform) * ray, time, &obj2);
         *object = obj2;
      }
   }
   //printf("2\n");
   if (q3) {
      if (t >= TOLERANCE) {
         tempT = q3->boundingBox.checkCollision(start, ray, time);
         if (tempT >= TOLERANCE && tempT < t) {
            if (q3->transformed) {
               tempT = q3->checkCollision(q3, glm::vec3(q3->transform * startTransform), glm::mat3(q3->transform) * ray, time, &obj3);
            } else {
               tempT = q3->checkCollision(q3, start, ray, time, &obj3);
            }
            if (tempT >= TOLERANCE && tempT < t) {
               t = tempT;
               *object = obj3;
            }
         }
      } else {
         t = q3->checkCollision(q3, glm::vec3(q3->transform * startTransform), glm::mat3(q3->transform) * ray, time, &obj3);
         *object = obj3;
      }
   }
   //printf("3\n");
   if (q4) {
      if (t >= TOLERANCE) {
         tempT = q4->boundingBox.checkCollision(start, ray, time);
         if (tempT >= TOLERANCE && tempT < t) {
            if (q4->transformed) {
               tempT = q4->checkCollision(q4, glm::vec3(q4->transform * startTransform), glm::mat3(q4->transform) * ray, time, &obj4);
            } else {
               tempT = q4->checkCollision(q4, start, ray, time, &obj4);
            }
            if (tempT >= TOLERANCE && tempT < t) {
               t = tempT;
               *object = obj4;
            }
         }
      } else {
         t = q4->checkCollision(q4, glm::vec3(q4->transform * startTransform), glm::mat3(q4->transform) * ray, time, &obj4);
         *object = obj4;
      }
   }
   
   if (t < TOLERANCE) {
      *object = NULL;
   }
   //printf("4\n");
   return t;
}

float checkSphereCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
   float t = -1.0f, t0, t1, innerRoot, A, B, C;
   Sphere *thisObj = reinterpret_cast<Sphere*>(obj);
   float radius = thisObj->radius;
   glm::vec3 position = thisObj->position;
   
   A = glm::dot(ray, ray);
   B = 2.0f * glm::dot(start - position, ray);
   C = glm::dot(start - position, start - position) - (radius * radius);
   innerRoot = (B * B) - (4.0f * A * C);
   
   if (innerRoot >= 0.0f) {
      t0 = (-B - sqrt(innerRoot)) / (2.0f * A);
      t1 = (-B + sqrt(innerRoot)) / (2.0f * A);
      if (t0 >= TOLERANCE && t0 < t1) {
         t = t0;
      } else if (t1 >= TOLERANCE) {
         t = t1;
      }
   }
   
   *object = obj;
   return t;
}

float checkGWCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
   printf("GW Collision\n");
   *object = obj;
   return -1.0f;
}

float checkConeCollision(SceneObject *obj, glm::vec3 start, glm::vec3 ray, float time, SceneObject **object) {
   printf("Cone Collision\n");
   *object = obj;
   return -1.0f;
}
