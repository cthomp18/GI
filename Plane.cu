#include "hip/hip_runtime.h"
/*
   Cody Thompson
   CPE 473: Rendering
   Spring 2016
*/

#include "glm/glm.hpp"
#include "Plane.h"

Plane::Plane(glm::vec3 n, float d) : SceneObject() {
   normal = n;
   normal = glm::normalize(normal);
   distance = d;
   planePt = normal * distance;
   checkCollision = &(checkPlaneCollision);
   getNormal = &(getPlaneNormal);
}

Plane::Plane() : SceneObject() {
   checkCollision = &(checkPlaneCollision);
   getNormal = &(getPlaneNormal);
}
Plane::~Plane() {}

/*float Plane::checkCollision(glm::vec3 start, glm::vec3 ray, float time) {
   float t = -1.0f;
   //glm::vec3 origin = glm::vec3(0.0f, 0.0f, 0.0f);
   
   if (glm::dot(ray, normal) != 0.0f) {
      t = glm::dot(planePt - start, normal) / glm::dot(ray, normal);
   }
   
   return t;
   /*float t = -1.0f, tAccrue = 0.0f;
   
   //Ray Marching Boys
   
   Eigen::Vector3f pt0 = start; //Current point
   Eigen::Vector3f pt1 = start; //Next point
   Eigen::Vector3f ptW = planePt + (pt1 - Eigen::Vector3f(normal.x() * pt1.x(), normal.y() * pt1.y(), normal.z() * pt1.z()));//getPoint(pt1.x(), pt1.z(), time); //Point on wave with same x, z (i think)
   //Eigen::Vector3f normal; //Normal of said point on wave
   float step = 2.0f; //Current step (May or may not go through wave if (near) parallel to y plane)
   
   float povertwo = M_PI / 2.0f;
   if (ray.dot(normal) >= 0.0) return t;
   while (fabs(pt1.y() - ptW.y()) > TOLERANCE || fabs(pt1.x() - ptW.x()) > TOLERANCE || fabs(pt1.z() - ptW.z()) > TOLERANCE) {
      if (tAccrue > 22.0f) return t;
      //if (abs(normal.y() - 1.0) < TOLERANCE) std::cout << "ptW: " << ptW.x() << " " << ptW.y() << " " << ptW.z() << " pt1: " << pt1.x() << " " << pt1.y() << " " << pt1.z() << std::endl;
      //if (abs(normal.y() - 1.0) < TOLERANCE) std::cout << "dotthing: " << normal.dot(pt1 - ptW) << std::endl;
      //if (abs(normal.y() - 1.0) < TOLERANCE) std::cout << "tAcc " << tAccrue << std::endl;
      if (normal.dot(pt1 - ptW) < 0.0f) { //If pt1 on the other side of the wave
         //if (abs(normal.y() - 1.0) < TOLERANCE) std::cout << "ptW: " << ptW.x() << " " << ptW.y() << " " << ptW.z() << " pt1: " << pt1.x() << " " << pt1.y() << " " << pt1.z() << std::endl;
         //if (abs(normal.y() - 1.0) < TOLERANCE) std::cout << "dotthing: " << acos(normal.dot((pt1 - ptW) / (pt1 - ptW).norm())) << std::endl;
         //if (abs(normal.y() - 1.0) < TOLERANCE) std::cout << "tAcc " << tAccrue << std::endl;
         //if (abs(normal.y() - 1.0) < TOLERANCE) std::cout << "HIYYA " << tAccrue << " " << step << std::endl;
         pt1 = pt0;
         tAccrue -= step;
         step /= 4.0f;
      } else {
         tAccrue += step;
      }
      
      pt0 = pt1;
      pt1 = start + (ray * tAccrue);
      //ptW = getPoint(pt1.x(), pt1.z(), time);
      ptW = planePt + (pt1 - Eigen::Vector3f(fabs(normal.x()) * pt1.x(), fabs(normal.y()) * pt1.y(), fabs(normal.z()) * pt1.z()));
      //normal = getNormal(ptW, time);
   }
   
   t = tAccrue;
   return t;*/
//}

/*glm::vec3 Plane::getNormal(glm::vec3 iPt, float time) {
   return normal;
}*/
