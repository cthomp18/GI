#include "hip/hip_runtime.h"
/*
   Cody Thompson
   Photon Mapping
*/

#include <iostream>
#include "RayTracer.h"

using namespace std;

//extern __shared__ float sh[];

RayTracer::RayTracer(std::vector<Light*>* l, std::vector<SceneObject*>* o) {
   lights = &((*l)[0]);
   objects = &((*o)[0]);
   objSize = (*o).size();
   
   /*printf("OBJ SIZE: %d\n", (*o).size());
   printf("ADSFASDF %d\n", objSize);
   printf("TYPE   %d\n", objects[0]->type);
   for (int i = 0; i < (*o).size(); i++) {
      if (!(*o)[i]) printf("FUCKKaadsfasdf %d\n", i);
   }*/
   
   cudaStack = NULL;
}

RayTracer::RayTracer(std::vector<Light*>* l, std::vector<SceneObject*>* o, int gM, int cM, KDTreeNode* gr, KDTreeNode* cr) {
   lights = &((*l)[0]);
   objects = &((*o)[0]);
   objSize = (*o).size();
   numGPhotons = gM;
   numCPhotons = cM;
   root = gr;
   rootC1 = cr;
   /*printf("OBJ SIZE: %d\n", (*o).size());
   printf("ADSFASDF %d\n", objSize);
   printf("TYPE   %d\n", objects[0]->type);
   for (int i = 0; i < (*o).size(); i++) {
      if (!objects[i]) printf("FUCKKaadsfasdf %d\n", i);
   }*/
   
   cudaStack = NULL;
}

RayTracer::RayTracer(SceneObject** o, int osize, int gM, int cM, KDTreeNode* gr, KDTreeNode* cr) {
   objects = o;
   objSize = osize;
   numGPhotons = gM;
   numCPhotons = cM;
   root = gr;
   rootC1 = cr;
   
   cudaStack = NULL;
}

RayTracer::RayTracer() {
   cudaStack = NULL;
}
RayTracer::~RayTracer() { }

//Collision* RayTracer::trace(glm::vec3 start, glm::vec3 ray, bool unit) {
Collision* RayTracer::trace(glm::vec3 start, glm::vec3 ray, bool unit, int *shI, float *shF) {
   Collision* c = new Collision();
   /*printf("OBJ SIZE: %d\n", objSize);
   printf("TTYPE   %d\n", objects[0]->type);
   for (int i = 0; i < objSize; i++) {
      if (!objects[i]) printf("FUCKK %d\n", i);
   }*/
   /*printf("Here :)\n");
   printf("COLL %f %f %f\n", objects[0]->boundingBox.minPt.x, objects[0]->boundingBox.minPt.y, objects[0]->boundingBox.minPt.z);
   printf("COLL %f %f %f\n", objects[0]->boundingBox.maxPt.x, objects[0]->boundingBox.maxPt.y, objects[0]->boundingBox.maxPt.z);*/
   shF[6] = start.x;
   shF[7] = start.y;
   shF[8] = start.z;
   //printf("uhhhhh %f %f %f\n", start.x, start.y, start.z);
   //printf("hello? %f %f %f\n", shF[6], shF[7], shF[8]);
   c->detectRayCollision2(ray, objects, objSize, -1, unit, shI, shF);
   return c;
}

Collision* RayTracer::trace(glm::vec3 ray, int *shI, float *shF) {
//Collision* RayTracer::trace(glm::vec3 start, glm::vec3 ray, int *shI, float *shF) {

   //I01: Omit Index

   Collision* c = new Collision();
   
   shI[2] = objSize;
   
   //c->detectRayCollision(start, ray, objects, shI, shF);
   c->detectRayCollision(ray, objects, shI, shF);
   return c;
}

glm::vec3 RayTracer::calcRadiance(glm::vec3 start, glm::vec3 iPt, SceneObject* obj, bool unit, float scale, float n1, float dropoff, int threadNum, int depth, int *shI, float *shF) {
   /*float e = 2.71828;
   float alpha = 0.918;
   float beta = 1.953;*/
      
   int causts;
   float x = 0.0f, y = 0.0f, z = 0.0f, t = 0.0f, c = 0.0f, s = 0.0f;
   //printf("THREADNUM: %d\n", threadNum);
   //float sampleDistSqrd, newRadSqrd;//, scaleN;
   Photon** locateHeap;// = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));;
   int heapSize;
   glm::vec3 eRay = glm::vec3(0.0, 0.0, 1.0);
   glm::mat3 mat, matInv;
   
   glm::vec3 clr, absorbClr, reflectClr, refractClr;
   float n2 = 0.0f, time = 0.0f, reflectScale = 0.0f, tempDO = 0.0f;//, dots1 = 0.0, dots2 = 0.0, temp, temp2, mainDist, mainT, dist, reflectance = 1.0f / obj->roughness, D, F, G, m, sroot, R = 0.0f, R0 = 0.0f, innersqr = 1.0f;
   glm::vec3 colorD, colorS, colorA, color, normal, reflectRay, newStart, newIPt, crossP;
   glm::vec4 tempNormal, tempStart, tempIPt;
   glm::vec3 pigment(obj->pigment.x, obj->pigment.y, obj->pigment.z);
   glm::vec3 l, v, h, lcol, dir;
   
   color = glm::vec3(0.0f, 0.0f, 0.0f);
   clr.x = clr.y = clr.z = 0.0;
   //locateHeap.clear();
   //sampleDistSqrd = newRadSqrd = INITIAL_SAMPLE_DIST_SQRD;
   
   normal = obj->getNormal(obj, iPt, 2.0f);
   normal = glm::normalize(normal);
   
   v = start - iPt;
   v = glm::normalize(v);
   dir = -v;

   Collision* col;
   //bool shadow;
   
   absorbClr.x = reflectClr.x = refractClr.x = 0.0;
   absorbClr.y = reflectClr.y = refractClr.y = 0.0;
   absorbClr.z = reflectClr.z = refractClr.z = 0.0;
   
   colorD = pigment * obj->diffuse;
   colorS = pigment * obj->specular;
   colorA = pigment * obj->ambient;
   
   /*if (threadNum == 0) {
      printf("STACK PART: %d\n", stackPartition);
   }*/
   /*if (threadNum == 0) {
      printf("TREE START\n");
      root->printTree(root);
      printf("TREE END\n");
   } */ 
   //if (threadNum == 0) {
   if (fabs(1.0f - obj->refraction) > TOLERANCE || depth <= 0) {
      
      /*for (int lightnum = 0; lightnum < lights.size(); lightnum++) {
         dots1 = dots2 = 0.0f;
         shadow = false;
         lcol = lights[lightnum]->getColor();
         lcol = Eigen::Vector3f(1.0f, 1.0f, 1.0f);
         l = (lights[lightnum]->getPosition() - iPt);
         mainDist = l.norm();
         l.normalize();
         h = (l + v);
         h.normalize();
         l.normalize();
         col = new Collision();
         col->detectRayCollision(iPt, l, objects, -1, unit);
         if (col->time >= TOLERANCE && (l*col->time).norm() < mainDist) shadow = true;
         delete(col);
         if (shadow == false) {
            dots1 = l.dot(normal);
            if (dots1 < TOLERANCE) { 
               dots1 = 0.0f;
            }
            
            temp = dots2 = h.dot(normal);
            if (dots2 < TOLERANCE) { 
               dots2 = 0.0f;
            } else {
               for (int i = 0; i < int(reflectance); i++) {
            	   dots2 *= temp;
            	}
            }
         }
         color.x() += ((colorA.x() / float(lights.size())) + (colorD.x() * dots1) + (colorS.x() * dots2)) * lcol.x();
         color.y() += ((colorA.y() / float(lights.size())) + (colorD.y() * dots1) + (colorS.y() * dots2)) * lcol.y();
         color.z() += ((colorA.z() / float(lights.size())) + (colorD.z() * dots1) + (colorS.z() * dots2)) * lcol.z();
      }*/
      reflectScale = obj->reflection;
      
      //glm::mat3 matInv;
      if (fabs(ELLIPSOID_SCALE - 1.0) > TOLERANCE &&
          (fabs(fabs(normal[0]) - eRay[0]) > TOLERANCE ||
           fabs(fabs(normal[1]) - eRay[1]) > TOLERANCE ||
           fabs(fabs(normal[2]) - eRay[2]) > TOLERANCE)) {
         crossP = glm::cross(eRay, normal);
         crossP = glm::normalize(crossP);
         
         //float x = 0.0f, y = 0.0f, z = 0.0f, t = 0.0f, c = 0.0f, s = 0.0f;
         x = crossP.x; y = crossP.y; z = crossP.z;
         c = glm::dot(eRay, normal); s = sin(acos(glm::dot(eRay, normal))); t = 1.0 - c;
         mat = glm::mat3(t*x*x + c, t*x*y - z*s, t*x*z + y*s,
                                   t*x*y + z*s, t*y*y + c, t*y*z - x*s,
                                   t*x*z - y*s, t*y*z + x*s,	t*z*z + c);
         matInv = glm::inverse(mat);
      } else {
         matInv = glm::mat3(1.0f);
      }
      
      //inv stuff
      //volatile glm::mat3 volatile mInv;
      /*mInv.value[0].x = matInv[0][0];
      mInv.value[0].y = matInv[0][1];
      mInv.value[0].z = matInv[0][2];
      mInv.value[1].x = matInv[1][0];
      mInv.value[1].y = matInv[1][1];
      mInv.value[1].z = matInv[1][2];
      mInv.value[2].x = matInv[2][0];
      mInv.value[2].y = matInv[2][1];
      mInv.value[2].z = matInv[2][2];*/
      
      volatile float* volatile mInv = glm::value_ptr(matInv);
      
      //printf("%d\n", threadNum);
      //Photon** locateHeap = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));
      
      locateHeap = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));
      //int heapSize = 0;
      heapSize = 0;
      //if (numCPhotons > 0) { printf("fucking wut\n"); rootC1->locatePhotons(1, iPt, locateHeap, &heapSize, 0.05, &newRadSqrd, matInv, numCPhotons, cudaStack + (threadNum * stackPartition)); }
      causts = heapSize;
      
      //printf("I'm guesssing here\n");
      
      //sh[threadSpot] = iPt.x;
      //sh[threadSpot+1] = iPt.y;
      //sh[threadSpot+2] = iPt.z;
      shF[7] = INITIAL_SAMPLE_DIST_SQRD;
      shF[8] = INITIAL_SAMPLE_DIST_SQRD;
      shI[2] = 0;//heapSize;
      //if (numGPhotons > 0) root->locatePhotons(iPt, locateHeap, matInv, numGPhotons, shF + threadSpotF, shI + threadSpotI);// cudaStack + (threadNum * stackPartition));
      if (numGPhotons > 0) root->locatePhotons(iPt, locateHeap, mInv, numGPhotons, shF, shI);// cudaStack + (threadNum * stackPartition));
      //printf("HEAPSIZE FAM: %d\n", heapSize);
      //sh[threadSpot] = 0.1f;
      //if (numGPhotons > 0) root->locatePhotons(iPt, threadSpot, locateHeap, sampleDistSqrd, &newRadSqrd, numGPhotons, sh);
      //printf("sheeeet\n");
      heapSize = shI[2];
      //printf("HS: %d\n", heapSize);
      //if (heapSize) {
      //   printf("PTN INT: %f %f %f\n", locateHeap[0]->intensity.x, locateHeap[0]->intensity.y, locateHeap[0]->intensity.z);
      //}
      //for (int i = 0; i < heapSize; i++) {
      for (int i = 0; i < heapSize; i++) {
         //BRDF
         float dotProd = glm::dot(-locateHeap[i]->incidence, normal);
         //volatile float dotProd = -(locateHeap[i]->incidence.x) * normal.x;//glm::dot(-locateHeap[i]->incidence, normal);
         //dotProd += -(locateHeap[i]->incidence.y) * normal.y;
         //dotProd += -(locateHeap[i]->incidence.z) * normal.z;
         if (i < causts) {
            //float d = glm::length(locateHeap[i]->pt - iPt);
            //float w = alpha * (1 - ((1 - pow(e, -1 * beta * ((d * d) / (2 * newRadSqrd)))) / (1 - pow(e, -1 * beta))));
            color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);// * w;//* (1.0 - ((locateHeap[i]->pt - intersectPt).norm() / sqrt(newRadSqrd)));
         } else {
            //dotProd = (dotProd > 0.0f ? dotProd : 0.0f);
            //color.x += locateHeap[i]->intensity.x * dotProd;
            //color.y += locateHeap[i]->intensity.x * dotProd;
            //color.z += locateHeap[i]->intensity.x * dotProd;
            //if (dotProd < TOLERANCE) dotProd = 0.0f;
            //color += locateHeap[i]->intensity * dotProd;
            color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
         }
      }
      color /= shF[8] * M_PI;
      
      color *= (1.0 - obj->reflection) * scale;
      absorbClr.x = color.x;
      absorbClr.y = color.y;
      absorbClr.z = color.z;
      
      
      free(locateHeap);
      
   } else {
      if (depth > 0) {         
         //Do Refraction
         reflectRay = findRefract(dir, normal, obj, n1, &n2, &reflectScale, &tempDO);
         if (fabs(reflectScale - 1.0f) >= TOLERANCE) { //Total internal reflection carry-over check
            shF[6] = iPt.x;
            shF[7] = iPt.y;
            shF[8] = iPt.z;
            col = trace(reflectRay, shI, shF);
            //col = trace(iPt, reflectRay, shI, shF);
            if (col->time >TOLERANCE) {
               refractClr = calcRadiance(iPt, iPt + reflectRay * col->time, col->object, unit, scale * (1.0f - reflectScale), n2, tempDO, threadNum, depth - 1, shI, shF);
            }
            delete(col);
         }
      }
   }

   if ((obj->reflection > TOLERANCE || fabs(obj->refraction - 1.0) < TOLERANCE) && depth > 0) {
      //float randFloat;
      reflectRay = findReflect(dir, normal, obj);
      shF[6] = iPt.x;
      shF[7] = iPt.y;
      shF[8] = iPt.z;
      col = trace(reflectRay, shI, shF);
      //col = trace(iPt, reflectRay, shI, shF);
      if (col->time >= TOLERANCE) {
         reflectClr = calcRadiance(iPt, iPt + reflectRay * col->time, col->object, unit, scale * reflectScale, n1, dropoff, threadNum, depth - 1, shI, shF);
      }
      delete(col);
   }

   time = glm::length(iPt - start);
   float dropoffCalc = pow(dropoff, time);
   clr.x += (absorbClr.x + reflectClr.x + refractClr.x) * dropoffCalc;
   clr.y += (absorbClr.y + reflectClr.y + refractClr.y) * dropoffCalc;
   clr.z += (absorbClr.z + reflectClr.z + refractClr.z) * dropoffCalc;
   
	return clr;
}
//}
                  
glm::vec3 RayTracer::findReflect(glm::vec3 ray, glm::vec3 normal, SceneObject* obj) {
   glm::vec3 reflectRay;
   
   reflectRay = ray + (2.0f*normal*(glm::dot(normal, -ray)));
   reflectRay = glm::normalize(reflectRay);
   
   return reflectRay;
}

glm::vec3 RayTracer::findRefract(glm::vec3 ray, glm::vec3 normalI, SceneObject* obj, float n1, float* n2, float* R, float* dropoff) {
   glm::vec3 refractRay, normal = normalI;
   float dots1, R0, sroot, innersqr;
   
   //Determine object-ray status
   dots1 = glm::dot(-ray, normal);
   if (dots1 < 0.0f) { //Exitting
      *n2 = 1.0f; //Assume no refract object collision
      *dropoff = 1.0f;
      normal *= -1.0f;
      dots1 = glm::dot(-ray, normal);
   } else { //Entering
      *n2 = obj->indexRefraction;
      *dropoff = obj->dropoff;
   }
   
   sroot = 1.0f - ((n1 / *n2) * (n1 / *n2) * (1.0f - (dots1 * dots1)));
   if (sroot < 0.0f) { //Total internal reflection check
      *R = 1.0f;
   } else {
      //Schlick Overhead
      R0 = (n1 - *n2) / (n1 + *n2);
      R0 *= R0;
      
      innersqr = 1.0f - dots1;
      innersqr = pow(innersqr, 5);
      *R = R0 + ((1.0f - R0) * innersqr);
      
      refractRay = ((n1 / *n2) * (ray + (normal * dots1))) - (normal * sqrt(sroot));
      refractRay = glm::normalize(refractRay);
   }
   
   return refractRay;
}
