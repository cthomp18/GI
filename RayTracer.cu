#include "hip/hip_runtime.h"
/*
   Cody Thompson
   Photon Mapping
*/

#include <iostream>
//#include "RayTracer.h"
#include "RayTracer.cuh"

using namespace std;

//extern __shared__ float sh[];

RayTracer::RayTracer(std::vector<Light*>* l, std::vector<SceneObject*>* o) {
   lights = &((*l)[0]);
   objects = &((*o)[0]);
   objSize = (*o).size();
   
   /*printf("OBJ SIZE: %d\n", (*o).size());
   printf("ADSFASDF %d\n", objSize);
   printf("TYPE   %d\n", objects[0]->type);
   for (int i = 0; i < (*o).size(); i++) {
      if (!(*o)[i]) printf("FUCKKaadsfasdf %d\n", i);
   }*/
   
   cudaStack = NULL;
}

RayTracer::RayTracer(std::vector<Light*>* l, std::vector<SceneObject*>* o, int gM, int cM, KDTreeNode* gr, KDTreeNode* cr) {
   lights = &((*l)[0]);
   objects = &((*o)[0]);
   objSize = (*o).size();
   numGPhotons = gM;
   numCPhotons = cM;
   root = gr;
   rootC1 = cr;
   /*printf("OBJ SIZE: %d\n", (*o).size());
   printf("ADSFASDF %d\n", objSize);
   printf("TYPE   %d\n", objects[0]->type);
   for (int i = 0; i < (*o).size(); i++) {
      if (!objects[i]) printf("FUCKKaadsfasdf %d\n", i);
   }*/
   
   cudaStack = NULL;
}

RayTracer::RayTracer(SceneObject** o, int osize, int gM, int cM, KDTreeNode* gr, KDTreeNode* cr) {
   objects = o;
   objSize = osize;
   numGPhotons = gM;
   numCPhotons = cM;
   root = gr;
   rootC1 = cr;
   
   cudaStack = NULL;
}

RayTracer::RayTracer() {
   cudaStack = NULL;
}
RayTracer::~RayTracer() { }

//Collision* RayTracer::trace(glm::vec3 start, glm::vec3 ray, bool unit) {
Collision* RayTracer::trace(glm::vec3 start, glm::vec3 ray, bool unit, int *shI, float *shF) {
   Collision* c = new Collision();

   shF[6] = start.x;
   shF[7] = start.y;
   shF[8] = start.z;

   c->detectRayCollision2(ray, objects, objSize, -1, unit, shI, shF);
   return c;
}

Collision* RayTracer::trace(glm::vec3 ray, int *shI, float *shF) {
//Collision* RayTracer::trace(glm::vec3 start, glm::vec3 ray, int *shI, float *shF) {

   //I01: Omit Index

   Collision* c = new Collision();
   
   shI[2] = objSize;
   
   c->detectRayCollision(ray, objects, shI, shF);
   return c;
}

__device__ __noinline__ volatile float * volatile RayTracer::getMatInv(glm::vec3 iPt, SceneObject* obj, float *shF) {
   /*
   7: c
   8: s
   9: t
   */
   
   glm::mat3 matInv;
   glm::vec3 normal = obj->getNormal(obj, iPt, shF);
   glm::vec3 eRay = glm::vec3(0.0, 0.0, 1.0);
   
   normal = glm::normalize(normal);
   if (fabs(ELLIPSOID_SCALE - 1.0) > TOLERANCE &&
       (fabs(fabs(normal[0]) - eRay[0]) > TOLERANCE ||
        fabs(fabs(normal[1]) - eRay[1]) > TOLERANCE ||
        fabs(fabs(normal[2]) - eRay[2]) > TOLERANCE)) {
        
      glm::vec3 crossP = glm::cross(eRay, normal);
      crossP = glm::normalize(crossP);
      
      shF[6] = glm::dot(eRay, normal); shF[7] = sin(acos(glm::dot(eRay, normal))); shF[8] = 1.0 - shF[6];
      glm::mat3 mat = glm::mat3(shF[8]*crossP.x*crossP.x + shF[6], shF[8]*crossP.x*crossP.y - crossP.z*shF[7], shF[8]*crossP.x*crossP.z + crossP.y*shF[7],
                                shF[8]*crossP.x*crossP.y + crossP.z*shF[7], shF[8]*crossP.y*crossP.y + shF[6], shF[8]*crossP.y*crossP.z - crossP.x*shF[7],
                                shF[8]*crossP.x*crossP.z - crossP.y*shF[7], shF[8]*crossP.y*crossP.z + crossP.x*shF[7], shF[8]*crossP.z*crossP.z + shF[6]);
      matInv = glm::inverse(mat);
   } else {
      matInv = glm::mat3(1.0f);
   }
   
   //volatile float* volatile mInv;// = 
   return glm::value_ptr(matInv);
}

__device__ __noinline__ glm::vec3 RayTracer::accumulatePhotons(Photon **locateHeap, glm::vec3 iPt, SceneObject* obj, float *shF, int *shI) {
   glm::vec3 clr(0.0f, 0.0f, 0.0f);
   for (volatile int i = 0; i < shI[2]; i++) {
      //BRDF
      glm::vec3 normal = obj->getNormal(obj, iPt, shF);
      normal = glm::normalize(normal);
      float dotProd = glm::dot(-locateHeap[i]->incidence, normal);
      //if (shI[0] < causts) {
         //color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
      clr += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
      //} else {
         //color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
         //clr += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
      //}
   }
   return clr;
}

__attribute__ ((noinline)) void __attribute__ ((noinline)) RayTracer::getNormal(SceneObject *obj, glm::vec3 iPt, float *shF) {
   glm::vec3 normal = obj->getNormal(obj, iPt, shF);
   normal = glm::normalize(normal);
   shF[3] = normal.x;
   shF[4] = normal.y;
   shF[5] = normal.z;
}

glm::vec3 RayTracer::calcRadiance(glm::vec3 start, glm::vec3 iPt, SceneObject* obj, bool unit, float scale, float n1, float dropoff, int threadNum, int depth, int *shI, float *shF) {
   /*float e = 2.71828;
   float alpha = 0.918;
   float beta = 1.953;*/
      
   //int causts;
   //volatile float x = 0.0f, y = 0.0f, z = 0.0f, t = 0.0f, c = 0.0f, s = 0.0f;
   //printf("THREADNUM: %d\n", threadNum);
   //float sampleDistSqrd, newRadSqrd;//, scaleN;
   //Photon** locateHeap;// = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));;
   //int heapSize;
   //glm::vec3 eRay = glm::vec3(0.0, 0.0, 1.0);
   //glm::mat3 mat, matInv;
   //volatile float reflectScale = obj->reflection;
   //volatile float * volatile matInv;
      //if (1) matInv = (volatile float * volatile)glm::value_ptr(glm::mat3(1.0f));//getMatInv(iPt, obj, shF);
      //if (1) matInv = getMatInv(iPt, obj, shF);
   //glm::vec3 normal = obj->getNormal(obj, iPt, shF);
   getNormal(obj, iPt, shF);
      //normal = glm::normalize(normal);
   shF[1] = obj->reflection;
   //volatile float refract = obj->refraction;
   shF[2] = obj->refraction;
   //float dropoffCalc = glm::length(iPt - start);
   shF[0] = glm::length(iPt - start);
   //dropoffCalc = pow(dropoff, dropoffCalc);
   shF[0] = pow(dropoff, shF[0]);
   
   //glm::vec3 clr, absorbClr, reflectClr, refractClr;
   glm::vec3 clr(0.0f, 0.0f, 0.0f);
   //float n2 = 0.0f;//, reflectScale = 0.0f, tempDO = 0.0f;//, dots1 = 0.0, dots2 = 0.0, temp, temp2, mainDist, mainT, dist, reflectance = 1.0f / obj->roughness, D, F, G, m, sroot, R = 0.0f, R0 = 0.0f, innersqr = 1.0f;
   //glm::vec3 colorD, colorS, colorA, color, normal, reflectRay, newStart, newIPt, crossP;
   //glm::vec4 tempNormal, tempStart, tempIPt;
   //glm::vec3 pigment(obj->pigment.x, obj->pigment.y, obj->pigment.z);
   //glm::vec3 l, v, h, lcol, dir;
   
   //color = glm::vec3(0.0f, 0.0f, 0.0f);
   
   //glm::vec3 normal = obj->getNormal(obj, iPt, shF);
   //normal = glm::normalize(normal);
   
   //v = start - iPt;
   //v = glm::normalize(v);
   //dir = -v;

   //Collision* col;
   
   //float time = glm::length(iPt - start);
   //float dropoffCalc = pow(dropoff, time);
   
   //absorbClr.x = reflectClr.x = refractClr.x = 0.0;
   //absorbClr.y = reflectClr.y = refractClr.y = 0.0;
   //absorbClr.z = reflectClr.z = refractClr.z = 0.0;
   
   //colorD = pigment * obj->diffuse;
   //colorS = pigment * obj->specular;
   //colorA = pigment * obj->ambient;
   
   //if (fabs(1.0f - refract) > TOLERANCE || depth <= 0) {
      //reflectScale = obj->reflection;
      
      
      
      //printf("%d\n", threadNum);
      //Photon** locateHeap = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));
      
      Photon** locateHeap = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));
      //int heapSize = 0;
      //heapSize = 0;
      //if (numCPhotons > 0) rootC1->locatePhotons(1, iPt, locateHeap, &heapSize, 0.05, &newRadSqrd, matInv, numCPhotons, cudaStack + (threadNum * stackPartition)); 
      //causts = heapSize;
      //volatile int causts = 0;//shI[2];
      
      //sh[threadSpot] = iPt.x;
      //sh[threadSpot+1] = iPt.y;
      //sh[threadSpot+2] = iPt.z;
      shF[7] = INITIAL_SAMPLE_DIST_SQRD;
      shF[8] = INITIAL_SAMPLE_DIST_SQRD;
      shI[2] = 0;//heapSize;
      //asm volatile("membar.cta;");
      volatile float * volatile matInv;
      if (1) matInv = (volatile float * volatile)glm::value_ptr(glm::mat3(1.0f));//getMatInv(iPt, obj, shF);
      //matInv = getMatInv(iPt, obj, shF);
      //if (numGPhotons > 0) root->locatePhotons(iPt, locateHeap, matInv, numGPhotons, shF + threadSpotF, shI + threadSpotI);// cudaStack + (threadNum * stackPartition));
      shI[0] = numGPhotons;
      //asm volatile("membar.cta;");
      if (numGPhotons) root->locatePhotons(iPt, locateHeap, matInv, numGPhotons, shF, shI);// cudaStack + (threadNum * stackPartition));
      //printf("HEAPSIZE FAM: %d\n", heapSize);
      //sh[threadSpot] = 0.1f;
      //if (numGPhotons > 0) root->locatePhotons(iPt, threadSpot, locateHeap, sampleDistSqrd, &newRadSqrd, numGPhotons, sh);
      //printf("sheeeet\n");
      //heapSize = shI[2];
      //printf("HS: %d\n", heapSize);
      //if (heapSize) {
      //   printf("PTN INT: %f %f %f\n", locateHeap[0]->intensity.x, locateHeap[0]->intensity.y, locateHeap[0]->intensity.z);
      //}
      //for (int i = 0; i < heapSize; i++) {
      //clr = accumulatePhotons(locateHeap, iPt, obj, shF, shI);
      /*for (volatile int i = 0; i < shI[2]; i++) {
         //BRDF
         glm::vec3 normal = obj->getNormal(obj, iPt, shF);
         normal = glm::normalize(normal);
         float dotProd = glm::dot(-locateHeap[i]->incidence, normal);
         //if (shI[0] < causts) {
            //color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
            clr += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
         //} else {
            //color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
            //clr += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
         //}
      }*/
      //color /= shF[8] * M_PI;
      //clr /= shF[8] * M_PI;
      
      //color *= (1.0 - obj->reflection) * scale;
      //clr *= (1.0 - reflectScale) * scale;
      //clr.x = clr.x * (1.0f - reflectScale) * scale;
      //absorbClr.x = color.x;
      //absorbClr.y = color.y;
      //absorbClr.z = color.z;
      //clr.x += color.x * dropoffCalc;
      //clr.y += color.y * dropoffCalc;
      //clr.z += color.z * dropoffCalc;
      //clr += color;
      
      free(locateHeap);
      
   //} 
   /*else {
      if (depth > 0) {         
         //Do Refraction
         reflectRay = findRefract(dir, normal, obj, n1, &n2, &reflectScale, &tempDO);
         if (fabs(reflectScale - 1.0f) >= TOLERANCE) { //Total internal reflection carry-over check
            shF[6] = iPt.x;
            shF[7] = iPt.y;
            shF[8] = iPt.z;
            col = trace(reflectRay, shI, shF);
            //col = trace(iPt, reflectRay, shI, shF);
            if (col->time >TOLERANCE) {
               //refractClr = calcRadiance(iPt, iPt + reflectRay * col->time, col->object, unit, scale * (1.0f - reflectScale), n2, tempDO, threadNum, depth - 1, shI, shF);
               color = calcRadiance(iPt, iPt + reflectRay * col->time, col->object, unit, scale * (1.0f - reflectScale), n2, tempDO, threadNum, depth - 1, shI, shF);
               clr.x += color.x * dropoffCalc;
               clr.y += color.y * dropoffCalc;
               clr.z += color.z * dropoffCalc;
            }
            delete(col);
         }
      }
   }*/

   /*if ((obj->reflection > TOLERANCE || fabs(obj->refraction - 1.0) < TOLERANCE) && depth > 0) {
      //float randFloat;
      reflectRay = findReflect(dir, normal, obj);
      shF[6] = iPt.x;
      shF[7] = iPt.y;
      shF[8] = iPt.z;
      col = trace(reflectRay, shI, shF);
      //col = trace(iPt, reflectRay, shI, shF);
      if (col->time >= TOLERANCE) {
         //reflectClr = calcRadiance(iPt, iPt + reflectRay * col->time, col->object, unit, scale * reflectScale, n1, dropoff, threadNum, depth - 1, shI, shF);
         color = calcRadiance(iPt, iPt + reflectRay * col->time, col->object, unit, scale * reflectScale, n1, dropoff, threadNum, depth - 1, shI, shF);
         clr.x += color.x * dropoffCalc;
         clr.y += color.y * dropoffCalc;
         clr.z += color.z * dropoffCalc;
      }
      delete(col);
   }*/

   //time = glm::length(iPt - start);
   //float dropoffCalc = pow(dropoff, time);
   //clr.x += (absorbClr.x + reflectClr.x + refractClr.x) * dropoffCalc;
   //clr.y += (absorbClr.y + reflectClr.y + refractClr.y) * dropoffCalc;
   //clr.z += (absorbClr.z + reflectClr.z + refractClr.z) * dropoffCalc;
   
   
   clr *= shF[0];
   
	return clr;
}

/*glm::vec3 RayTracer::calcRadiance(glm::vec3 start, glm::vec3 iPt, SceneObject* obj, bool unit, float scale, float n1, float dropoff, int threadNum, int depth, int *shI, float *shF) {
   //float e = 2.71828;
   //float alpha = 0.918;
   //float beta = 1.953;
      
   int causts;
   float x = 0.0f, y = 0.0f, z = 0.0f, t = 0.0f, c = 0.0f, s = 0.0f;
   //printf("THREADNUM: %d\n", threadNum);
   //float sampleDistSqrd, newRadSqrd;//, scaleN;
   Photon** locateHeap;// = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));;
   int heapSize;
   glm::vec3 eRay = glm::vec3(0.0, 0.0, 1.0);
   glm::mat3 mat, matInv;
   
   glm::vec3 clr, absorbClr, reflectClr, refractClr;
   float n2 = 0.0f, time = 0.0f, reflectScale = 0.0f, tempDO = 0.0f;//, dots1 = 0.0, dots2 = 0.0, temp, temp2, mainDist, mainT, dist, reflectance = 1.0f / obj->roughness, D, F, G, m, sroot, R = 0.0f, R0 = 0.0f, innersqr = 1.0f;
   glm::vec3 colorD, colorS, colorA, color, normal, reflectRay, newStart, newIPt, crossP;
   glm::vec4 tempNormal, tempStart, tempIPt;
   glm::vec3 pigment(obj->pigment.x, obj->pigment.y, obj->pigment.z);
   glm::vec3 l, v, h, lcol, dir;
   
   color = glm::vec3(0.0f, 0.0f, 0.0f);
   clr.x = clr.y = clr.z = 0.0;
   //locateHeap.clear();
   //sampleDistSqrd = newRadSqrd = INITIAL_SAMPLE_DIST_SQRD;
   
   normal = obj->getNormal(obj, iPt, shF);
   normal = glm::normalize(normal);
   
   v = start - iPt;
   v = glm::normalize(v);
   dir = -v;

   Collision* col;
   //bool shadow;
   
   absorbClr.x = reflectClr.x = refractClr.x = 0.0;
   absorbClr.y = reflectClr.y = refractClr.y = 0.0;
   absorbClr.z = reflectClr.z = refractClr.z = 0.0;
   
   colorD = pigment * obj->diffuse;
   colorS = pigment * obj->specular;
   colorA = pigment * obj->ambient;
   
   //if (threadNum == 0) {
   //   printf("STACK PART: %d\n", stackPartition);
   //}
   //if (threadNum == 0) {
   //   printf("TREE START\n");
   //   root->printTree(root);
   //   printf("TREE END\n");
   //}  
   //if (threadNum == 0) {
   if (fabs(1.0f - obj->refraction) > TOLERANCE || depth <= 0) {
      
      /*for (int lightnum = 0; lightnum < lights.size(); lightnum++) {
         dots1 = dots2 = 0.0f;
         shadow = false;
         lcol = lights[lightnum]->getColor();
         lcol = Eigen::Vector3f(1.0f, 1.0f, 1.0f);
         l = (lights[lightnum]->getPosition() - iPt);
         mainDist = l.norm();
         l.normalize();
         h = (l + v);
         h.normalize();
         l.normalize();
         col = new Collision();
         col->detectRayCollision(iPt, l, objects, -1, unit);
         if (col->time >= TOLERANCE && (l*col->time).norm() < mainDist) shadow = true;
         delete(col);
         if (shadow == false) {
            dots1 = l.dot(normal);
            if (dots1 < TOLERANCE) { 
               dots1 = 0.0f;
            }
            
            temp = dots2 = h.dot(normal);
            if (dots2 < TOLERANCE) { 
               dots2 = 0.0f;
            } else {
               for (int i = 0; i < int(reflectance); i++) {
            	   dots2 *= temp;
            	}
            }
         }
         color.x() += ((colorA.x() / float(lights.size())) + (colorD.x() * dots1) + (colorS.x() * dots2)) * lcol.x();
         color.y() += ((colorA.y() / float(lights.size())) + (colorD.y() * dots1) + (colorS.y() * dots2)) * lcol.y();
         color.z() += ((colorA.z() / float(lights.size())) + (colorD.z() * dots1) + (colorS.z() * dots2)) * lcol.z();
      }*//*
      reflectScale = obj->reflection;
      
      //glm::mat3 matInv;
      if (fabs(ELLIPSOID_SCALE - 1.0) > TOLERANCE &&
          (fabs(fabs(normal[0]) - eRay[0]) > TOLERANCE ||
           fabs(fabs(normal[1]) - eRay[1]) > TOLERANCE ||
           fabs(fabs(normal[2]) - eRay[2]) > TOLERANCE)) {
         crossP = glm::cross(eRay, normal);
         crossP = glm::normalize(crossP);
         
         //float x = 0.0f, y = 0.0f, z = 0.0f, t = 0.0f, c = 0.0f, s = 0.0f;
         x = crossP.x; y = crossP.y; z = crossP.z;
         c = glm::dot(eRay, normal); s = sin(acos(glm::dot(eRay, normal))); t = 1.0 - c;
         mat = glm::mat3(t*x*x + c, t*x*y - z*s, t*x*z + y*s,
                                   t*x*y + z*s, t*y*y + c, t*y*z - x*s,
                                   t*x*z - y*s, t*y*z + x*s,	t*z*z + c);
         matInv = glm::inverse(mat);
      } else {
         matInv = glm::mat3(1.0f);
      }
      
      //inv stuff
      //volatile glm::mat3 volatile mInv;
      //mInv.value[0].x = matInv[0][0];
      //mInv.value[0].y = matInv[0][1];
      //mInv.value[0].z = matInv[0][2];
      //mInv.value[1].x = matInv[1][0];
      //mInv.value[1].y = matInv[1][1];
      //mInv.value[1].z = matInv[1][2];
      //mInv.value[2].x = matInv[2][0];
      //mInv.value[2].y = matInv[2][1];
      //mInv.value[2].z = matInv[2][2];
      
      volatile float* volatile mInv = glm::value_ptr(matInv);
      
      //printf("%d\n", threadNum);
      //Photon** locateHeap = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));
      
      locateHeap = (Photon**)malloc(CUTOFF_HEAP_SIZE * sizeof(Photon*));
      //int heapSize = 0;
      heapSize = 0;
      //if (numCPhotons > 0) rootC1->locatePhotons(1, iPt, locateHeap, &heapSize, 0.05, &newRadSqrd, matInv, numCPhotons, cudaStack + (threadNum * stackPartition));
      causts = heapSize;
      
      //printf("I'm guesssing here\n");
      
      //sh[threadSpot] = iPt.x;
      //sh[threadSpot+1] = iPt.y;
      //sh[threadSpot+2] = iPt.z;
      shF[7] = INITIAL_SAMPLE_DIST_SQRD;
      shF[8] = INITIAL_SAMPLE_DIST_SQRD;
      shI[2] = 0;//heapSize;
      //if (numGPhotons > 0) root->locatePhotons(iPt, locateHeap, matInv, numGPhotons, shF + threadSpotF, shI + threadSpotI);// cudaStack + (threadNum * stackPartition));
      if (numGPhotons > 0) root->locatePhotons(iPt, locateHeap, mInv, numGPhotons, shF, shI);// cudaStack + (threadNum * stackPartition));
      //sh[threadSpot] = 0.1f;
      //if (numGPhotons > 0) root->locatePhotons(iPt, threadSpot, locateHeap, sampleDistSqrd, &newRadSqrd, numGPhotons, sh);
      heapSize = shI[2];
      //printf("HS: %d\n", heapSize);
      //if (heapSize) {
      //   printf("PTN INT: %f %f %f\n", locateHeap[0]->intensity.x, locateHeap[0]->intensity.y, locateHeap[0]->intensity.z);
      //}
      //for (int i = 0; i < heapSize; i++) {
      for (int i = 0; i < heapSize; i++) {
         //BRDF
         float dotProd = glm::dot(-locateHeap[i]->incidence, normal);
         //volatile float dotProd = -(locateHeap[i]->incidence.x) * normal.x;//glm::dot(-locateHeap[i]->incidence, normal);
         //dotProd += -(locateHeap[i]->incidence.y) * normal.y;
         //dotProd += -(locateHeap[i]->incidence.z) * normal.z;
         if (i < causts) {
            //float d = glm::length(locateHeap[i]->pt - iPt);
            //float w = alpha * (1 - ((1 - pow(e, -1 * beta * ((d * d) / (2 * newRadSqrd)))) / (1 - pow(e, -1 * beta))));
            color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);// * w;//* (1.0 - ((locateHeap[i]->pt - intersectPt).norm() / sqrt(newRadSqrd)));
         } else {
            //dotProd = (dotProd > 0.0f ? dotProd : 0.0f);
            //color.x += locateHeap[i]->intensity.x * dotProd;
            //color.y += locateHeap[i]->intensity.x * dotProd;
            //color.z += locateHeap[i]->intensity.x * dotProd;
            //if (dotProd < TOLERANCE) dotProd = 0.0f;
            //color += locateHeap[i]->intensity * dotProd;
            color += (locateHeap[i]->intensity) * (dotProd > 0.0f ? dotProd : 0.0f);
         }
      }
      color /= shF[8] * M_PI;
      
      color *= (1.0 - obj->reflection) * scale;
      absorbClr.x = color.x;
      absorbClr.y = color.y;
      absorbClr.z = color.z;
      
      
      free(locateHeap);
      
   } else {
      if (depth > 0) {         
         //Do Refraction
         reflectRay = findRefract(dir, normal, obj, n1, &n2, &reflectScale, &tempDO);
         if (fabs(reflectScale - 1.0f) >= TOLERANCE) { //Total internal reflection carry-over check
            shF[6] = iPt.x;
            shF[7] = iPt.y;
            shF[8] = iPt.z;
            col = trace(reflectRay, shI, shF);
            //col = trace(iPt, reflectRay, shI, shF);
            if (col->time >TOLERANCE) {
               refractClr = calcRadiance(iPt, iPt + reflectRay * col->time, col->object, unit, scale * (1.0f - reflectScale), n2, tempDO, threadNum, depth - 1, shI, shF);
            }
            delete(col);
         }
      }
   }

   if ((obj->reflection > TOLERANCE || fabs(obj->refraction - 1.0) < TOLERANCE) && depth > 0) {
      //float randFloat;
      reflectRay = findReflect(dir, normal, obj);
      shF[6] = iPt.x;
      shF[7] = iPt.y;
      shF[8] = iPt.z;
      col = trace(reflectRay, shI, shF);
      //col = trace(iPt, reflectRay, shI, shF);
      if (col->time >= TOLERANCE) {
         reflectClr = calcRadiance(iPt, iPt + reflectRay * col->time, col->object, unit, scale * reflectScale, n1, dropoff, threadNum, depth - 1, shI, shF);
      }
      delete(col);
   }

   time = glm::length(iPt - start);
   float dropoffCalc = pow(dropoff, time);
   clr.x += (absorbClr.x + reflectClr.x + refractClr.x) * dropoffCalc;
   clr.y += (absorbClr.y + reflectClr.y + refractClr.y) * dropoffCalc;
   clr.z += (absorbClr.z + reflectClr.z + refractClr.z) * dropoffCalc;
   
	return clr;
}
//}*/
                  
glm::vec3 RayTracer::findReflect(glm::vec3 ray, glm::vec3 normal, SceneObject* obj) {
   glm::vec3 reflectRay;
   
   reflectRay = ray + (2.0f*normal*(glm::dot(normal, -ray)));
   reflectRay = glm::normalize(reflectRay);
   
   return reflectRay;
}

glm::vec3 RayTracer::findRefract(glm::vec3 ray, glm::vec3 normalI, SceneObject* obj, float n1, float* n2, float* R, float* dropoff) {
   glm::vec3 refractRay, normal = normalI;
   float dots1, R0, sroot, innersqr;
   
   //Determine object-ray status
   dots1 = glm::dot(-ray, normal);
   if (dots1 < 0.0f) { //Exitting
      *n2 = 1.0f; //Assume no refract object collision
      *dropoff = 1.0f;
      normal *= -1.0f;
      dots1 = glm::dot(-ray, normal);
   } else { //Entering
      *n2 = obj->indexRefraction;
      *dropoff = obj->dropoff;
   }
   
   sroot = 1.0f - ((n1 / *n2) * (n1 / *n2) * (1.0f - (dots1 * dots1)));
   if (sroot < 0.0f) { //Total internal reflection check
      *R = 1.0f;
   } else {
      //Schlick Overhead
      R0 = (n1 - *n2) / (n1 + *n2);
      R0 *= R0;
      
      innersqr = 1.0f - dots1;
      innersqr = pow(innersqr, 5);
      *R = R0 + ((1.0f - R0) * innersqr);
      
      refractRay = ((n1 / *n2) * (ray + (normal * dots1))) - (normal * sqrt(sroot));
      refractRay = glm::normalize(refractRay);
   }
   
   return refractRay;
}
