#include "hip/hip_runtime.h"
/*
   Cody Thompson
   CPE 473: Rendering
   Spring 2016
*/

#include "glm/glm.hpp"
#include "glm/gtc/type_ptr.hpp"
#include "Triangle.h"

Triangle::Triangle(glm::vec3 pt1, glm::vec3 pt2, glm::vec3 pt3) : SceneObject() {
   a = pt1;
   b = pt2;
   c = pt3;
   
   normal = glm::cross(b - a, c - a);
   normal = glm::normalize(normal);
   smooth = false;
   type = 3;
   
   checkCollision = (&checkTriCollision);
   getNormal = (&getTriNormal);
}

Triangle::Triangle(glm::vec3 pt1, glm::vec3 pt2, glm::vec3 pt3, bool smoothCheck) : SceneObject() {
   a = pt1;
   b = pt2;
   c = pt3;
   
   normal = glm::cross(b - a, c - a);
   normal = glm::normalize(normal);
   smooth = false;
   
   if (smoothCheck) {
      smooth = true;
      glm::vec3 v1 = b - a, v2 = c - a;
      dotAB = glm::dot(v1, v1);
      dotAC = glm::dot(v2, v2);
      dotABC = glm::dot(v1, v2);
      multiplier = 1.0f / ((dotAB * dotAC) - (dotABC * dotABC));
   }
   
   checkCollision = (&checkTriCollision);
   getNormal = (&getTriNormal);
}

Triangle::Triangle() : SceneObject() {}
Triangle::~Triangle() {
   //printf("yo dawg wuts gud\n");
}

Triangle::Triangle(Triangle* o) : SceneObject() {
   a = o->a;
   b = o->b;
   c = o->c;
   aNor = o->aNor;
   bNor = o->bNor;
   cNor = o->cNor;
   
   dotAB = o->dotAB;
   dotAC = o->dotAC;
   dotABC = o->dotABC;
   multiplier = o->multiplier;
      
   normal = o->normal;
   smooth = o->smooth;
   
   checkCollision = (&checkTriCollision);
   getNormal = (&getTriNormal);
   
   copyData(o);
}

/*float Triangle::checkCollision(glm::vec3 start, glm::vec3 ray, float time) {
   std::cout << "Triangle Collision" << std::endl;
   glm::mat3 A, Ai;
   double detA, t, beta, gamma;
   A = glm::mat3(a.x - b.x, a.x - c.x, ray.x,
                 a.y - b.y, a.y - c.y, ray.y,
                 a.z - b.z, a.z - c.z, ray.z);
   detA = glm::determinant(A);
   //std::cout << "making sure lol" << std::endl;
   //std::cout << detA << std::endl;
   //if (std::fabs(detA) > 0.0f) {
      //std::cout << "hi?" << std::endl;
      Ai = A;
      Ai[0][2] = a.x - start.x; Ai[1][2] = a.y - start.y; Ai[2][2] = a.z - start.z;
      t = glm::determinant(Ai) / detA;
      if (t > TOLERANCE) {
         //std::cout << "hiya" << std::endl;
         Ai = A;
         Ai[0][0] = a.x - start.x; Ai[1][0] = a.y - start.y; Ai[2][0] = a.z - start.z;
         gamma = glm::determinant(Ai) / detA;
         if (gamma >= 0.0f && gamma <= 1.0f) {
            //std::cout << "hello" << std::endl;
            Ai = A;
            Ai[0][1] = a.x - start.x; Ai[1][1] = a.y - start.y; Ai[2][1] = a.z - start.z;
            beta = glm::determinant(Ai) / detA;
            //std::cout << "Beta: " << beta << " Gamma: " << gamma << std::endl;
            if (beta >= 0.0f && beta + gamma <= 1.0f) {
               //std::cout << t << std::endl;
               return t;
            }
         }
      }
   //}
   
   return -1.0f;
}*/

/*glm::vec3 Triangle::getNormal(glm::vec3 iPt, float time) {
   float alpha, beta, gamma;
   glm::vec3 normalConstructor = normal;
   printf("sup dawg\n");
   if (smooth) {
      float dotABP = glm::dot(b - a, iPt - a), dotACP = glm::dot(c - a, iPt - a);
      alpha = ((dotAC * dotABP) - (dotABC * dotACP)) * multiplier;
      beta = ((dotAB * dotACP) - (dotABC * dotABP)) * multiplier;
      gamma = (1.0f - alpha) - beta;
      
      normalConstructor = bNor * alpha;
      normalConstructor += cNor * beta;
      normalConstructor += aNor * gamma;
      normalConstructor = glm::normalize(normalConstructor);
   }
   
   return normalConstructor;
}*/

void Triangle::constructBB() {
   glm::vec3 minPt = glm::vec3(FLT_MAX, FLT_MAX, FLT_MAX);
   glm::vec3 maxPt = glm::vec3(FLT_MIN, FLT_MIN, FLT_MIN);
   
   for (int i = 0; i < 3; i++) {
      minPt[i] = fmin(fmin(a[i], b[i]), c[i]);
      maxPt[i] = fmax(fmax(a[i], b[i]), c[i]);
   }
   
   boundingBox = BoundingBox(minPt, maxPt);
}

void Triangle::printObj() {
   std::cout << a.x << " " << a.y << " " << a.z << std::endl;
   std::cout << b.x << " " << b.y << " " << b.z << std::endl;
   std::cout << c.x << " " << c.y << " " << c.z << std::endl;
}
