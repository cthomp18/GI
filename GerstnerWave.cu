#include "hip/hip_runtime.h"
/*
   Cody Thompson
   Photon Mapping
*/

#include "GerstnerWave.h"
using namespace std;
GerstnerWave::GerstnerWave(float a, float w, float s, glm::vec3 d, glm::vec3 lowerleft, glm::vec3 upperright, float yPosition) : SceneObject() {
   amplitude = (float*)malloc(sizeof(float));
   wavelength = (float*)malloc(sizeof(float));
   frequency = (float*)malloc(sizeof(float));
   speedPC = (float*)malloc(sizeof(float));
   direction = (glm::vec3*)malloc(sizeof(glm::vec3));
   steepness = (float*)malloc(sizeof(float));
   
   amplitude[0] = a;
   wavelength[0] = w;
   frequency[0] = sqrt(9.81f * ((2.0f * M_PI) / w));
   speedPC[0] = s * sqrt(9.81f * ((2.0f * M_PI) / w));
   d = glm::normalize(d);
   direction[0] = d;
   steepness[0] = 1.0f / (sqrt(9.81f * ((2.0f * M_PI) / w)) * a * 2.0f);
   lb = lowerleft;
   ub = upperright;
   yPos = yPosition;
   
   waves = 1;
}

GerstnerWave::GerstnerWave() : SceneObject() {
   amplitude = NULL;
   wavelength = NULL;
   frequency = NULL;
   speedPC = NULL;
   direction = NULL;
   steepness = NULL;

   lb = ub = glm::vec3(0.0f,0.0f,0.0f);
   yPos = 0.0f;
   
   waves = 0;
}

GerstnerWave::~GerstnerWave() {}

// Not using this function anymore btw

float GerstnerWave::checkCollision(glm::vec3 start, glm::vec3 ray, float time) {
   /*float t = -1.0f, tAccrue = 0.0f;
   
   //Ray Marching Boys
   
   Eigen::Vector3f pt0 = start; //Current point
   Eigen::Vector3f pt1 = start; //Next point
   Eigen::Vector3f ptW = getPoint(pt1.x(), pt1.z(), time); //Point on wave with same x, z (i think)
   Eigen::Vector3f normal; //Normal of said point on wave
   float step = 0.01f; //Current step (May or may not go through wave if (near) parallel to y plane)
   
   float povertwo = M_PI / 2.0f;
   
   while (pt1.y() - ptW.y() > TOLERANCE) {// (|| abs(pt1.x() - ptW.x()) > TOLERANCE || abs(pt1.z() - ptW.z()) > TOLERANCE)
      if (tAccrue > 22.0f) break;
      //std::cout << "ptW: " << ptW.x() << " " << ptW.y() << " " << ptW.z() << " pt1: " << pt1.x() << " " << pt1.y() << " " << pt1.z() << std::endl;
      //std::cout << "dotthing: " << normal.dot(pt1 - ptW) << std::endl;
      //std::cout << "tAcc " << tAccrue << std::endl;
      if (normal.dot(pt1 - ptW) < TOLERANCE || pt1.y() - ptW.y() < TOLERANCE) { //If pt1 on the other side of the wave
         pt1 = pt0;
         tAccrue -= step;
         step /= 4.0f;
      } else {
         tAccrue += step;
      }
      
      pt0 = pt1;
      pt1 = pt0 + (ray * step);
      ptW = getPoint(pt1.x(), pt1.z(), time);
      normal = getNormal(pt1, time);
      //std::cout << tAccrue << std::endl;
   }
   
   if (pt1.y() - ptW.y() <= TOLERANCE) { t = tAccrue; }
   return t;*/
   return -1.0f;
}

glm::vec3 GerstnerWave::getNormal(glm::vec3 iPt, float time) {
   glm::vec3 normal = glm::vec3(0.0f, 1.0f, 0.0f);
   float wa, wdpt, coswa;
   
   for (int i = 0; i < waves; i++) {
      wa = frequency[i] * amplitude[i];
      wdpt = glm::dot(frequency[i] * glm::vec3(direction[i].x, 0.0f, direction[i].z), iPt) + (time * speedPC[i]);
      coswa = wa * cos(wdpt);
      normal -= glm::vec3(direction[i].x * coswa, (steepness[i] * wa * sin(wdpt)), direction[i].z * coswa);
   }
   normal = glm::normalize(normal);
   
   return normal;
}

glm::vec3 GerstnerWave::getPoint(float x, float z, float time) {
   glm::vec3 iPt = glm::vec3(x, 0.0f, z);
   float wdpt, qacos;
   
   for (int i = 0; i < waves; i++) {
      wdpt = glm::dot(frequency[i] * direction[i], glm::vec3(x, 0.0f, z)) + (time * speedPC[i]);
      qacos = steepness[i] * amplitude[i] * cos(wdpt);
      iPt += glm::vec3(qacos * direction[i].x, (amplitude[i] * sin(wdpt)) + yPos, qacos * direction[i].z);
   }
   
   return iPt;
}

void GerstnerWave::addWave(float a, float w, float s, glm::vec3 d) {
   waves++;
   
   amplitude = (float*)realloc(amplitude, sizeof(float) * waves);
   wavelength = (float*)realloc(wavelength, sizeof(float) * waves);
   frequency = (float*)realloc(frequency, sizeof(float) * waves);
   speedPC = (float*)realloc(speedPC, sizeof(float) * waves);
   direction = (glm::vec3*)realloc(direction, sizeof(glm::vec3) * waves);
   steepness = (float*)realloc(steepness, sizeof(float) * waves);
   
   amplitude[waves - 1] = a;
   wavelength[waves - 1] = w;
   frequency[waves - 1] = sqrt(9.81f * ((2.0f * M_PI) / w));
   speedPC[waves - 1] = s * sqrt(9.81f * ((2.0f * M_PI) / w));
   d = glm::normalize(d);
   direction[waves - 1] = d;
   steepness[waves - 1] = 1.0f / (sqrt(9.81f * ((2.0f * M_PI) / w)) * a * float(waves + 1));
   
   for (int i = 0; i < waves; i++) {
      steepness[i] *= float(waves);
      steepness[i] /= float(waves + 1);
   }
}

void GerstnerWave::addTriangles(std::vector<SceneObject*> *objects, float step, float time) {
   float depth = ub[2] - lb[2], width = ub[0] - lb[0];
   int matDepth = depth / step, matWidth = width / step;
   
   glm::vec3 wavePts[matDepth][matWidth];
   glm::vec3 waveNorms[matDepth][matWidth];
  
   //FILL MATS
   for (int i = 0; i < matDepth; i++) {
      for (int j = 0; j < matWidth; j++) {
         wavePts[i][j] = getPoint((j * step) + lb[0], (i * step) + lb[2], time);
         waveNorms[i][j] = getNormal(wavePts[i][j], time);
      }
   }
   
   //WRITE
   //EACH ITERATION = 2 TRIANGLES
   //SQUARES
   glm::vec3 a, b, c;
   Triangle *triangle;
   cout << matDepth << endl;
   cout << matWidth << endl;
   for (int i = 0; i < matDepth - 1; i++) {
      for (int j = 0; j < matWidth - 1; j++) {
         a = wavePts[i][j];
         b = wavePts[i+1][j];
         c = wavePts[i][j+1];
         
         triangle = new Triangle(a, b, c, true);
         triangle->aNor = waveNorms[i][j];
         triangle->bNor = waveNorms[i+1][j];
         triangle->cNor = waveNorms[i][j+1];
         
         triangle->pigment = pigment;
         triangle->refraction = refraction;
         triangle->indexRefraction = indexRefraction;
         triangle->photonReflectance = photonReflectance;
         triangle->photonRefractance = photonRefractance;
         triangle->dropoff = dropoff;
         triangle->type = 2;
         triangle->constructBB();
         
         objects->push_back(triangle);
         
         a = wavePts[i+1][j];
         b = wavePts[i+1][j+1];
         c = wavePts[i][j+1];
         
         triangle = new Triangle(a, b, c, true);
         triangle->aNor = waveNorms[i+1][j];
         triangle->bNor = waveNorms[i+1][j+1];
         triangle->cNor = waveNorms[i][j+1];
         
         triangle->pigment = pigment;
         triangle->refraction = refraction;
         triangle->indexRefraction = indexRefraction;
         triangle->photonReflectance = photonReflectance;
         triangle->photonRefractance = photonRefractance;
         triangle->dropoff = dropoff;
         triangle->type = 2;
         triangle->constructBB();
         
         objects->push_back(triangle);
      }
   }
   cout << objects->size() << endl;
}

void GerstnerWave::toPovFileMesh(char* fileName, float step, float time) {
   //GOING TO BUILD 2D MATRIX OF WAVE POINTS
   //WRITE INTO FILE TRIANGLES IN POV FORMAT
   
   float depth = ub[2] - lb[2], width = ub[0] - lb[0];
   int matDepth = depth / step, matWidth = width / step;
   
   glm::vec3 wavePts[matDepth][matWidth];
   glm::vec3 waveNorms[matDepth][matWidth];
   
   ofstream meshFile;
   meshFile.open(fileName);
   //myfile << "Writing this to a file.\n";
  
   //FILL MAT
   for (int i = 0; i < matDepth; i++) {
      for (int j = 0; j < matWidth; j++) {
         wavePts[i][j] = getPoint((j * step) + lb[0], (i * step) + lb[2], time);
         waveNorms[i][j] = getNormal(wavePts[i][j], time);
      }
   }
   
   //WRITE
   //EACH ITERATION = 2 TRIANGLES
   //SQUARES
   glm::vec3 a, b, c;
   for (int i = 0; i < matDepth - 1; i++) {
      for (int j = 0; j < matWidth - 1; j++) {
         a = wavePts[i][j];
         b = wavePts[i+1][j];
         c = wavePts[i][j+1];
         
         meshFile << "triangle {\n";
         meshFile << "<" << a.x << ", " << a.y << ", " << a.z << ">,\n";
         meshFile << "<" << b.x << ", " << b.y << ", " << b.z << ">,\n";
         meshFile << "<" << c.x << ", " << c.y << ", " << c.z << ">\n";
         meshFile << "pigment { color rgb <" << pigment.x << ", " << pigment.y << ", " << pigment.z << ">}\n";
         meshFile << "finish {refraction " << refraction << " ior " << indexRefraction << " preflect " << photonReflectance << " prefract " << photonRefractance << "}\n";
         meshFile << "}\n";
         
         a = wavePts[i+1][j];
         b = wavePts[i+1][j+1];
         c = wavePts[i][j+1];
         
         meshFile << "triangle {\n";
         meshFile << "<" << a.x << ", " << a.y << ", " << a.z << ">,\n";
         meshFile << "<" << b.x << ", " << b.y << ", " << b.z << ">,\n";
         meshFile << "<" << c.x << ", " << c.y << ", " << c.z << ">\n";
         meshFile << "pigment { color rgb <" << pigment.x << ", " << pigment.y << ", " << pigment.z << ">}\n";
         meshFile << "finish {refraction " << refraction << " ior " << indexRefraction << " preflect " << photonReflectance << " prefract " << photonRefractance << "}\n";
         meshFile << "}\n";
      }
   }
   
   meshFile.close();
}
