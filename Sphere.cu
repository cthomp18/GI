#include "hip/hip_runtime.h"
/*
   Cody Thompson
   CPE 473: Rendering
   Spring 2016
*/

#include "glm/glm.hpp"
#include "Sphere.h"

Sphere::Sphere(glm::vec3 pos, float rad) : SceneObject() {
   position = pos;
   radius = rad;
   
   checkCollision = &(checkSphereCollision);
   getNormal = &(getSphereNormal);
}

Sphere::Sphere() : SceneObject() {}
Sphere::~Sphere() {}

/*float Sphere::checkCollision(glm::vec3 start, glm::vec3 ray, float time) {
   float t = -1.0f, t0, t1, innerRoot, A, B, C;

   A = glm::dot(ray, ray);
   B = 2.0f * glm::dot(start - position, ray);
   C = glm::dot(start - position, start - position) - (radius * radius);
   innerRoot = (B * B) - (4.0f * A * C);
   
   if (innerRoot >= 0.0f) {
      t0 = (-B - sqrt(innerRoot)) / (2.0f * A);
      t1 = (-B + sqrt(innerRoot)) / (2.0f * A);
      if (t0 >= TOLERANCE && t0 < t1) {
         t = t0;
      } else if (t1 >= TOLERANCE) {
         t = t1;
      }
   }
   
   return t;
}*/

/*glm::vec3 Sphere::getNormal(glm::vec3 iPt, float time) {
   return (iPt - position) / radius;
}*/

void Sphere::constructBB() {
   boundingBox = BoundingBox(glm::vec3(position[0] - radius, position[1] - radius, position[2] - radius),
                             glm::vec3(position[0] + radius, position[1] + radius, position[2] + radius));
}
