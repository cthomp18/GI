#include "hip/hip_runtime.h"
/*
   Cody Thompson
   CPE 473: Rendering
   Spring 2016
*/

#include "Box.h"

Box::Box(glm::vec3 cornerPt1, glm::vec3 cornerPt2) : SceneObject() {
   for (int i = 0; i < 3; i++) {
      if (cornerPt1[i] < cornerPt2[i]) {
         minPt[i] = cornerPt1[i];
         maxPt[i] = cornerPt2[i];
      } else {
         minPt[i] = cornerPt2[i];
         maxPt[i] = cornerPt1[i];
      }
   }
   middle = glm::vec3((maxPt[0] + minPt[0]) / 2.0f, (maxPt[1] + minPt[1]) / 2.0f, (maxPt[2] + minPt[2]) / 2.0f);
   unit = false;
}

Box::Box() : SceneObject() {}
Box::~Box() {}

float Box::checkCollision(glm::vec3 start, glm::vec3 ray, float time) {
   //std::cout << "Box Collision" << std::endl;
   float tgmin = FLT_MIN, tgmax = FLT_MAX, t1, t2, temp, t = -1.0f;

   for (int i = 0; i < 3; i++) {
      temp = start[i];
      
      if (fabs(ray[i]) < TOLERANCE) { // Ray along 2D Plane checks
         if (temp > maxPt[i] || temp < minPt[i]) return -1.0f;
      }
      
      t1 = (minPt[i] - temp) / ray[i];
      t2 = (maxPt[i] - temp) / ray[i];
      if (t2 < t1) {
         temp = t2;
         t2 = t1;
         t1 = temp;
      }
      if (t1 > tgmin) tgmin = t1;
      if (t2 < tgmax) tgmax = t2;
   }
   
   /*if (start.x() >= minPt.x() && start.x() <= maxPt.x() &&
       start.y() >= minPt.y() && start.y() <= maxPt.y() && 
       start.z() >= minPt.z() && start.z() <= maxPt.z()) {
       t = 10.0f;
       
       std::cout << "Inside!!!" << std::endl;
   }*/
   //if (tgmin < TOLERANCE) return new Collision(tgmin, this);
   //if (tgmin > tgmax || tgmax < 0.001f) return new Collision(t, this);
   if (tgmin > tgmax) return -1.0f;
   if (tgmin < TOLERANCE) return tgmax;
   return tgmin;
}

glm::vec3 Box::getNormal(glm::vec3 iPt) {
   glm::vec3 normal = glm::vec3(0.0f, 0.0f, 0.0f);
   
   for (int i = 0; i < 3; i++) {
      if (fabs(iPt[i] - minPt[i]) < TOLERANCE) {
         normal[i] = -1.0f;
      } else if (fabs(iPt[i] - maxPt[i]) < TOLERANCE) {
         normal[i] = 1.0f;
      }
   }
   
   return normal;
}

void Box::constructBB() {
   boundingBox = BoundingBox(minPt, maxPt);
}
